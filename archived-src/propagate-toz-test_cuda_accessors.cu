#include "hip/hip_runtime.h"
/*
nvc++ -O2 -std=c++17 -stdpar=gpu -gpu=cc75 -gpu=managed -gpu=fma -gpu=fastmath -gpu=autocollapse -gpu=loadcache:L1 -gpu=unroll  src/propagate-tor-test_pstl.cpp   -o ./propagate_nvcpp_pstl
nvc++ -O2 -std=c++17 -stdpar=multicore src/propagate-tor-test_pstl.cpp   -o ./propagate_nvcpp_pstl 
g++ -O3 -I. -fopenmp -mavx512f -std=c++17 src/propagate-tor-test_pstl.cpp -lm -lgomp -Lpath-to-tbb-lib -ltbb  -o ./propagate_gcc_pstl
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <iostream>
#include <chrono>
#include <iomanip>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <cassert>

#include <algorithm>
#include <vector>
#include <memory>
#include <numeric>
#include <random>


#ifndef ntrks
#define ntrks 9600
#endif

//#define ntrks    (ntrks/bsize)

#ifndef nevts
#define nevts 100
#endif
#define smear 0.00001

#ifndef NITER
#define NITER 5
#endif
#ifndef nlayer
#define nlayer 20
#endif

#ifndef num_streams
#define num_streams 1
#endif

#ifndef threadsperblock
#define threadsperblock 32
#endif

namespace impl {

  /**
     Simple array object which mimics std::array
  */
  template <typename T, int n> struct array {
    using value_type = T;
    T data[n];

    constexpr T &operator[](int i) { return data[i]; }
    constexpr const T &operator[](int i) const { return data[i]; }
    constexpr int size() const { return n; }

    array() = default;
    array(const array<T, n> &) = default;
    array(array<T, n> &&) = default;

    array<T, n> &operator=(const array<T, n> &) = default;
    array<T, n> &operator=(array<T, n> &&) = default;
  };
  
  template<typename Tp>
  struct UVMAllocator {
    public:

      typedef Tp value_type;

      UVMAllocator () {};

      UVMAllocator(const UVMAllocator&) { }
       
      template<typename Tp1> constexpr UVMAllocator(const UVMAllocator<Tp1>&) { }

      ~UVMAllocator() { }

      Tp* address(Tp& x) const { return &x; }

      std::size_t  max_size() const throw() { return size_t(-1) / sizeof(Tp); }

      [[nodiscard]] Tp* allocate(std::size_t n){

        Tp* ptr = nullptr;

        auto err = hipMallocManaged((void **)&ptr,n*sizeof(Tp));

        if( err != hipSuccess ) {
          ptr = (Tp *) NULL;
          std::cerr << " hipMallocManaged failed for " << n*sizeof(Tp) << " bytes " <<hipGetErrorString(err)<< std::endl;
          assert(0);
        }

        return ptr;
      }
      void deallocate( Tp* p, std::size_t n) noexcept {
        hipFree((void *)p);
        return;
      }
    };
    
   template <typename IntType>
   class counting_iterator {
       static_assert(std::numeric_limits<IntType>::is_integer, "Cannot instantiate counting_iterator with a non-integer type");
     public:
       using value_type = IntType;
       using difference_type = typename std::make_signed<IntType>::type;
       using pointer = IntType*;
       using reference = IntType&;
       using iterator_category = std::random_access_iterator_tag;

       counting_iterator() : value(0) { }
       explicit counting_iterator(IntType v) : value(v) { }

       value_type operator*() const { return value; }
       value_type operator[](difference_type n) const { return value + n; }

       counting_iterator& operator++() { ++value; return *this; }
       counting_iterator operator++(int) {
         counting_iterator result{value};
         ++value;
         return result;
       }  
       counting_iterator& operator--() { --value; return *this; }
       counting_iterator operator--(int) {
         counting_iterator result{value};
         --value;
         return result;
       }
       counting_iterator& operator+=(difference_type n) { value += n; return *this; }
       counting_iterator& operator-=(difference_type n) { value -= n; return *this; }

       friend counting_iterator operator+(counting_iterator const& i, difference_type n)          { return counting_iterator(i.value + n);  }
       friend counting_iterator operator+(difference_type n, counting_iterator const& i)          { return counting_iterator(i.value + n);  }
       friend difference_type   operator-(counting_iterator const& x, counting_iterator const& y) { return x.value - y.value;  }
       friend counting_iterator operator-(counting_iterator const& i, difference_type n)          { return counting_iterator(i.value - n);  }

       friend bool operator==(counting_iterator const& x, counting_iterator const& y) { return x.value == y.value;  }
       friend bool operator!=(counting_iterator const& x, counting_iterator const& y) { return x.value != y.value;  }
       friend bool operator<(counting_iterator const& x, counting_iterator const& y)  { return x.value < y.value; }
       friend bool operator<=(counting_iterator const& x, counting_iterator const& y) { return x.value <= y.value; }
       friend bool operator>(counting_iterator const& x, counting_iterator const& y)  { return x.value > y.value; }
       friend bool operator>=(counting_iterator const& x, counting_iterator const& y) { return x.value >= y.value; }

     private:
       IntType value;
   };

} //impl


enum class FieldOrder{P2Z_TRACKBLK_EVENT_LAYER_MATIDX_ORDER,
                      P2Z_TRACKBLK_EVENT_MATIDX_LAYER_ORDER};
                      
enum class ConversionType{P2Z_CONVERT_TO_INTERNAL_ORDER, P2Z_CONVERT_FROM_INTERNAL_ORDER};   

const std::array<int, 36> SymOffsets66{0, 1, 3, 6, 10, 15, 1, 2, 4, 7, 11, 16, 3, 4, 5, 8, 12, 17, 6, 7, 8, 9, 13, 18, 10, 11, 12, 13, 14, 19, 15, 16, 17, 18, 19, 20};

struct ATRK {
  std::array<float,6> par;
  std::array<float,21> cov;
  int q;
};

struct AHIT {
  std::array<float,3> pos;
  std::array<float,6> cov;
};

constexpr int iparX     = 0;
constexpr int iparY     = 1;
constexpr int iparZ     = 2;
constexpr int iparIpt   = 3;
constexpr int iparPhi   = 4;
constexpr int iparTheta = 5;

template <typename T, int N>
struct MPNX_ {
   impl::array<T,N> data;
   //basic accessors
   __device__ __host__ inline const T& operator[](const int idx) const {return data[idx];}
   __device__ __host__ inline T& operator[](const int idx) {return data[idx];}
};

using MP1I_    = MPNX_<int,   1 >;
using MP1F_    = MPNX_<float, 1 >;
using MP2F_    = MPNX_<float, 2 >;
using MP3F_    = MPNX_<float, 3 >;
using MP6F_    = MPNX_<float, 6 >;
using MP2x2SF_ = MPNX_<float, 3 >;
using MP3x3SF_ = MPNX_<float, 6 >;
using MP6x6SF_ = MPNX_<float, 21>;
using MP6x6F_  = MPNX_<float, 36>;
using MP3x3_   = MPNX_<float, 9 >;
using MP3x6_   = MPNX_<float, 18>;

struct MPTRK_ {
  MP6F_    par;
  MP6x6SF_ cov;
  MP1I_    q;

  //  MP22I   hitidx;
};

struct MPHIT_ {
  MP3F_    pos;
  MP3x3SF_ cov;
};

using IntAllocator   = impl::UVMAllocator<int>;
using FloatAllocator = impl::UVMAllocator<float>;
using MPTRKAllocator = impl::UVMAllocator<MPTRK_>;
using MPHITAllocator = impl::UVMAllocator<MPHIT_>;

template <typename T, typename Allocator, int n>
struct MPNX {
   using DataType = T;

   static constexpr int N    = n;

   const int nTrks;//note that bSize is a tuning parameter!
   const int nEvts;
   const int nLayers;

   std::vector<T, Allocator> data;

   MPNX() : nTrks(0), nEvts(0), nLayers(0), data(n){}

   MPNX(const int ntrks_, const int nevts_, const int nlayers_ = 1) :
      nTrks(ntrks_),
      nEvts(nevts_),
      nLayers(nlayers_),
      data(n*nTrks*nEvts*nLayers){
   }

   MPNX(const std::vector<T, Allocator> data_, const int ntrks_, const int nevts_, const int nlayers_ = 1) :
      nTrks(ntrks_),
      nEvts(nevts_),
      nLayers(nlayers_),
      data(data_) {
     if(data_.size() > n*nTrks*nEvts*nLayers) {std::cerr << "Incorrect dim parameters."; }
   }
};

using MP1I    = MPNX<int,  IntAllocator,   1 >;
using MP1F    = MPNX<float,FloatAllocator, 1 >;
using MP2F    = MPNX<float,FloatAllocator, 2 >;
using MP3F    = MPNX<float,FloatAllocator, 3 >;
using MP6F    = MPNX<float,FloatAllocator, 6 >;
using MP3x3   = MPNX<float,FloatAllocator, 9 >;
using MP3x6   = MPNX<float,FloatAllocator, 18>;
using MP2x2SF = MPNX<float,FloatAllocator, 3 >;
using MP3x3SF = MPNX<float,FloatAllocator, 6 >;
using MP6x6SF = MPNX<float,FloatAllocator, 21>;
using MP6x6F  = MPNX<float,FloatAllocator, 36>;


template <typename MPNTp, FieldOrder Order>
struct MPNXAccessor {
   typedef typename MPNTp::DataType T;

   static constexpr int n   = MPNTp::N;//matrix linear dim (total number of els)

   int nTrks;
   int nEvts;
   int nLayers;

   int NevtsNtrks;

   int stride;
   
   int thread_stride;

   T* data_; //accessor field only for the data access, not allocated here

   MPNXAccessor() = default;

   MPNXAccessor(const MPNTp &v) :
        nTrks(v.nTrks),
        nEvts(v.nEvts),
        nLayers(v.nLayers),
        NevtsNtrks(nEvts*nTrks),
        stride(Order == FieldOrder::P2Z_TRACKBLK_EVENT_LAYER_MATIDX_ORDER ? nTrks*nEvts*nLayers  : nTrks*nEvts*n),
        thread_stride(Order == FieldOrder::P2Z_TRACKBLK_EVENT_LAYER_MATIDX_ORDER ? stride  : NevtsNtrks),              
        data_(const_cast<T*>(v.data.data())){ }

   __device__ __host__ inline T& operator[](const int idx) const {return data_[idx];}

   __device__ __host__ inline T& operator()(const int mat_idx, const int trkev_idx, const int layer_idx) const {
     if      constexpr (Order == FieldOrder::P2Z_TRACKBLK_EVENT_LAYER_MATIDX_ORDER)
       return data_[mat_idx*stride + layer_idx*NevtsNtrks + trkev_idx];//using defualt order batch id (the fastest) > track id > event id > layer id (the slowest)
     else //(Order == FieldOrder::P2Z_TRACKBLK_EVENT_MATIDX_LAYER_ORDER)
       return data_[layer_idx*stride + mat_idx*NevtsNtrks + trkev_idx];
   }//i is the internal dof index

   __device__ __host__ inline T& operator()(const int thrd_idx, const int blk_offset) const { return data_[thrd_idx*thread_stride + blk_offset];}//

   __device__ __host__ inline int GetThreadOffset(const int thrd_idx, const int layer_idx = 0) const {
     if      constexpr (Order == FieldOrder::P2Z_TRACKBLK_EVENT_LAYER_MATIDX_ORDER)
       return (layer_idx*NevtsNtrks + thrd_idx);//using defualt order batch id (the fastest) > track id > event id > layer id (the slowest)
     else //(Order == FieldOrder::P2Z_TRACKBLK_EVENT_MATIDX_LAYER_ORDER)
       return (layer_idx*stride + thrd_idx);
   }
   
   __device__ __host__ inline void load(MPNX_<T, n>& dest, const int tid, const int layer = 0) const {
      auto tid_offset = GetThreadOffset(tid, layer);
#pragma unroll
      for(int id = 0; id < n; id++){
          dest[id] = this->operator()(id, tid_offset);
      }
      return;
   }
   __device__ __host__ inline void save(const MPNX_<T, n>& src, const int tid, const int layer = 0){
      auto tid_offset = GetThreadOffset(tid, layer); 
#pragma unroll
      for(int id = 0; id < n; id++){
        this->operator()(id, tid_offset) = src[id];
      }
      return;
   }  
  
};

struct MPTRK {
  MP6F    par;
  MP6x6SF cov;
  MP1I    q;

  MPTRK() : par(), cov(), q() {}
  MPTRK(const int ntrks_, const int nevts_) : par(ntrks_, nevts_), cov(ntrks_, nevts_), q(ntrks_, nevts_) {}
  //  MP22I   hitidx;
};

template <FieldOrder Order>
struct MPTRKAccessor {
  using MP6FAccessor   = MPNXAccessor<MP6F,    Order>;
  using MP6x6SFAccessor= MPNXAccessor<MP6x6SF, Order>;
  using MP1IAccessor   = MPNXAccessor<MP1I,    Order>;

  MP6FAccessor    par;
  MP6x6SFAccessor cov;
  MP1IAccessor    q;

  MPTRKAccessor() : par(), cov(), q() {}
  MPTRKAccessor(const MPTRK &in) : par(in.par), cov(in.cov), q(in.q) {}
  
   __device__ __host__ inline const auto load(const int tid) const {
    MPTRK_ dst;

    this->par.load(dst.par, tid, 0);
    this->cov.load(dst.cov, tid, 0);
    this->q.load(dst.q, tid, 0);
    
    return dst;
  }
  
  __device__ __host__ inline void save(MPTRK_ &src, const int tid, const int layer = 0) {
    this->par.save(src.par, tid, layer);
    this->cov.save(src.cov, tid, layer);
    this->q.save(src.q, tid, layer);
    
    return;
  }
};

struct MPHIT {
  MP3F    pos;
  MP3x3SF cov;

  MPHIT() : pos(), cov(){}
  MPHIT(const int ntrks_, const int nevts_, const int nlayers_) : pos(ntrks_, nevts_, nlayers_), cov(ntrks_, nevts_, nlayers_) {}
};

template <FieldOrder Order>
struct MPHITAccessor {
  using MP3FAccessor   = MPNXAccessor<MP3F,    Order>;
  using MP3x3SFAccessor= MPNXAccessor<MP3x3SF, Order>;

  MP3FAccessor    pos;
  MP3x3SFAccessor cov;

  MPHITAccessor() : pos(), cov() {}
  MPHITAccessor(const MPHIT &in) : pos(in.pos), cov(in.cov) {}
  
  __device__ __host__ inline const auto load(const int tid, const int layer = 0) const {
    MPHIT_ dst;

    this->pos.load(dst.pos, tid, layer);
    this->cov.load(dst.cov, tid, layer);
    
    return dst;
  } 
};


template<FieldOrder order, ConversionType convers_tp>
void convertTracks(std::vector<MPTRK_> &external_order_data, MPTRK* internal_order_data) {
  //create an accessor field:
  std::unique_ptr<MPTRKAccessor<order>> ind(new MPTRKAccessor<order>(*internal_order_data));
  // store in element order for bunches of bsize matrices (a la matriplex)
  const int outer_loop_range = nevts*ntrks;
  //
  std::for_each(impl::counting_iterator(0),
                impl::counting_iterator(outer_loop_range),
                [=, exd_ = external_order_data.data(), &ind_ = *ind] (const auto tid) {
                  {
                  //const int l = it+ib*bsize+ie*ntrks*bsize;
                    //par
    	            for (int ip=0;ip<6;++ip) {
    	              if constexpr (convers_tp == ConversionType::P2Z_CONVERT_FROM_INTERNAL_ORDER)
    	                exd_[tid].par.data[ip] = ind_.par(ip, tid, 0);
    	              else
    	                ind_.par(ip, tid, 0) = exd_[tid].par.data[ip];  
    	            }
    	            //cov
    	            for (int ip=0;ip<21;++ip) {
    	              if constexpr (convers_tp == ConversionType::P2Z_CONVERT_FROM_INTERNAL_ORDER)
    	                exd_[tid].cov.data[ip] = ind_.cov(ip, tid, 0);
    	              else
    	                ind_.cov(ip, tid, 0) = exd_[tid].cov.data[ip];
    	            }
    	            //q
    	            if constexpr (convers_tp == ConversionType::P2Z_CONVERT_FROM_INTERNAL_ORDER)
    	              exd_[tid].q.data[0] = ind_.q(0, tid, 0);//fixme check
    	            else
    	              ind_.q(0, tid, 0) = exd_[tid].q.data[0];
                  }
                });
   //
   return;
}


template<FieldOrder order, ConversionType convers_tp>
void convertHits(std::vector<MPHIT_> &external_order_data, MPHIT* internal_order_data) {
  //create an accessor field:
  std::unique_ptr<MPHITAccessor<order>> ind(new MPHITAccessor<order>(*internal_order_data));
  // store in element order for bunches of bsize matrices (a la matriplex)
  const int outer_loop_range = nevts*ntrks;
  
  std::for_each(impl::counting_iterator(0),
                impl::counting_iterator(outer_loop_range),
                [=, exd_ = external_order_data.data(), &ind_ = *ind] (const auto tid) {
                   //  
                   for(int layer=0; layer<nlayer; ++layer) {  
                     {
                       //pos
                       for (int ip=0;ip<3;++ip) {
                         if constexpr (convers_tp == ConversionType::P2Z_CONVERT_FROM_INTERNAL_ORDER)
                           exd_[layer+nlayer*tid].pos.data[ip] = ind_.pos(ip, tid, layer);
                         else
                           ind_.pos(ip, tid, layer) = exd_[layer+nlayer*tid].pos.data[ip];
                       }
                       //cov
                       for (int ip=0;ip<6;++ip) {
                         if constexpr (convers_tp == ConversionType::P2Z_CONVERT_FROM_INTERNAL_ORDER)
                           exd_[layer+nlayer*tid].cov.data[ip] = ind_.cov(ip, tid, layer);
                         else
                           ind_.cov(ip, tid, layer) = exd_[layer+nlayer*tid].cov.data[ip];
                       }
                     } 
                  }
               });
  
  return;
}

///////////////////////////////////////
//Gen. utils

float randn(float mu, float sigma) {
  float U1, U2, W, mult;
  static float X1, X2;
  static int call = 0;
  if (call == 1) {
    call = !call;
    return (mu + sigma * (float) X2);
  } do {
    U1 = -1 + ((float) rand () / RAND_MAX) * 2;
    U2 = -1 + ((float) rand () / RAND_MAX) * 2;
    W = pow (U1, 2) + pow (U2, 2);
  }
  while (W >= 1 || W == 0); 
  mult = sqrt ((-2 * log (W)) / W);
  X1 = U1 * mult;
  X2 = U2 * mult; 
  call = !call; 
  return (mu + sigma * (float) X1);
}


void prepareTracks(std::vector<MPTRK_> &trcks, ATRK &inputtrk) {
  //
  for (int ie=0;ie<nevts;++ie) {
    for (int ib=0;ib<ntrks;++ib) {
      {
	      //par
	      for (int ip=0;ip<6;++ip) {
	        trcks[ib + ntrks*ie].par.data[ip] = (1+smear*randn(0,1))*inputtrk.par[ip];
	      }
	      //cov, scale by factor 100
	      for (int ip=0;ip<21;++ip) {
	        trcks[ib + ntrks*ie].cov.data[ip] = (1+smear*randn(0,1))*inputtrk.cov[ip]*100;
	      }
	      //q
	      trcks[ib + ntrks*ie].q.data[0] = inputtrk.q;//can't really smear this or fit will be wrong
      }
    }
  }
  //
  return;
}

void prepareHits(std::vector<MPHIT_> &hits, std::vector<AHIT>& inputhits) {
  // store in element order for bunches of bsize matrices (a la matriplex)
  for (int lay=0;lay<nlayer;++lay) {

    int mylay = lay;
    if (lay>=inputhits.size()) {
      // int wraplay = inputhits.size()/lay;
      exit(1);
    }
    AHIT& inputhit = inputhits[mylay];

    for (int ie=0;ie<nevts;++ie) {
      for (int ib=0;ib<ntrks;++ib) {
        {
        	//pos
        	for (int ip=0;ip<3;++ip) {
        	  hits[lay+nlayer*(ib + ntrks*ie)].pos.data[ip] = (1+smear*randn(0,1))*inputhit.pos[ip];
        	}
        	//cov
        	for (int ip=0;ip<6;++ip) {
        	  hits[lay+nlayer*(ib + ntrks*ie)].cov.data[ip] = (1+smear*randn(0,1))*inputhit.cov[ip];
        	}
        }
      }
    }
  }
  return;
}


//////////////////////////////////////////////////////////////////////////////////////
// Aux utils 
MPTRK_* bTk(MPTRK_* tracks, int ev, int ib) {
  return &(tracks[ib + ntrks*ev]);
}

const MPTRK_* bTk(const MPTRK_* tracks, int ev, int ib) {
  return &(tracks[ib + ntrks*ev]);
}

float q(const MP1I_* bq, int it){
  return (*bq).data[0];
}
//
float par(const MP6F_* bpars, int it, int ipar){
  return (*bpars).data[it + ipar];
}
float x    (const MP6F_* bpars, int it){ return par(bpars, it, 0); }
float y    (const MP6F_* bpars, int it){ return par(bpars, it, 1); }
float z    (const MP6F_* bpars, int it){ return par(bpars, it, 2); }
float ipt  (const MP6F_* bpars, int it){ return par(bpars, it, 3); }
float phi  (const MP6F_* bpars, int it){ return par(bpars, it, 4); }
float theta(const MP6F_* bpars, int it){ return par(bpars, it, 5); }
//
float par(const MPTRK_* btracks, int it, int ipar){
  return par(&(*btracks).par,it,ipar);
}
float x    (const MPTRK_* btracks, int it){ return par(btracks, it, 0); }
float y    (const MPTRK_* btracks, int it){ return par(btracks, it, 1); }
float z    (const MPTRK_* btracks, int it){ return par(btracks, it, 2); }
float ipt  (const MPTRK_* btracks, int it){ return par(btracks, it, 3); }
float phi  (const MPTRK_* btracks, int it){ return par(btracks, it, 4); }
float theta(const MPTRK_* btracks, int it){ return par(btracks, it, 5); }
//
float par(const MPTRK_* tracks, int ev, int tk, int ipar){
  int ib = tk;
  const MPTRK_* btracks = bTk(tracks, ev, ib);
  int it = 0;
  return par(btracks, it, ipar);
}
float x    (const MPTRK_* tracks, int ev, int tk){ return par(tracks, ev, tk, 0); }
float y    (const MPTRK_* tracks, int ev, int tk){ return par(tracks, ev, tk, 1); }
float z    (const MPTRK_* tracks, int ev, int tk){ return par(tracks, ev, tk, 2); }
float ipt  (const MPTRK_* tracks, int ev, int tk){ return par(tracks, ev, tk, 3); }
float phi  (const MPTRK_* tracks, int ev, int tk){ return par(tracks, ev, tk, 4); }
float theta(const MPTRK_* tracks, int ev, int tk){ return par(tracks, ev, tk, 5); }
//

const MPHIT_* bHit(const MPHIT_* hits, int ev, int ib) {
  return &(hits[ib + ntrks*ev]);
}
const MPHIT_* bHit(const MPHIT_* hits, int ev, int ib,int lay) {
return &(hits[lay + (ib*nlayer) +(ev*nlayer*ntrks)]);
}
//
float Pos(const MP3F_* hpos, int it, int ipar){
  return (*hpos).data[it + ipar];
}
float x(const MP3F_* hpos, int it)    { return Pos(hpos, it, 0); }
float y(const MP3F_* hpos, int it)    { return Pos(hpos, it, 1); }
float z(const MP3F_* hpos, int it)    { return Pos(hpos, it, 2); }
//
float Pos(const MPHIT_* hits, int it, int ipar){
  return Pos(&(*hits).pos,it,ipar);
}
float x(const MPHIT_* hits, int it)    { return Pos(hits, it, 0); }
float y(const MPHIT_* hits, int it)    { return Pos(hits, it, 1); }
float z(const MPHIT_* hits, int it)    { return Pos(hits, it, 2); }
//
float Pos(const MPHIT_* hits, int ev, int tk, int ipar){
  int ib = tk;
  const MPHIT_* bhits = bHit(hits, ev, ib);
  int it = 0;
  return Pos(bhits,it,ipar);
}
float x(const MPHIT_* hits, int ev, int tk)    { return Pos(hits, ev, tk, 0); }
float y(const MPHIT_* hits, int ev, int tk)    { return Pos(hits, ev, tk, 1); }
float z(const MPHIT_* hits, int ev, int tk)    { return Pos(hits, ev, tk, 2); }


////////////////////////////////////////////////////////////////////////
///MAIN compute kernels

__device__ inline void MultHelixPropEndcap(const MP6x6F_ &a, const MP6x6SF_ &b, MP6x6F_ &c) {
  {
    c[ 0] = b[ 0] + a[ 2]*b[ 3] + a[ 3]*b[ 6] + a[ 4]*b[10] + a[ 5]*b[15];
    c[ 1] = b[ 1] + a[ 2]*b[ 4] + a[ 3]*b[ 7] + a[ 4]*b[11] + a[ 5]*b[16];
    c[ 2] = b[ 3] + a[ 2]*b[ 5] + a[ 3]*b[ 8] + a[ 4]*b[12] + a[ 5]*b[17];
    c[ 3] = b[ 6] + a[ 2]*b[ 8] + a[ 3]*b[ 9] + a[ 4]*b[13] + a[ 5]*b[18];
    c[ 4] = b[10] + a[ 2]*b[12] + a[ 3]*b[13] + a[ 4]*b[14] + a[ 5]*b[19];
    c[ 5] = b[15] + a[ 2]*b[17] + a[ 3]*b[18] + a[ 4]*b[19] + a[ 5]*b[20];
    c[ 6] = b[ 1] + a[ 8]*b[ 3] + a[ 9]*b[ 6] + a[10]*b[10] + a[11]*b[15];
    c[ 7] = b[ 2] + a[ 8]*b[ 4] + a[ 9]*b[ 7] + a[10]*b[11] + a[11]*b[16];
    c[ 8] = b[ 4] + a[ 8]*b[ 5] + a[ 9]*b[ 8] + a[10]*b[12] + a[11]*b[17];
    c[ 9] = b[ 7] + a[ 8]*b[ 8] + a[ 9]*b[ 9] + a[10]*b[13] + a[11]*b[18];
    c[10] = b[11] + a[ 8]*b[12] + a[ 9]*b[13] + a[10]*b[14] + a[11]*b[19];
    c[11] = b[16] + a[ 8]*b[17] + a[ 9]*b[18] + a[10]*b[19] + a[11]*b[20];
    c[12] = 0.f;
    c[13] = 0.f;
    c[14] = 0.f;
    c[15] = 0.f;
    c[16] = 0.f;
    c[17] = 0.f;
    c[18] = b[ 6];
    c[19] = b[ 7];
    c[20] = b[ 8];
    c[21] = b[ 9];
    c[22] = b[13];
    c[23] = b[18];
    c[24] = a[26]*b[ 3] + a[27]*b[ 6] + b[10] + a[29]*b[15];
    c[25] = a[26]*b[ 4] + a[27]*b[ 7] + b[11] + a[29]*b[16];
    c[26] = a[26]*b[ 5] + a[27]*b[ 8] + b[12] + a[29]*b[17];
    c[27] = a[26]*b[ 8] + a[27]*b[ 9] + b[13] + a[29]*b[18];
    c[28] = a[26]*b[12] + a[27]*b[13] + b[14] + a[29]*b[19];
    c[29] = a[26]*b[17] + a[27]*b[18] + b[19] + a[29]*b[20];
    c[30] = b[15];
    c[31] = b[16];
    c[32] = b[17];
    c[33] = b[18];
    c[34] = b[19];
    c[35] = b[20];
  }
  return;
}

__device__ inline void MultHelixPropTranspEndcap(const MP6x6F_ &a, const MP6x6F_ &b, MP6x6SF_ &c) {
  {
    c[ 0] = b[ 0] + b[ 2]*a[ 2] + b[ 3]*a[ 3] + b[ 4]*a[ 4] + b[ 5]*a[ 5];
    c[ 1] = b[ 6] + b[ 8]*a[ 2] + b[ 9]*a[ 3] + b[10]*a[ 4] + b[11]*a[ 5];
    c[ 2] = b[ 7] + b[ 8]*a[ 8] + b[ 9]*a[ 9] + b[10]*a[10] + b[11]*a[11];
    c[ 3] = b[12] + b[14]*a[ 2] + b[15]*a[ 3] + b[16]*a[ 4] + b[17]*a[ 5];
    c[ 4] = b[13] + b[14]*a[ 8] + b[15]*a[ 9] + b[16]*a[10] + b[17]*a[11];
    c[ 5] = 0.f;
    c[ 6] = b[18] + b[20]*a[ 2] + b[21]*a[ 3] + b[22]*a[ 4] + b[23]*a[ 5];
    c[ 7] = b[19] + b[20]*a[ 8] + b[21]*a[ 9] + b[22]*a[10] + b[23]*a[11];
    c[ 8] = 0.f;
    c[ 9] = b[21];
    c[10] = b[24] + b[26]*a[ 2] + b[27]*a[ 3] + b[28]*a[ 4] + b[29]*a[ 5];
    c[11] = b[25] + b[26]*a[ 8] + b[27]*a[ 9] + b[28]*a[10] + b[29]*a[11];
    c[12] = 0.f;
    c[13] = b[27];
    c[14] = b[26]*a[26] + b[27]*a[27] + b[28] + b[29]*a[29];
    c[15] = b[30] + b[32]*a[ 2] + b[33]*a[ 3] + b[34]*a[ 4] + b[35]*a[ 5];
    c[16] = b[31] + b[32]*a[ 8] + b[33]*a[ 9] + b[34]*a[10] + b[35]*a[11];
    c[17] = 0.f;
    c[18] = b[33];
    c[19] = b[32]*a[26] + b[33]*a[27] + b[34] + b[35]*a[29];
    c[20] = b[35];
  }
  return;
}

__device__ inline void KalmanGainInv(const MP6x6SF_ &a, const MP3x3SF_ &b, MP3x3_ &c) {

  {
    double det =
      ((a[0]+b[0])*(((a[ 6]+b[ 3]) *(a[11]+b[5])) - ((a[7]+b[4]) *(a[7]+b[4])))) -
      ((a[1]+b[1])*(((a[ 1]+b[ 1]) *(a[11]+b[5])) - ((a[7]+b[4]) *(a[2]+b[2])))) +
      ((a[2]+b[2])*(((a[ 1]+b[ 1]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[6]+b[3]))));
    double invdet = 1.0/det;

    c[ 0] =   invdet*(((a[ 6]+b[ 3]) *(a[11]+b[5])) - ((a[7]+b[4]) *(a[7]+b[4])));
    c[ 1] =  -invdet*(((a[ 1]+b[ 1]) *(a[11]+b[5])) - ((a[2]+b[2]) *(a[7]+b[4])));
    c[ 2] =   invdet*(((a[ 1]+b[ 1]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[7]+b[4])));
    c[ 3] =  -invdet*(((a[ 1]+b[ 1]) *(a[11]+b[5])) - ((a[7]+b[4]) *(a[2]+b[2])));
    c[ 4] =   invdet*(((a[ 0]+b[ 0]) *(a[11]+b[5])) - ((a[2]+b[2]) *(a[2]+b[2])));
    c[ 5] =  -invdet*(((a[ 0]+b[ 0]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[1]+b[1])));
    c[ 6] =   invdet*(((a[ 1]+b[ 1]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[6]+b[3])));
    c[ 7] =  -invdet*(((a[ 0]+b[ 0]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[1]+b[1])));
    c[ 8] =   invdet*(((a[ 0]+b[ 0]) *(a[6]+b[3])) - ((a[1]+b[1]) *(a[1]+b[1])));
  }
  
  return;
}

__device__ inline void KalmanGain(const MP6x6SF_ &a, const MP3x3_ &b, MP3x6_ &c) {

  {
    c[ 0] = a[0]*b[0] + a[ 1]*b[3] + a[2]*b[6];
    c[ 1] = a[0]*b[1] + a[ 1]*b[4] + a[2]*b[7];
    c[ 2] = a[0]*b[2] + a[ 1]*b[5] + a[2]*b[8];
    c[ 3] = a[1]*b[0] + a[ 6]*b[3] + a[7]*b[6];
    c[ 4] = a[1]*b[1] + a[ 6]*b[4] + a[7]*b[7];
    c[ 5] = a[1]*b[2] + a[ 6]*b[5] + a[7]*b[8];
    c[ 6] = a[2]*b[0] + a[ 7]*b[3] + a[11]*b[6];
    c[ 7] = a[2]*b[1] + a[ 7]*b[4] + a[11]*b[7];
    c[ 8] = a[2]*b[2] + a[ 7]*b[5] + a[11]*b[8];
    c[ 9] = a[3]*b[0] + a[ 8]*b[3] + a[12]*b[6];
    c[10] = a[3]*b[1] + a[ 8]*b[4] + a[12]*b[7];
    c[11] = a[3]*b[2] + a[ 8]*b[5] + a[12]*b[8];
    c[12] = a[4]*b[0] + a[ 9]*b[3] + a[13]*b[6];
    c[13] = a[4]*b[1] + a[ 9]*b[4] + a[13]*b[7];
    c[14] = a[4]*b[2] + a[ 9]*b[5] + a[13]*b[8];
    c[15] = a[5]*b[0] + a[10]*b[3] + a[14]*b[6];
    c[16] = a[5]*b[1] + a[10]*b[4] + a[14]*b[7];
    c[17] = a[5]*b[2] + a[10]*b[5] + a[14]*b[8];
  }
  
  return;
}

__device__ void KalmanUpdate(MP6x6SF_ &trkErr, MP6F_ &inPar, const MP3x3SF_ &hitErr, const MP3F_ &msP){

  MP3x3_ inverse_temp;
  MP3x6_ kGain;
  MP6x6SF_ newErr;
  
  KalmanGainInv(trkErr, hitErr, inverse_temp);
  KalmanGain(trkErr, inverse_temp, kGain);

  {
    const auto xin     = inPar[iparX];
    const auto yin     = inPar[iparY];
    const auto zin     = inPar[iparZ];
    const auto ptin    = 1.f/ inPar[iparIpt];
    const auto phiin   = inPar[iparPhi];
    const auto thetain = inPar[iparTheta];
    const auto xout    = msP[iparX];
    const auto yout    = msP[iparY];
    //const auto zout    = msP[iparZ];

    auto xnew     = xin + (kGain[0]*(xout-xin)) +(kGain[1]*(yout-yin)); 
    auto ynew     = yin + (kGain[3]*(xout-xin)) +(kGain[4]*(yout-yin)); 
    auto znew     = zin + (kGain[6]*(xout-xin)) +(kGain[7]*(yout-yin)); 
    auto ptnew    = ptin + (kGain[9]*(xout-xin)) +(kGain[10]*(yout-yin)); 
    auto phinew   = phiin + (kGain[12]*(xout-xin)) +(kGain[13]*(yout-yin)); 
    auto thetanew = thetain + (kGain[15]*(xout-xin)) +(kGain[16]*(yout-yin)); 

    newErr[ 0] = trkErr[ 0] - (kGain[ 0]*trkErr[0]+kGain[1]*trkErr[1]+kGain[2]*trkErr[2]);
    newErr[ 1] = trkErr[ 1] - (kGain[ 0]*trkErr[1]+kGain[1]*trkErr[6]+kGain[2]*trkErr[7]);
    newErr[ 2] = trkErr[ 2] - (kGain[ 0]*trkErr[2]+kGain[1]*trkErr[7]+kGain[2]*trkErr[11]);
    newErr[ 3] = trkErr[ 3] - (kGain[ 0]*trkErr[3]+kGain[1]*trkErr[8]+kGain[2]*trkErr[12]);
    newErr[ 4] = trkErr[ 4] - (kGain[ 0]*trkErr[4]+kGain[1]*trkErr[9]+kGain[2]*trkErr[13]);
    newErr[ 5] = trkErr[ 5] - (kGain[ 0]*trkErr[5]+kGain[1]*trkErr[10]+kGain[2]*trkErr[14]);

    newErr[ 6] = trkErr[ 6] - (kGain[ 3]*trkErr[1]+kGain[4]*trkErr[6]+kGain[5]*trkErr[7]);
    newErr[ 7] = trkErr[ 7] - (kGain[ 3]*trkErr[2]+kGain[4]*trkErr[7]+kGain[5]*trkErr[11]);
    newErr[ 8] = trkErr[ 8] - (kGain[ 3]*trkErr[3]+kGain[4]*trkErr[8]+kGain[5]*trkErr[12]);
    newErr[ 9] = trkErr[ 9] - (kGain[ 3]*trkErr[4]+kGain[4]*trkErr[9]+kGain[5]*trkErr[13]);
    newErr[10] = trkErr[10] - (kGain[ 3]*trkErr[5]+kGain[4]*trkErr[10]+kGain[5]*trkErr[14]);

    newErr[11] = trkErr[11] - (kGain[ 6]*trkErr[2]+kGain[7]*trkErr[7]+kGain[8]*trkErr[11]);
    newErr[12] = trkErr[12] - (kGain[ 6]*trkErr[3]+kGain[7]*trkErr[8]+kGain[8]*trkErr[12]);
    newErr[13] = trkErr[13] - (kGain[ 6]*trkErr[4]+kGain[7]*trkErr[9]+kGain[8]*trkErr[13]);
    newErr[14] = trkErr[14] - (kGain[ 6]*trkErr[5]+kGain[7]*trkErr[10]+kGain[8]*trkErr[14]);

    newErr[15] = trkErr[15] - (kGain[ 9]*trkErr[3]+kGain[10]*trkErr[8]+kGain[11]*trkErr[12]);
    newErr[16] = trkErr[16] - (kGain[ 9]*trkErr[4]+kGain[10]*trkErr[9]+kGain[11]*trkErr[13]);
    newErr[17] = trkErr[17] - (kGain[ 9]*trkErr[5]+kGain[10]*trkErr[10]+kGain[11]*trkErr[14]);

    newErr[18] = trkErr[18] - (kGain[12]*trkErr[4]+kGain[13]*trkErr[9]+kGain[14]*trkErr[13]);
    newErr[19] = trkErr[19] - (kGain[12]*trkErr[5]+kGain[13]*trkErr[10]+kGain[14]*trkErr[14]);

    newErr[20] = trkErr[20] - (kGain[15]*trkErr[5]+kGain[16]*trkErr[10]+kGain[17]*trkErr[14]);
    
    inPar[iparX]     = xnew;
    inPar[iparY]     = ynew;
    inPar[iparZ]     = znew;
    inPar[iparIpt]   = ptnew;
    inPar[iparPhi]   = phinew;
    inPar[iparTheta] = thetanew;
    
 #pragma unroll
    for (int i = 0; i < 21; i++){
      trkErr[ i] = trkErr[ i] - newErr[ i];
    }

  }
  
  return;
}              

//constexpr auto kfact= 100/(-0.299792458*3.8112);
constexpr auto kfact= 100/3.8;

__device__ void propagateToZ(const MP6x6SF_ &inErr, const MP6F_ &inPar, const MP1I_ &inChg, 
                  const MP3F_ &msP, MP6x6SF_ &outErr, MP6F_ &outPar) {
  
  MP6x6F_ errorProp;
  MP6x6F_ temp;
 
  auto PosInMtrx = [=] (int i, int j, int D) constexpr {return (i*D+j);};
//#pragma omp simd
  {	
    const auto zout = msP[iparZ];
    //note: in principle charge is not needed and could be the sign of ipt
    const auto k = inChg[0]*kfact;
    const auto deltaZ = zout - inPar[iparZ];
    const auto ipt  = inPar[iparIpt];
    const auto pt   = 1.f/ipt;
    const auto phi  = inPar[iparPhi];
    const auto cosP = cosf(phi);
    const auto sinP = sinf(phi);
    const auto theta= inPar[iparTheta];
    const auto cosT = cosf(theta);
    const auto sinT = sinf(theta);
    const auto pxin = cosP*pt;
    const auto pyin = sinP*pt;
    const auto icosT  = 1.f/cosT;
    const auto icosTk = icosT/k;
    const auto alpha  = deltaZ*sinT*ipt*icosTk;
    //const auto alpha = deltaZ*sinT*ipt(inPar]/(cosT*k);
    const auto sina = sinf(alpha); // this can be approximated;
    const auto cosa = cosf(alpha); // this can be approximated;
    //
    outPar[iparX]     = inPar[iparX] + k*(pxin*sina - pyin*(1.f-cosa));
    outPar[iparY]     = inPar[iparY] + k*(pyin*sina + pxin*(1.f-cosa));
    outPar[iparZ]     = zout;
    outPar[iparIpt]   = ipt;
    outPar[iparPhi]   = phi +alpha;
    outPar[iparTheta] = theta;
    
    const auto sCosPsina = sinf(cosP*sina);
    const auto cCosPsina = cosf(cosP*sina);
    
    //for (size_t i=0;i<6;++i) errorProp[bsize*PosInMtrx(i,i,6) + it] = 1.;
    errorProp[PosInMtrx(0,0,6)] = 1.0f;
    errorProp[PosInMtrx(1,1,6)] = 1.0f;
    errorProp[PosInMtrx(2,2,6)] = 1.0f;
    errorProp[PosInMtrx(3,3,6)] = 1.0f;
    errorProp[PosInMtrx(4,4,6)] = 1.0f;
    errorProp[PosInMtrx(5,5,6)] = 1.0f;
    //
    errorProp[PosInMtrx(0,1,6)] = 0.f;
    errorProp[PosInMtrx(0,2,6)] = cosP*sinT*(sinP*cosa*sCosPsina-cosa)*icosT;
    errorProp[PosInMtrx(0,3,6)] = cosP*sinT*deltaZ*cosa*(1.f-sinP*sCosPsina)*(icosT*pt)-k*(cosP*sina-sinP*(1.f-cCosPsina))*(pt*pt);
    errorProp[PosInMtrx(0,4,6)] = (k*pt)*(-sinP*sina+sinP*sinP*sina*sCosPsina-cosP*(1.f-cCosPsina));
    errorProp[PosInMtrx(0,5,6)] = cosP*deltaZ*cosa*(1.f-sinP*sCosPsina)*(icosT*icosT);
    errorProp[PosInMtrx(1,2,6)] = cosa*sinT*(cosP*cosP*sCosPsina-sinP)*icosT;
    errorProp[PosInMtrx(1,3,6)] = sinT*deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*pt)-k*(sinP*sina+cosP*(1.f-cCosPsina))*(pt*pt);
    errorProp[PosInMtrx(1,4,6)] = (k*pt)*(-sinP*(1.f-cCosPsina)-sinP*cosP*sina*sCosPsina+cosP*sina);
    errorProp[PosInMtrx(1,5,6)] = deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*icosT);
    errorProp[PosInMtrx(4,2,6)] = -ipt*sinT*(icosTk);//!
    errorProp[PosInMtrx(4,3,6)] = sinT*deltaZ*(icosTk);
    errorProp[PosInMtrx(4,5,6)] = ipt*deltaZ*(icosT*icosTk);//!
  }
  
  MultHelixPropEndcap(errorProp, inErr, temp);
  MultHelixPropTranspEndcap(errorProp, temp, outErr);
  
  return;
}


template <int layers, FieldOrder order = FieldOrder::P2Z_TRACKBLK_EVENT_LAYER_MATIDX_ORDER, bool grid_stride = true>
__global__ void launch_p2z_kernels(MPTRKAccessor<order> &obtracksAcc, MPTRKAccessor<order> &btracksAcc, MPHITAccessor<order> &bhitsAcc, const int length){
   auto i = threadIdx.x + blockIdx.x * blockDim.x;

   MPTRK_ obtracks;

   while (i < length) {
     //
     const auto& btracks = btracksAcc.load(i);
     
     for(int layer = 0; layer < layers; ++layer) {  
       //
       const auto& bhits = bhitsAcc.load(i, layer);
       //
       propagateToZ(btracks.cov, btracks.par, btracks.q, bhits.pos, obtracks.cov, obtracks.par);
       KalmanUpdate(obtracks.cov, obtracks.par, bhits.cov, bhits.pos);
       //
     }
     //
     obtracksAcc.save(obtracks, i);
     
     if constexpr (grid_stride) i += gridDim.x * blockDim.x;
     else break;
  }
  return;
}

void p2z_check_error(){
  //	
  auto error = hipGetLastError();
  if(error != hipSuccess) std::cout << "Error detected, error " << error << std::endl;
  //
  return;
}


int main (int argc, char* argv[]) {

   #include "input_track.h"

   std::vector<AHIT> inputhits{inputhit21,inputhit20,inputhit19,inputhit18,inputhit17,inputhit16,inputhit15,inputhit14,
                               inputhit13,inputhit12,inputhit11,inputhit10,inputhit09,inputhit08,inputhit07,inputhit06,
                               inputhit05,inputhit04,inputhit03,inputhit02,inputhit01,inputhit00};

   printf("track in pos: x=%f, y=%f, z=%f, r=%f, pt=%f, phi=%f, theta=%f \n", inputtrk.par[0], inputtrk.par[1], inputtrk.par[2],
	  sqrtf(inputtrk.par[0]*inputtrk.par[0] + inputtrk.par[1]*inputtrk.par[1]),
	  1./inputtrk.par[3], inputtrk.par[4], inputtrk.par[5]);
   printf("track in cov: xx=%.2e, yy=%.2e, zz=%.2e \n", inputtrk.cov[SymOffsets66[0]],
	                                       inputtrk.cov[SymOffsets66[(1*6+1)]],
	                                       inputtrk.cov[SymOffsets66[(2*6+2)]]);
   for (int lay=0; lay<nlayer; lay++){
     printf("hit in layer=%lu, pos: x=%f, y=%f, z=%f, r=%f \n", lay, inputhits[lay].pos[0], inputhits[lay].pos[1], inputhits[lay].pos[2], sqrtf(inputhits[lay].pos[0]*inputhits[lay].pos[0] + inputhits[lay].pos[1]*inputhits[lay].pos[1]));
   }
   
   printf("produce nevts=%i ntrks=%i smearing by=%f \n", nevts, ntrks, smear);
   printf("NITER=%d\n", NITER);

   long setup_start, setup_stop;
   struct timeval timecheck;

   constexpr auto order = FieldOrder::P2Z_TRACKBLK_EVENT_LAYER_MATIDX_ORDER;

   using MPTRKAccessorTp = MPTRKAccessor<order>;
   using MPHITAccessorTp = MPHITAccessor<order>;

   impl::UVMAllocator<MPTRKAccessorTp> mptrk_uvm_alloc;
   impl::UVMAllocator<MPHITAccessorTp> mphit_uvm_alloc;

   gettimeofday(&timecheck, NULL);
   setup_start = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;

   std::unique_ptr<MPTRK> trcksPtr(new MPTRK(ntrks, nevts));
   auto trcksAccPtr = std::allocate_shared<MPTRKAccessorTp>(mptrk_uvm_alloc, *trcksPtr);
   //
   std::unique_ptr<MPHIT> hitsPtr(new MPHIT(ntrks, nevts, nlayer));
   auto hitsAccPtr = std::allocate_shared<MPHITAccessorTp>(mphit_uvm_alloc, *hitsPtr);
   //
   std::unique_ptr<MPTRK> outtrcksPtr(new MPTRK(ntrks, nevts));
   auto outtrcksAccPtr = std::allocate_shared<MPTRKAccessorTp>(mptrk_uvm_alloc, *outtrcksPtr);
   //
   std::vector<MPTRK_> trcks(nevts*ntrks); 
   prepareTracks(trcks, inputtrk);
   //
   std::vector<MPHIT_> hits(nlayer*nevts*ntrks);
   prepareHits(hits, inputhits);
   //
   std::vector<MPTRK_> outtrcks(nevts*ntrks);
   
   convertHits<order,   ConversionType::P2Z_CONVERT_TO_INTERNAL_ORDER>(hits,     hitsPtr.get());
   convertTracks<order, ConversionType::P2Z_CONVERT_TO_INTERNAL_ORDER>(trcks,    trcksPtr.get());
   convertTracks<order, ConversionType::P2Z_CONVERT_TO_INTERNAL_ORDER>(outtrcks, outtrcksPtr.get());

   gettimeofday(&timecheck, NULL);
   setup_stop = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;

   printf("done preparing!\n");

   printf("Size of struct MPTRK trk[] = %ld\n", nevts*ntrks*sizeof(MPTRK));
   printf("Size of struct MPTRK outtrk[] = %ld\n", nevts*ntrks*sizeof(MPTRK));
   printf("Size of struct struct MPHIT hit[] = %ld\n", nevts*ntrks*sizeof(MPHIT));

   const int phys_length      = nevts*ntrks;
   const int outer_loop_range = phys_length;
   //
   dim3 blocks(threadsperblock, 1, 1);
   dim3 grid(((outer_loop_range + threadsperblock - 1)/ threadsperblock),1,1);
   // A warmup run to migrate data on the device
   launch_p2z_kernels<nlayer><<<grid, blocks>>>(*outtrcksAccPtr, *trcksAccPtr, *hitsAccPtr, phys_length);

   hipDeviceSynchronize();

   p2z_check_error();

   auto wall_start = std::chrono::high_resolution_clock::now();

   for(int itr=0; itr<NITER; itr++) {

     launch_p2z_kernels<nlayer><<<grid, blocks>>>(*outtrcksAccPtr, *trcksAccPtr, *hitsAccPtr, phys_length);

   } //end of itr loop

   hipDeviceSynchronize();
   p2z_check_error();

   auto wall_stop = std::chrono::high_resolution_clock::now();

   auto wall_diff = wall_stop - wall_start;
   auto wall_time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(wall_diff).count()) / 1e6;   

   printf("setup time time=%f (s)\n", (setup_stop-setup_start)*0.001);
   printf("done ntracks=%i tot time=%f (s) time/trk=%e (s)\n", nevts*ntrks*int(NITER), wall_time, wall_time/(nevts*ntrks*int(NITER)));
   printf("formatted %i %i %i %i %i %f 0 %f %i\n",int(NITER),nevts, ntrks, 1, ntrks, wall_time, (setup_stop-setup_start)*0.001, -1);

   convertTracks<order, ConversionType::P2Z_CONVERT_FROM_INTERNAL_ORDER>(outtrcks, outtrcksPtr.get());
   auto outtrk = outtrcks.data();

   int nnans = 0, nfail = 0;
   float avgx = 0, avgy = 0, avgz = 0, avgr = 0;
   float avgpt = 0, avgphi = 0, avgtheta = 0;
   float avgdx = 0, avgdy = 0, avgdz = 0, avgdr = 0;

   for (int ie=0;ie<nevts;++ie) {
     for (int it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       float r_ = sqrtf(x_*x_ + y_*y_);
       float pt_ = std::abs(1./ipt(outtrk,ie, it));
       float phi_ = phi(outtrk,ie,it);
       float theta_ = theta(outtrk,ie,it);
       float hx_ = inputhits[nlayer-1].pos[0];
       float hy_ = inputhits[nlayer-1].pos[1];
       float hz_ = inputhits[nlayer-1].pos[2];
       float hr_ = sqrtf(hx_*hx_ + hy_*hy_);
 
       if (std::isfinite(x_)==false ||
          std::isfinite(y_)==false ||
          std::isfinite(z_)==false ||
          std::isfinite(pt_)==false ||
          std::isfinite(phi_)==false ||
          std::isfinite(theta_)==false
          ) {
        nnans++;
        continue;
       }
       if (fabs( (x_-hx_)/hx_ )>1. ||
	   fabs( (y_-hy_)/hy_ )>1. ||
	   fabs( (z_-hz_)/hz_ )>1.) {
	 nfail++;
	 continue;
       }

       avgpt += pt_;
       avgphi += phi_;
       avgtheta += theta_;
       avgx += x_;
       avgy += y_;
       avgz += z_;
       avgr += r_;
       avgdx += (x_-hx_)/x_;
       avgdy += (y_-hy_)/y_;
       avgdz += (z_-hz_)/z_;
       avgdr += (r_-hr_)/r_;
       //if((it+ie*ntrks) < 64) printf("iTrk = %i,  track (x,y,z,r)=(%.6f,%.6f,%.6f,%.6f) \n", it+ie*ntrks, x_,y_,z_,r_);
     }
   }

   avgpt = avgpt/float(nevts*ntrks);
   avgphi = avgphi/float(nevts*ntrks);
   avgtheta = avgtheta/float(nevts*ntrks);
   avgx = avgx/float(nevts*ntrks);
   avgy = avgy/float(nevts*ntrks);
   avgz = avgz/float(nevts*ntrks);
   avgr = avgr/float(nevts*ntrks);
   avgdx = avgdx/float(nevts*ntrks);
   avgdy = avgdy/float(nevts*ntrks);
   avgdz = avgdz/float(nevts*ntrks);
   avgdr = avgdr/float(nevts*ntrks);

   float stdx = 0, stdy = 0, stdz = 0, stdr = 0;
   float stddx = 0, stddy = 0, stddz = 0, stddr = 0;
   for (int ie=0;ie<nevts;++ie) {
     for (int it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       float r_ = sqrtf(x_*x_ + y_*y_);
       float hx_ = inputhits[nlayer-1].pos[0];
       float hy_ = inputhits[nlayer-1].pos[1];
       float hz_ = inputhits[nlayer-1].pos[2];
       float hr_ = sqrtf(hx_*hx_ + hy_*hy_);
       if (std::isfinite(x_)==false ||
          std::isfinite(y_)==false ||
          std::isfinite(z_)==false
          ) {
        continue;
       }
       if (fabs( (x_-hx_)/hx_ )>1. ||
	   fabs( (y_-hy_)/hy_ )>1. ||
	   fabs( (z_-hz_)/hz_ )>1.) {
	 continue;
       }
       stdx += (x_-avgx)*(x_-avgx);
       stdy += (y_-avgy)*(y_-avgy);
       stdz += (z_-avgz)*(z_-avgz);
       stdr += (r_-avgr)*(r_-avgr);
       stddx += ((x_-hx_)/x_-avgdx)*((x_-hx_)/x_-avgdx);
       stddy += ((y_-hy_)/y_-avgdy)*((y_-hy_)/y_-avgdy);
       stddz += ((z_-hz_)/z_-avgdz)*((z_-hz_)/z_-avgdz);
       stddr += ((r_-hr_)/r_-avgdr)*((r_-hr_)/r_-avgdr);
     }
   }

   stdx = sqrtf(stdx/float(nevts*ntrks));
   stdy = sqrtf(stdy/float(nevts*ntrks));
   stdz = sqrtf(stdz/float(nevts*ntrks));
   stdr = sqrtf(stdr/float(nevts*ntrks));
   stddx = sqrtf(stddx/float(nevts*ntrks));
   stddy = sqrtf(stddy/float(nevts*ntrks));
   stddz = sqrtf(stddz/float(nevts*ntrks));
   stddr = sqrtf(stddr/float(nevts*ntrks));

   printf("track x avg=%f std/avg=%f\n", avgx, fabs(stdx/avgx));
   printf("track y avg=%f std/avg=%f\n", avgy, fabs(stdy/avgy));
   printf("track z avg=%f std/avg=%f\n", avgz, fabs(stdz/avgz));
   printf("track r avg=%f std/avg=%f\n", avgr, fabs(stdr/avgz));
   printf("track dx/x avg=%f std=%f\n", avgdx, stddx);
   printf("track dy/y avg=%f std=%f\n", avgdy, stddy);
   printf("track dz/z avg=%f std=%f\n", avgdz, stddz);
   printf("track dr/r avg=%f std=%f\n", avgdr, stddr);
   printf("track pt avg=%f\n", avgpt);
   printf("track phi avg=%f\n", avgphi);
   printf("track theta avg=%f\n", avgtheta);
   printf("number of tracks with nans=%i\n", nnans);
   printf("number of tracks failed=%i\n", nfail);

   return 0;
}
