#include "hip/hip_runtime.h"
/*
icc propagate-toz-test.C -o propagate-toz-test.exe -fopenmp -O3
*/
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <iostream>
#include <chrono>
#include <iomanip>

//#define DUMP_OUTPUT
#define FIXED_RSEED
//#define USE_ASYNC
#ifndef USE_ASYNC
#define num_streams 1
#endif

#ifndef nevts
#define nevts 100
#endif
#ifndef bsize
#define bsize 32
#endif
#ifndef ntrks
#define ntrks 9600
#endif

#define nb    (ntrks/bsize)
#define smear 0.00001

#ifndef NITER
#define NITER 5
#endif
#ifndef nlayer
#define nlayer 20
#endif
#ifndef num_streams
#define num_streams 10
#endif

#ifndef threadsperblockx
#define threadsperblockx bsize
#endif
#define threadsperblocky 1
#ifndef blockspergrid
#define blockspergrid nevts*nb/num_streams
#endif

#define HOSTDEV __host__ __device__

HOSTDEV size_t PosInMtrx(size_t i, size_t j, size_t D) {
  return i*D+j;
}

HOSTDEV size_t SymOffsets33(size_t i) {
  const size_t offs[9] = {0, 1, 3, 1, 2, 4, 3, 4, 5};
  return offs[i];
}

HOSTDEV size_t SymOffsets66(size_t i) {
  const size_t offs[36] = {0, 1, 3, 6, 10, 15, 1, 2, 4, 7, 11, 16, 3, 4, 5, 8, 12, 17, 6, 7, 8, 9, 13, 18, 10, 11, 12, 13, 14, 19, 15, 16, 17, 18, 19, 20};
  return offs[i];
}


struct ATRK {
  float par[6];
  float cov[21];
  int q;
//  int hitidx[22];
};

struct AHIT {
  float pos[3];
  float cov[6];
};

struct MP1I {
  int data[1*bsize];
};

struct MP22I {
  int data[22*bsize];
};

struct MP3F {
  float data[3*bsize];
};

struct MP6F {
  float data[6*bsize];
};

struct MP3x3 {
  float data[9*bsize];
};

struct MP3x6 {
  float data[18*bsize];
};

struct MP3x3SF {
  float data[6*bsize];
};

struct MP6x6SF {
  float data[21*bsize];
};

struct MP6x6F {
  float data[36*bsize];
};

struct MP2x2SF {
  float data[3*bsize];
};

struct MP2x6 {
  float data[12*bsize];
};

struct MP2F {
  float data[2*bsize];
};

struct MPTRK {
  MP6F    par;
  MP6x6SF cov;
  MP1I    q;
//  MP22I   hitidx;
};

struct MPHIT {
  MP3F    pos;
  MP3x3SF cov;
};

float randn(float mu, float sigma) {
  float U1, U2, W, mult;
  static float X1, X2;
  static int call = 0;
  if (call == 1) {
    call = !call;
    return (mu + sigma * (float) X2);
  } do {
    U1 = -1 + ((float) rand () / RAND_MAX) * 2;
    U2 = -1 + ((float) rand () / RAND_MAX) * 2;
    W = pow (U1, 2) + pow (U2, 2);
  }
  while (W >= 1 || W == 0);
  mult = sqrt ((-2 * log (W)) / W);
  X1 = U1 * mult;
  X2 = U2 * mult;
  call = !call;
  return (mu + sigma * (float) X1);
}

MPTRK* prepareTracks(ATRK inputtrk) {
  MPTRK* result;
  hipMallocManaged((void**)&result,nevts*nb*sizeof(MPTRK));
  for (size_t ie=0;ie<nevts;++ie) {
    for (size_t ib=0;ib<nb;++ib) {
      for (size_t it=0;it<bsize;++it) {
        //par
        for (size_t ip=0;ip<6;++ip) {
          result[ib + nb*ie].par.data[it + ip*bsize] = (1+smear*randn(0,1))*inputtrk.par[ip];
        }
        //cov, scale by factor 100
        for (size_t ip=0;ip<21;++ip) {
          result[ib + nb*ie].cov.data[it + ip*bsize] = (1+smear*randn(0,1))*inputtrk.cov[ip]*100;
        }
        //q
        result[ib + nb*ie].q.data[it] = inputtrk.q;//can't really smear this or fit will be wrong
      }
    }
  }
  return result;
}

MPHIT* prepareHits(AHIT* inputhits) {
  MPHIT* result;
  hipMallocManaged((void**)&result,nlayer*nevts*nb*sizeof(MPHIT));
  for (size_t lay=0;lay<nlayer;++lay) {

    struct AHIT inputhit = inputhits[lay]; 

    for (size_t ie=0;ie<nevts;++ie) {
      for (size_t ib=0;ib<nb;++ib) {
        for (size_t it=0;it<bsize;++it) {
          //pos
          for (size_t ip=0;ip<3;++ip) {
            result[lay+nlayer*(ib + nb*ie)].pos.data[it + ip*bsize] = (1+smear*randn(0,1))*inputhit.pos[ip];
          }
          //cov
          for (size_t ip=0;ip<6;++ip) {
            result[lay+nlayer*(ib + nb*ie)].cov.data[it + ip*bsize] = (1+smear*randn(0,1))*inputhit.cov[ip];
          }
        }
      }
    }
  }
  return result;
}


HOSTDEV MPTRK* bTk(MPTRK* tracks, size_t ev, size_t ib) {
  return &(tracks[ib + nb*ev]);
}

HOSTDEV const MPTRK* bTk(const MPTRK* tracks, size_t ev, size_t ib) {
  return &(tracks[ib + nb*ev]);
}


HOSTDEV int q(const MP1I* bq, size_t it){
  return (*bq).data[it];
}

HOSTDEV float par(const MP6F* bpars, size_t it, size_t ipar){
  return (*bpars).data[it + ipar*bsize];
}
HOSTDEV float x    (const MP6F* bpars, size_t it){ return par(bpars, it, 0); }
HOSTDEV float y    (const MP6F* bpars, size_t it){ return par(bpars, it, 1); }
HOSTDEV float z    (const MP6F* bpars, size_t it){ return par(bpars, it, 2); }
HOSTDEV float ipt  (const MP6F* bpars, size_t it){ return par(bpars, it, 3); }
HOSTDEV float phi  (const MP6F* bpars, size_t it){ return par(bpars, it, 4); }
HOSTDEV float theta(const MP6F* bpars, size_t it){ return par(bpars, it, 5); }

HOSTDEV float par(const MPTRK* btracks, size_t it, size_t ipar){
  return par(&(*btracks).par,it,ipar);
}
HOSTDEV float x    (const MPTRK* btracks, size_t it){ return par(btracks, it, 0); }
HOSTDEV float y    (const MPTRK* btracks, size_t it){ return par(btracks, it, 1); }
HOSTDEV float z    (const MPTRK* btracks, size_t it){ return par(btracks, it, 2); }
HOSTDEV float ipt  (const MPTRK* btracks, size_t it){ return par(btracks, it, 3); }
HOSTDEV float phi  (const MPTRK* btracks, size_t it){ return par(btracks, it, 4); }
HOSTDEV float theta(const MPTRK* btracks, size_t it){ return par(btracks, it, 5); }

HOSTDEV float par(const MPTRK* tracks, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  const MPTRK* btracks = bTk(tracks, ev, ib);
  size_t it = tk % bsize;
  return par(btracks, it, ipar);
}

HOSTDEV float x    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 0); }
HOSTDEV float y    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 1); }
HOSTDEV float z    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 2); }
HOSTDEV float ipt  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 3); }
HOSTDEV float phi  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 4); }
HOSTDEV float theta(const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 5); }

HOSTDEV void setpar(MP6F* bpars, size_t it, size_t ipar, float val){
  (*bpars).data[it + ipar*bsize] = val;
}
HOSTDEV void setx    (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 0, val); }
HOSTDEV void sety    (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 1, val); }
HOSTDEV void setz    (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 2, val); }
HOSTDEV void setipt  (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 3, val); }
HOSTDEV void setphi  (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 4, val); }
HOSTDEV void settheta(MP6F* bpars, size_t it, float val){ setpar(bpars, it, 5, val); }

HOSTDEV void setpar(MPTRK* btracks, size_t it, size_t ipar, float val){
  setpar(&(*btracks).par,it,ipar,val);
}
HOSTDEV void setx    (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 0, val); }
HOSTDEV void sety    (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 1, val); }
HOSTDEV void setz    (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 2, val); }
HOSTDEV void setipt  (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 3, val); }
HOSTDEV void setphi  (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 4, val); }
HOSTDEV void settheta(MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 5, val); }

HOSTDEV MPHIT* bHit(MPHIT* hits, size_t ev, size_t ib) {
  return &(hits[ib + nb*ev]);
}
HOSTDEV const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib) {
  return &(hits[ib + nb*ev]);
}
HOSTDEV const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib,int lay) {
  return &(hits[lay + (ib*nlayer) +(ev*nlayer*nb)]);
}

HOSTDEV float pos(const MP3F* hpos, size_t it, size_t ipar){
  return (*hpos).data[it + ipar*bsize];
}
HOSTDEV float x(const MP3F* hpos, size_t it)    { return pos(hpos, it, 0); }
HOSTDEV float y(const MP3F* hpos, size_t it)    { return pos(hpos, it, 1); }
HOSTDEV float z(const MP3F* hpos, size_t it)    { return pos(hpos, it, 2); }

HOSTDEV float pos(const MPHIT* hits, size_t it, size_t ipar){
  return pos(&(*hits).pos,it,ipar);
}
HOSTDEV float x(const MPHIT* hits, size_t it)    { return pos(hits, it, 0); }
HOSTDEV float y(const MPHIT* hits, size_t it)    { return pos(hits, it, 1); }
HOSTDEV float z(const MPHIT* hits, size_t it)    { return pos(hits, it, 2); }

HOSTDEV float pos(const MPHIT* hits, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  //[DEBUG by Seyong on Dec. 28, 2020] add 4th argument(nlayer-1) to bHit() below.
  const MPHIT* bhits = bHit(hits, ev, ib, nlayer-1);
  size_t it = tk % bsize;
  return pos(bhits,it,ipar);
}
HOSTDEV float x(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 0); }
HOSTDEV float y(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 1); }
HOSTDEV float z(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 2); }



#define N bsize
__forceinline__ __device__ void MultHelixPropEndcap(const MP6x6F* A, const MP6x6SF* B, MP6x6F* C) {
  const float* a; //ASSUME_ALIGNED(a, 64);
  const float* b; //ASSUME_ALIGNED(b, 64);
  float* c;       //ASSUME_ALIGNED(c, 64);
  a = A->data; //ASSUME_ALIGNED(a, 64);
  b = B->data; //ASSUME_ALIGNED(b, 64);
  c = C->data;       //ASSUME_ALIGNED(c, 64);
  for(int n=threadIdx.x;n<N;n+=blockDim.x)
  {
    c[ 0*N+n] = b[ 0*N+n] + a[ 2*N+n]*b[ 3*N+n] + a[ 3*N+n]*b[ 6*N+n] + a[ 4*N+n]*b[10*N+n] + a[ 5*N+n]*b[15*N+n];
    c[ 1*N+n] = b[ 1*N+n] + a[ 2*N+n]*b[ 4*N+n] + a[ 3*N+n]*b[ 7*N+n] + a[ 4*N+n]*b[11*N+n] + a[ 5*N+n]*b[16*N+n];
    c[ 2*N+n] = b[ 3*N+n] + a[ 2*N+n]*b[ 5*N+n] + a[ 3*N+n]*b[ 8*N+n] + a[ 4*N+n]*b[12*N+n] + a[ 5*N+n]*b[17*N+n];
    c[ 3*N+n] = b[ 6*N+n] + a[ 2*N+n]*b[ 8*N+n] + a[ 3*N+n]*b[ 9*N+n] + a[ 4*N+n]*b[13*N+n] + a[ 5*N+n]*b[18*N+n];
    c[ 4*N+n] = b[10*N+n] + a[ 2*N+n]*b[12*N+n] + a[ 3*N+n]*b[13*N+n] + a[ 4*N+n]*b[14*N+n] + a[ 5*N+n]*b[19*N+n];
    c[ 5*N+n] = b[15*N+n] + a[ 2*N+n]*b[17*N+n] + a[ 3*N+n]*b[18*N+n] + a[ 4*N+n]*b[19*N+n] + a[ 5*N+n]*b[20*N+n];
    c[ 6*N+n] = b[ 1*N+n] + a[ 8*N+n]*b[ 3*N+n] + a[ 9*N+n]*b[ 6*N+n] + a[10*N+n]*b[10*N+n] + a[11*N+n]*b[15*N+n];
    c[ 7*N+n] = b[ 2*N+n] + a[ 8*N+n]*b[ 4*N+n] + a[ 9*N+n]*b[ 7*N+n] + a[10*N+n]*b[11*N+n] + a[11*N+n]*b[16*N+n];
    c[ 8*N+n] = b[ 4*N+n] + a[ 8*N+n]*b[ 5*N+n] + a[ 9*N+n]*b[ 8*N+n] + a[10*N+n]*b[12*N+n] + a[11*N+n]*b[17*N+n];
    c[ 9*N+n] = b[ 7*N+n] + a[ 8*N+n]*b[ 8*N+n] + a[ 9*N+n]*b[ 9*N+n] + a[10*N+n]*b[13*N+n] + a[11*N+n]*b[18*N+n];
    c[10*N+n] = b[11*N+n] + a[ 8*N+n]*b[12*N+n] + a[ 9*N+n]*b[13*N+n] + a[10*N+n]*b[14*N+n] + a[11*N+n]*b[19*N+n];
    c[11*N+n] = b[16*N+n] + a[ 8*N+n]*b[17*N+n] + a[ 9*N+n]*b[18*N+n] + a[10*N+n]*b[19*N+n] + a[11*N+n]*b[20*N+n];
    c[12*N+n] = 0;
    c[13*N+n] = 0;
    c[14*N+n] = 0;
    c[15*N+n] = 0;
    c[16*N+n] = 0;
    c[17*N+n] = 0;
    c[18*N+n] = b[ 6*N+n];
    c[19*N+n] = b[ 7*N+n];
    c[20*N+n] = b[ 8*N+n];
    c[21*N+n] = b[ 9*N+n];
    c[22*N+n] = b[13*N+n];
    c[23*N+n] = b[18*N+n];
    c[24*N+n] = a[26*N+n]*b[ 3*N+n] + a[27*N+n]*b[ 6*N+n] + b[10*N+n] + a[29*N+n]*b[15*N+n];
    c[25*N+n] = a[26*N+n]*b[ 4*N+n] + a[27*N+n]*b[ 7*N+n] + b[11*N+n] + a[29*N+n]*b[16*N+n];
    c[26*N+n] = a[26*N+n]*b[ 5*N+n] + a[27*N+n]*b[ 8*N+n] + b[12*N+n] + a[29*N+n]*b[17*N+n];
    c[27*N+n] = a[26*N+n]*b[ 8*N+n] + a[27*N+n]*b[ 9*N+n] + b[13*N+n] + a[29*N+n]*b[18*N+n];
    c[28*N+n] = a[26*N+n]*b[12*N+n] + a[27*N+n]*b[13*N+n] + b[14*N+n] + a[29*N+n]*b[19*N+n];
    c[29*N+n] = a[26*N+n]*b[17*N+n] + a[27*N+n]*b[18*N+n] + b[19*N+n] + a[29*N+n]*b[20*N+n];
    c[30*N+n] = b[15*N+n];
    c[31*N+n] = b[16*N+n];
    c[32*N+n] = b[17*N+n];
    c[33*N+n] = b[18*N+n];
    c[34*N+n] = b[19*N+n];
    c[35*N+n] = b[20*N+n];
  }
}

__forceinline__ __device__ void MultHelixPropTranspEndcap(MP6x6F* A, MP6x6F* B, MP6x6SF* C) {
  const float* a; //ASSUME_ALIGNED(a, 64);
  const float* b; //ASSUME_ALIGNED(b, 64);
  float* c;       //ASSUME_ALIGNED(c, 64);
  a = A->data; //ASSUME_ALIGNED(a, 64);
  b = B->data; //ASSUME_ALIGNED(b, 64);
  c = C->data;       //ASSUME_ALIGNED(c, 64);
  for(int n=threadIdx.x;n<N;n+=blockDim.x)
  {
    c[ 0*N+n] = b[ 0*N+n] + b[ 2*N+n]*a[ 2*N+n] + b[ 3*N+n]*a[ 3*N+n] + b[ 4*N+n]*a[ 4*N+n] + b[ 5*N+n]*a[ 5*N+n];
    c[ 1*N+n] = b[ 6*N+n] + b[ 8*N+n]*a[ 2*N+n] + b[ 9*N+n]*a[ 3*N+n] + b[10*N+n]*a[ 4*N+n] + b[11*N+n]*a[ 5*N+n];
    c[ 2*N+n] = b[ 7*N+n] + b[ 8*N+n]*a[ 8*N+n] + b[ 9*N+n]*a[ 9*N+n] + b[10*N+n]*a[10*N+n] + b[11*N+n]*a[11*N+n];
    c[ 3*N+n] = b[12*N+n] + b[14*N+n]*a[ 2*N+n] + b[15*N+n]*a[ 3*N+n] + b[16*N+n]*a[ 4*N+n] + b[17*N+n]*a[ 5*N+n];
    c[ 4*N+n] = b[13*N+n] + b[14*N+n]*a[ 8*N+n] + b[15*N+n]*a[ 9*N+n] + b[16*N+n]*a[10*N+n] + b[17*N+n]*a[11*N+n];
    c[ 5*N+n] = 0;
    c[ 6*N+n] = b[18*N+n] + b[20*N+n]*a[ 2*N+n] + b[21*N+n]*a[ 3*N+n] + b[22*N+n]*a[ 4*N+n] + b[23*N+n]*a[ 5*N+n];
    c[ 7*N+n] = b[19*N+n] + b[20*N+n]*a[ 8*N+n] + b[21*N+n]*a[ 9*N+n] + b[22*N+n]*a[10*N+n] + b[23*N+n]*a[11*N+n];
    c[ 8*N+n] = 0;
    c[ 9*N+n] = b[21*N+n];
    c[10*N+n] = b[24*N+n] + b[26*N+n]*a[ 2*N+n] + b[27*N+n]*a[ 3*N+n] + b[28*N+n]*a[ 4*N+n] + b[29*N+n]*a[ 5*N+n];
    c[11*N+n] = b[25*N+n] + b[26*N+n]*a[ 8*N+n] + b[27*N+n]*a[ 9*N+n] + b[28*N+n]*a[10*N+n] + b[29*N+n]*a[11*N+n];
    c[12*N+n] = 0;
    c[13*N+n] = b[27*N+n];
    c[14*N+n] = b[26*N+n]*a[26*N+n] + b[27*N+n]*a[27*N+n] + b[28*N+n] + b[29*N+n]*a[29*N+n];
    c[15*N+n] = b[30*N+n] + b[32*N+n]*a[ 2*N+n] + b[33*N+n]*a[ 3*N+n] + b[34*N+n]*a[ 4*N+n] + b[35*N+n]*a[ 5*N+n];
    c[16*N+n] = b[31*N+n] + b[32*N+n]*a[ 8*N+n] + b[33*N+n]*a[ 9*N+n] + b[34*N+n]*a[10*N+n] + b[35*N+n]*a[11*N+n];
    c[17*N+n] = 0;
    c[18*N+n] = b[33*N+n];
    c[19*N+n] = b[32*N+n]*a[26*N+n] + b[33*N+n]*a[27*N+n] + b[34*N+n] + b[35*N+n]*a[29*N+n];
    c[20*N+n] = b[35*N+n];
  }
}

__forceinline__ __device__ void KalmanGainInv(const MP6x6SF* A, const MP3x3SF* B, MP3x3* C) {
  // k = P Ht(HPHt + R)^-1
  // HpHt -> cov of x,y,z. take upper 3x3 matrix of P
  // This calculates the inverse of HpHt +R
  const float* a; //ASSUME_ALIGNED(a, 64);
  const float* b; //ASSUME_ALIGNED(b, 64);
  float* c;       //ASSUME_ALIGNED(c, 64);
  a = (*A).data; //ASSUME_ALIGNED(a, 64);
  b = (*B).data; //ASSUME_ALIGNED(b, 64);
  c = (*C).data;       //ASSUME_ALIGNED(c, 64);
  for(int n=threadIdx.x;n<N;n+=blockDim.x)
  {
    double det =
      ((a[0*N+n]+b[0*N+n])*(((a[ 6*N+n]+b[ 3*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[7*N+n]+b[4*N+n])))) -
      ((a[1*N+n]+b[1*N+n])*(((a[ 1*N+n]+b[ 1*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[2*N+n]+b[2*N+n])))) +
      ((a[2*N+n]+b[2*N+n])*(((a[ 1*N+n]+b[ 1*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[6*N+n]+b[3*N+n]))));
    double invdet = 1.0/det;

    c[ 0*N+n] =  invdet*(((a[ 6*N+n]+b[ 3*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[7*N+n]+b[4*N+n])));
    c[ 1*N+n] =  -invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[7*N+n]+b[4*N+n])));
    c[ 2*N+n] =  invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[7*N+n]+b[4*N+n])));
    c[ 3*N+n] =  -invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[2*N+n]+b[2*N+n])));
    c[ 4*N+n] =  invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[2*N+n]+b[2*N+n])));
    c[ 5*N+n] =  -invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[1*N+n]+b[1*N+n])));
    c[ 6*N+n] =  invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[6*N+n]+b[3*N+n])));
    c[ 7*N+n] =  -invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[1*N+n]+b[1*N+n])));
    c[ 8*N+n] =  invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[6*N+n]+b[3*N+n])) - ((a[1*N+n]+b[1*N+n]) *(a[1*N+n]+b[1*N+n])));
  }
}

__forceinline__ __device__ void KalmanGain(const MP6x6SF* A, const MP3x3* B, MP3x6* C) {
  // k = P Ht(HPHt + R)^-1
  // HpHt -> cov of x,y,z. take upper 3x3 matrix of P
  // This calculates the kalman gain 
  const float* a; //ASSUME_ALIGNED(a, 64);
  const float* b; //ASSUME_ALIGNED(b, 64);
  float* c;       //ASSUME_ALIGNED(c, 64);
  a = (*A).data; //ASSUME_ALIGNED(a, 64);
  b = (*B).data; //ASSUME_ALIGNED(b, 64);
  c = (*C).data;       //ASSUME_ALIGNED(c, 64);
  for(int n=threadIdx.x;n<N;n+=blockDim.x)
  {
    c[ 0*N+n] = a[0*N+n]*b[0*N+n] + a[1*N+n]*b[3*N+n] + a[2*N+n]*b[6*N+n];
    c[ 1*N+n] = a[0*N+n]*b[1*N+n] + a[1*N+n]*b[4*N+n] + a[2*N+n]*b[7*N+n];
    c[ 2*N+n] = a[0*N+n]*b[2*N+n] + a[1*N+n]*b[5*N+n] + a[2*N+n]*b[8*N+n];
    c[ 3*N+n] = a[1*N+n]*b[0*N+n] + a[6*N+n]*b[3*N+n] + a[7*N+n]*b[6*N+n];
    c[ 4*N+n] = a[1*N+n]*b[1*N+n] + a[6*N+n]*b[4*N+n] + a[7*N+n]*b[7*N+n];
    c[ 5*N+n] = a[1*N+n]*b[2*N+n] + a[6*N+n]*b[5*N+n] + a[7*N+n]*b[8*N+n];
    c[ 6*N+n] = a[2*N+n]*b[0*N+n] + a[7*N+n]*b[3*N+n] + a[11*N+n]*b[6*N+n];
    c[ 7*N+n] = a[2*N+n]*b[1*N+n] + a[7*N+n]*b[4*N+n] + a[11*N+n]*b[7*N+n];
    c[ 8*N+n] = a[2*N+n]*b[2*N+n] + a[7*N+n]*b[5*N+n] + a[11*N+n]*b[8*N+n];
    c[ 9*N+n] = a[3*N+n]*b[0*N+n] + a[8*N+n]*b[3*N+n] + a[12*N+n]*b[6*N+n];
    c[ 10*N+n] = a[3*N+n]*b[1*N+n] + a[8*N+n]*b[4*N+n] + a[12*N+n]*b[7*N+n];
    c[ 11*N+n] = a[3*N+n]*b[2*N+n] + a[8*N+n]*b[5*N+n] + a[12*N+n]*b[8*N+n];
    c[ 12*N+n] = a[4*N+n]*b[0*N+n] + a[9*N+n]*b[3*N+n] + a[13*N+n]*b[6*N+n];
    c[ 13*N+n] = a[4*N+n]*b[1*N+n] + a[9*N+n]*b[4*N+n] + a[13*N+n]*b[7*N+n];
    c[ 14*N+n] = a[4*N+n]*b[2*N+n] + a[9*N+n]*b[5*N+n] + a[13*N+n]*b[8*N+n];
    c[ 15*N+n] = a[5*N+n]*b[0*N+n] + a[10*N+n]*b[3*N+n] + a[14*N+n]*b[6*N+n];
    c[ 16*N+n] = a[5*N+n]*b[1*N+n] + a[10*N+n]*b[4*N+n] + a[14*N+n]*b[7*N+n];
    c[ 17*N+n] = a[5*N+n]*b[2*N+n] + a[10*N+n]*b[5*N+n] + a[14*N+n]*b[8*N+n];
  }
}

__forceinline__ __device__ void KalmanUpdate(MP6x6SF* trkErr, MP6F* inPar, const MP3x3SF* hitErr, const MP3F* msP){
  __shared__ MP3x3 inverse_temp;
  __shared__ MP3x6 kGain;
  __shared__ MP6x6SF newErr;
  KalmanGainInv(trkErr,hitErr,&inverse_temp);
  KalmanGain(trkErr,&inverse_temp,&kGain);

  for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x){
    const float xin = x(inPar,it);
    const float yin = y(inPar,it);
    const float zin = z(inPar,it);
    const float ptin = 1.0f/ipt(inPar,it); // is this pt or ipt? 
    const float phiin = phi(inPar,it);
    const float thetain = theta(inPar,it);
    const float xout = x(msP,it);
    const float yout = y(msP,it);
    //const float zout = z(msP,it);
  
    float xnew = xin + (kGain.data[0*bsize+it]*(xout-xin)) +(kGain.data[1*bsize+it]*(yout-yin));
    float ynew = yin + (kGain.data[3*bsize+it]*(xout-xin)) +(kGain.data[4*bsize+it]*(yout-yin));
    float znew = zin + (kGain.data[6*bsize+it]*(xout-xin)) +(kGain.data[7*bsize+it]*(yout-yin));
    float ptnew = ptin + (kGain.data[9*bsize+it]*(xout-xin)) +(kGain.data[10*bsize+it]*(yout-yin));
    float phinew = phiin + (kGain.data[12*bsize+it]*(xout-xin)) +(kGain.data[13*bsize+it]*(yout-yin));
    float thetanew = thetain + (kGain.data[15*bsize+it]*(xout-xin)) +(kGain.data[16*bsize+it]*(yout-yin));
  
    newErr.data[0*bsize+it] = trkErr->data[0*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[0*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[1*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[2*bsize+it]);
    newErr.data[1*bsize+it] = trkErr->data[1*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[1*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[6*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[7*bsize+it]);
    newErr.data[2*bsize+it] = trkErr->data[2*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[2*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[7*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[11*bsize+it]);
    newErr.data[3*bsize+it] = trkErr->data[3*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[4*bsize+it] = trkErr->data[4*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[5*bsize+it] = trkErr->data[5*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[14*bsize+it]);
  
    newErr.data[6*bsize+it] = trkErr->data[6*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[1*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[6*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[7*bsize+it]);
    newErr.data[7*bsize+it] = trkErr->data[7*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[2*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[7*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[11*bsize+it]);
    newErr.data[8*bsize+it] = trkErr->data[8*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[9*bsize+it] = trkErr->data[9*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[10*bsize+it] = trkErr->data[10*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[14*bsize+it]);
  
    newErr.data[11*bsize+it] = trkErr->data[11*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[2*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[7*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[11*bsize+it]);
    newErr.data[12*bsize+it] = trkErr->data[12*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[13*bsize+it] = trkErr->data[13*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[14*bsize+it] = trkErr->data[14*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[14*bsize+it]);
  
    newErr.data[15*bsize+it] = trkErr->data[15*bsize+it] - (kGain.data[9*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[10*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[11*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[16*bsize+it] = trkErr->data[16*bsize+it] - (kGain.data[9*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[10*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[11*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[17*bsize+it] = trkErr->data[17*bsize+it] - (kGain.data[9*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[10*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[11*bsize+it]*trkErr->data[14*bsize+it]);
  
    newErr.data[18*bsize+it] = trkErr->data[18*bsize+it] - (kGain.data[12*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[13*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[14*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[19*bsize+it] = trkErr->data[19*bsize+it] - (kGain.data[12*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[13*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[14*bsize+it]*trkErr->data[14*bsize+it]);
  
    newErr.data[20*bsize+it] = trkErr->data[20*bsize+it] - (kGain.data[15*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[16*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[17*bsize+it]*trkErr->data[14*bsize+it]);
  
    setx(inPar,it,xnew );
    sety(inPar,it,ynew );
    setz(inPar,it,znew);
    setipt(inPar,it, ptnew);
    setphi(inPar,it, phinew);
    settheta(inPar,it, thetanew);
  }
  for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x){
    #pragma unroll
    for (int i = 0; i < 21; i++){
      trkErr->data[ i*bsize+it] = trkErr->data[ i*bsize+it] - newErr.data[ i*bsize+it];
    }
  }
}

__forceinline__ __device__ void KalmanUpdate_v2(struct MP6x6SF* trkErr, struct MP6F* inPar, const struct MP3x3SF* hitErr, const struct MP3F* msP, struct MP2x2SF* resErr_loc, struct MP2x6* kGain, struct MP2F* res_loc, struct MP6x6SF* newErr){

   // AddIntoUpperLeft2x2(psErr, msErr, resErr);
   for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x)
   {
     resErr_loc->data[0*bsize+it] = trkErr->data[0*bsize+it] + hitErr->data[0*bsize+it];
     resErr_loc->data[1*bsize+it] = trkErr->data[1*bsize+it] + hitErr->data[1*bsize+it];
     resErr_loc->data[2*bsize+it] = trkErr->data[2*bsize+it] + hitErr->data[2*bsize+it];
   }

   // Matriplex::InvertCramerSym(resErr);
   for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x)
   {
     const double det = (double)resErr_loc->data[0*bsize+it] * resErr_loc->data[2*bsize+it] -
                        (double)resErr_loc->data[1*bsize+it] * resErr_loc->data[1*bsize+it];
     const float s   = 1.f / det;
     const float tmp = s * resErr_loc->data[2*bsize+it];
     resErr_loc->data[1*bsize+it] *= -s;
     resErr_loc->data[2*bsize+it]  = s * resErr_loc->data[0*bsize+it];
     resErr_loc->data[0*bsize+it]  = tmp;
   }

   // KalmanGain(psErr, resErr, K);
   for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x)
   {
      kGain->data[ 0*bsize+it] = trkErr->data[ 0*bsize+it]*resErr_loc->data[ 0*bsize+it] + trkErr->data[ 1*bsize+it]*resErr_loc->data[ 1*bsize+it];
      kGain->data[ 1*bsize+it] = trkErr->data[ 0*bsize+it]*resErr_loc->data[ 1*bsize+it] + trkErr->data[ 1*bsize+it]*resErr_loc->data[ 2*bsize+it];
      kGain->data[ 2*bsize+it] = trkErr->data[ 1*bsize+it]*resErr_loc->data[ 0*bsize+it] + trkErr->data[ 2*bsize+it]*resErr_loc->data[ 1*bsize+it];
      kGain->data[ 3*bsize+it] = trkErr->data[ 1*bsize+it]*resErr_loc->data[ 1*bsize+it] + trkErr->data[ 2*bsize+it]*resErr_loc->data[ 2*bsize+it];
      kGain->data[ 4*bsize+it] = trkErr->data[ 3*bsize+it]*resErr_loc->data[ 0*bsize+it] + trkErr->data[ 4*bsize+it]*resErr_loc->data[ 1*bsize+it];
      kGain->data[ 5*bsize+it] = trkErr->data[ 3*bsize+it]*resErr_loc->data[ 1*bsize+it] + trkErr->data[ 4*bsize+it]*resErr_loc->data[ 2*bsize+it];
      kGain->data[ 6*bsize+it] = trkErr->data[ 6*bsize+it]*resErr_loc->data[ 0*bsize+it] + trkErr->data[ 7*bsize+it]*resErr_loc->data[ 1*bsize+it];
      kGain->data[ 7*bsize+it] = trkErr->data[ 6*bsize+it]*resErr_loc->data[ 1*bsize+it] + trkErr->data[ 7*bsize+it]*resErr_loc->data[ 2*bsize+it];
      kGain->data[ 8*bsize+it] = trkErr->data[10*bsize+it]*resErr_loc->data[ 0*bsize+it] + trkErr->data[11*bsize+it]*resErr_loc->data[ 1*bsize+it];
      kGain->data[ 9*bsize+it] = trkErr->data[10*bsize+it]*resErr_loc->data[ 1*bsize+it] + trkErr->data[11*bsize+it]*resErr_loc->data[ 2*bsize+it];
      kGain->data[10*bsize+it] = trkErr->data[15*bsize+it]*resErr_loc->data[ 0*bsize+it] + trkErr->data[16*bsize+it]*resErr_loc->data[ 1*bsize+it];
      kGain->data[11*bsize+it] = trkErr->data[15*bsize+it]*resErr_loc->data[ 1*bsize+it] + trkErr->data[16*bsize+it]*resErr_loc->data[ 2*bsize+it];
   }

   // SubtractFirst2(msPar, psPar, res);
   // MultResidualsAdd(K, psPar, res, outPar);
   for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x)
   {
     res_loc->data[0*bsize+it] =  x(msP,it) - x(inPar,it);
     res_loc->data[1*bsize+it] =  y(msP,it) - y(inPar,it);

     setx    (inPar, it, x    (inPar, it) + kGain->data[ 0*bsize+it] * res_loc->data[ 0*bsize+it] + kGain->data[ 1*bsize+it] * res_loc->data[ 1*bsize+it]);
     sety    (inPar, it, y    (inPar, it) + kGain->data[ 2*bsize+it] * res_loc->data[ 0*bsize+it] + kGain->data[ 3*bsize+it] * res_loc->data[ 1*bsize+it]);
     setz    (inPar, it, z    (inPar, it) + kGain->data[ 4*bsize+it] * res_loc->data[ 0*bsize+it] + kGain->data[ 5*bsize+it] * res_loc->data[ 1*bsize+it]);
     setipt  (inPar, it, ipt  (inPar, it) + kGain->data[ 6*bsize+it] * res_loc->data[ 0*bsize+it] + kGain->data[ 7*bsize+it] * res_loc->data[ 1*bsize+it]);
     setphi  (inPar, it, phi  (inPar, it) + kGain->data[ 8*bsize+it] * res_loc->data[ 0*bsize+it] + kGain->data[ 9*bsize+it] * res_loc->data[ 1*bsize+it]);
     settheta(inPar, it, theta(inPar, it) + kGain->data[10*bsize+it] * res_loc->data[ 0*bsize+it] + kGain->data[11*bsize+it] * res_loc->data[ 1*bsize+it]);
     //note: if ipt changes sign we should update the charge, or we should get rid of the charge altogether and just use the sign of ipt
   }

   // squashPhiMPlex(outPar,N_proc); // ensure phi is between |pi|
   // missing

   // KHC(K, psErr, outErr);
   // outErr.Subtract(psErr, outErr);
   for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x)
   {
      newErr->data[ 0*bsize+it] = kGain->data[ 0*bsize+it]*trkErr->data[ 0*bsize+it] + kGain->data[ 1*bsize+it]*trkErr->data[ 1*bsize+it];
      newErr->data[ 1*bsize+it] = kGain->data[ 2*bsize+it]*trkErr->data[ 0*bsize+it] + kGain->data[ 3*bsize+it]*trkErr->data[ 1*bsize+it];
      newErr->data[ 2*bsize+it] = kGain->data[ 2*bsize+it]*trkErr->data[ 1*bsize+it] + kGain->data[ 3*bsize+it]*trkErr->data[ 2*bsize+it];
      newErr->data[ 3*bsize+it] = kGain->data[ 4*bsize+it]*trkErr->data[ 0*bsize+it] + kGain->data[ 5*bsize+it]*trkErr->data[ 1*bsize+it];
      newErr->data[ 4*bsize+it] = kGain->data[ 4*bsize+it]*trkErr->data[ 1*bsize+it] + kGain->data[ 5*bsize+it]*trkErr->data[ 2*bsize+it];
      newErr->data[ 5*bsize+it] = kGain->data[ 4*bsize+it]*trkErr->data[ 3*bsize+it] + kGain->data[ 5*bsize+it]*trkErr->data[ 4*bsize+it];
      newErr->data[ 6*bsize+it] = kGain->data[ 6*bsize+it]*trkErr->data[ 0*bsize+it] + kGain->data[ 7*bsize+it]*trkErr->data[ 1*bsize+it];
      newErr->data[ 7*bsize+it] = kGain->data[ 6*bsize+it]*trkErr->data[ 1*bsize+it] + kGain->data[ 7*bsize+it]*trkErr->data[ 2*bsize+it];
      newErr->data[ 8*bsize+it] = kGain->data[ 6*bsize+it]*trkErr->data[ 3*bsize+it] + kGain->data[ 7*bsize+it]*trkErr->data[ 4*bsize+it];
      newErr->data[ 9*bsize+it] = kGain->data[ 6*bsize+it]*trkErr->data[ 6*bsize+it] + kGain->data[ 7*bsize+it]*trkErr->data[ 7*bsize+it];
      newErr->data[10*bsize+it] = kGain->data[ 8*bsize+it]*trkErr->data[ 0*bsize+it] + kGain->data[ 9*bsize+it]*trkErr->data[ 1*bsize+it];
      newErr->data[11*bsize+it] = kGain->data[ 8*bsize+it]*trkErr->data[ 1*bsize+it] + kGain->data[ 9*bsize+it]*trkErr->data[ 2*bsize+it];
      newErr->data[12*bsize+it] = kGain->data[ 8*bsize+it]*trkErr->data[ 3*bsize+it] + kGain->data[ 9*bsize+it]*trkErr->data[ 4*bsize+it];
      newErr->data[13*bsize+it] = kGain->data[ 8*bsize+it]*trkErr->data[ 6*bsize+it] + kGain->data[ 9*bsize+it]*trkErr->data[ 7*bsize+it];
      newErr->data[14*bsize+it] = kGain->data[ 8*bsize+it]*trkErr->data[10*bsize+it] + kGain->data[ 9*bsize+it]*trkErr->data[11*bsize+it];
      newErr->data[15*bsize+it] = kGain->data[10*bsize+it]*trkErr->data[ 0*bsize+it] + kGain->data[11*bsize+it]*trkErr->data[ 1*bsize+it];
      newErr->data[16*bsize+it] = kGain->data[10*bsize+it]*trkErr->data[ 1*bsize+it] + kGain->data[11*bsize+it]*trkErr->data[ 2*bsize+it];
      newErr->data[17*bsize+it] = kGain->data[10*bsize+it]*trkErr->data[ 3*bsize+it] + kGain->data[11*bsize+it]*trkErr->data[ 4*bsize+it];
      newErr->data[18*bsize+it] = kGain->data[10*bsize+it]*trkErr->data[ 6*bsize+it] + kGain->data[11*bsize+it]*trkErr->data[ 7*bsize+it];
      newErr->data[19*bsize+it] = kGain->data[10*bsize+it]*trkErr->data[10*bsize+it] + kGain->data[11*bsize+it]*trkErr->data[11*bsize+it];
      newErr->data[20*bsize+it] = kGain->data[10*bsize+it]*trkErr->data[15*bsize+it] + kGain->data[11*bsize+it]*trkErr->data[16*bsize+it];

      newErr->data[ 0*bsize+it] = trkErr->data[ 0*bsize+it] - newErr->data[ 0*bsize+it];
      newErr->data[ 1*bsize+it] = trkErr->data[ 1*bsize+it] - newErr->data[ 1*bsize+it];
      newErr->data[ 2*bsize+it] = trkErr->data[ 2*bsize+it] - newErr->data[ 2*bsize+it];
      newErr->data[ 3*bsize+it] = trkErr->data[ 3*bsize+it] - newErr->data[ 3*bsize+it];
      newErr->data[ 4*bsize+it] = trkErr->data[ 4*bsize+it] - newErr->data[ 4*bsize+it];
      newErr->data[ 5*bsize+it] = trkErr->data[ 5*bsize+it] - newErr->data[ 5*bsize+it];
      newErr->data[ 6*bsize+it] = trkErr->data[ 6*bsize+it] - newErr->data[ 6*bsize+it];
      newErr->data[ 7*bsize+it] = trkErr->data[ 7*bsize+it] - newErr->data[ 7*bsize+it];
      newErr->data[ 8*bsize+it] = trkErr->data[ 8*bsize+it] - newErr->data[ 8*bsize+it];
      newErr->data[ 9*bsize+it] = trkErr->data[ 9*bsize+it] - newErr->data[ 9*bsize+it];
      newErr->data[10*bsize+it] = trkErr->data[10*bsize+it] - newErr->data[10*bsize+it];
      newErr->data[11*bsize+it] = trkErr->data[11*bsize+it] - newErr->data[11*bsize+it];
      newErr->data[12*bsize+it] = trkErr->data[12*bsize+it] - newErr->data[12*bsize+it];
      newErr->data[13*bsize+it] = trkErr->data[13*bsize+it] - newErr->data[13*bsize+it];
      newErr->data[14*bsize+it] = trkErr->data[14*bsize+it] - newErr->data[14*bsize+it];
      newErr->data[15*bsize+it] = trkErr->data[15*bsize+it] - newErr->data[15*bsize+it];
      newErr->data[16*bsize+it] = trkErr->data[16*bsize+it] - newErr->data[16*bsize+it];
      newErr->data[17*bsize+it] = trkErr->data[17*bsize+it] - newErr->data[17*bsize+it];
      newErr->data[18*bsize+it] = trkErr->data[18*bsize+it] - newErr->data[18*bsize+it];
      newErr->data[19*bsize+it] = trkErr->data[19*bsize+it] - newErr->data[19*bsize+it];
      newErr->data[20*bsize+it] = trkErr->data[20*bsize+it] - newErr->data[20*bsize+it];
   }

  for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x)
  {
    #pragma unroll
    for (int i = 0; i < 21; i++){
      trkErr->data[ i*bsize+it] = trkErr->data[ i*bsize+it] - newErr->data[ i*bsize+it];
    }
  }
}

__device__ __constant__ float kfact = 100./(-0.299792458*3.8112);
__device__ __forceinline__ void propagateToZ(const MP6x6SF* inErr, const MP6F* inPar, const MP1I* inChg,const MP3F* msP, 
			  MP6x6SF* outErr, MP6F* outPar, struct MP6x6F* errorProp, struct MP6x6F* temp) {
        //struct MP6x6F* errorProp, temp; 
  for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x){
    const float zout = z(msP,it);
    const float k = q(inChg,it)*kfact;//*100/3.8;
    const float deltaZ = zout - z(inPar,it);
    const float pt = 1.0f/ipt(inPar,it);
    const float cosP = cosf(phi(inPar,it));
    const float sinP = sinf(phi(inPar,it));
    const float cosT = cosf(theta(inPar,it));
    const float sinT = sinf(theta(inPar,it));
    const float pxin = cosP*pt;
    const float pyin = sinP*pt;
    const float icosT = 1.0f/cosT;
    const float icosTk = icosT/k;
    const float alpha = deltaZ*sinT*ipt(inPar,it)*icosTk;
    const float sina = sinf(alpha); // this can be approximated;
    const float cosa = cosf(alpha); // this can be approximated;
    setx(outPar,it, x(inPar,it) + k*(pxin*sina - pyin*(1.0f-cosa)) );
    sety(outPar,it, y(inPar,it) + k*(pyin*sina + pxin*(1.0f-cosa)) );
    setz(outPar,it,zout);
    setipt(outPar,it, ipt(inPar,it));
    setphi(outPar,it, phi(inPar,it)+alpha );
    settheta(outPar,it, theta(inPar,it) );
    
    const float sCosPsina = sinf(cosP*sina);
    const float cCosPsina = cosf(cosP*sina);

    //for (size_t i=0;i<6;++i) errorProp->data[bsize*PosInMtrx(i,i,6) + it] = 1.0f;
    errorProp->data[bsize*PosInMtrx(0,0,6) + it] = 1.0f;
    errorProp->data[bsize*PosInMtrx(1,1,6) + it] = 1.0f;
    errorProp->data[bsize*PosInMtrx(2,2,6) + it] = 1.0f;
    errorProp->data[bsize*PosInMtrx(3,3,6) + it] = 1.0f;
    errorProp->data[bsize*PosInMtrx(4,4,6) + it] = 1.0f;
    errorProp->data[bsize*PosInMtrx(5,5,6) + it] = 1.0f;
    //[Dec. 21, 2022] Added to have the same pattern as the cudauvm version.
    errorProp->data[bsize*PosInMtrx(0,1,6) + it] = 0.0f;
    errorProp->data[bsize*PosInMtrx(0,2,6) + it] = cosP*sinT*(sinP*cosa*sCosPsina-cosa)*icosT;
    errorProp->data[bsize*PosInMtrx(0,3,6) + it] = cosP*sinT*deltaZ*cosa*(1.0f-sinP*sCosPsina)*(icosT*pt)-k*(cosP*sina-sinP*(1.0f-cCosPsina))*(pt*pt);
    errorProp->data[bsize*PosInMtrx(0,4,6) + it] = (k*pt)*(-sinP*sina+sinP*sinP*sina*sCosPsina-cosP*(1.0f-cCosPsina));
    errorProp->data[bsize*PosInMtrx(0,5,6) + it] = cosP*deltaZ*cosa*(1.0f-sinP*sCosPsina)*(icosT*icosT);
    errorProp->data[bsize*PosInMtrx(1,2,6) + it] = cosa*sinT*(cosP*cosP*sCosPsina-sinP)*icosT;
    errorProp->data[bsize*PosInMtrx(1,3,6) + it] = sinT*deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*pt)-k*(sinP*sina+cosP*(1.0f-cCosPsina))*(pt*pt);
    errorProp->data[bsize*PosInMtrx(1,4,6) + it] = (k*pt)*(-sinP*(1.0f-cCosPsina)-sinP*cosP*sina*sCosPsina+cosP*sina);
    errorProp->data[bsize*PosInMtrx(1,5,6) + it] = deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*icosT);
    errorProp->data[bsize*PosInMtrx(4,2,6) + it] = -ipt(inPar,it)*sinT*(icosTk);
    errorProp->data[bsize*PosInMtrx(4,3,6) + it] = sinT*deltaZ*(icosTk);
    errorProp->data[bsize*PosInMtrx(4,5,6) + it] = ipt(inPar,it)*deltaZ*(icosT*icosTk);
  }
  MultHelixPropEndcap(errorProp, inErr, temp);
  MultHelixPropTranspEndcap(errorProp, temp, outErr);
}



__device__ __constant__ int ie_range = (int) nevts/num_streams; 
__device__ __constant__ int ie_rangeR = (int) nevts%num_streams; 
__global__ void GPUsequence(MPTRK* trk, MPHIT* hit, MPTRK* outtrk, const int stream){
      __shared__ struct MP6x6F errorProp, temp;
      __shared__ struct MP2x2SF resErr_loc;
      __shared__ struct MP2x6 kGain;
      __shared__ struct MP2F res_loc;
      __shared__ struct MP6x6SF newErr;
      const MPTRK* __shared__ btracks;
      MPTRK* __shared__ obtracks;
      const MPHIT* __shared__ bhits;
      __shared__ int ie;
      __shared__ int ib;
      int ti;
      int lnb = nb;
  for (ti = blockIdx.x; ti<ie_range*nb; ti+=gridDim.x){
      if(threadIdx.x == 0) {
          ie = ti/lnb;
          ib = ti%lnb;
          btracks = bTk(trk,ie,ib);
          obtracks = bTk(outtrk,ie,ib);
          (*obtracks) = (*btracks);
      }
      for (int layer=0;layer<nlayer;++layer){	
        if(threadIdx.x == 0) {
            bhits = bHit(hit,ie,ib,layer);
        }
        __syncthreads();
     
        propagateToZ(&(*obtracks).cov, &(*obtracks).par, &(*obtracks).q, &(*bhits).pos, 
                     &(*obtracks).cov, &(*obtracks).par, &errorProp, &temp);
        //KalmanUpdate(&(*obtracks).cov,&(*obtracks).par,&(*bhits).cov,&(*bhits).pos);
        KalmanUpdate_v2(&(*obtracks).cov, &(*obtracks).par, &(*bhits).cov, &(*bhits).pos, &resErr_loc, &kGain, &res_loc, &newErr);
      }
  }
}
__global__ void GPUsequenceR(MPTRK* trk, MPHIT* hit, MPTRK* outtrk, const int stream){
      __shared__ struct MP6x6F errorProp, temp;
      __shared__ struct MP2x2SF resErr_loc;
      __shared__ struct MP2x6 kGain;
      __shared__ struct MP2F res_loc;
      __shared__ struct MP6x6SF newErr;
      const MPTRK* __shared__ btracks;
      MPTRK* __shared__ obtracks;
      const MPHIT* __shared__ bhits;
      __shared__ size_t ie;
      __shared__ size_t ib;
  for (size_t ti = blockIdx.x; ti<ie_rangeR*nb; ti+=gridDim.x){
      if(threadIdx.x == 0) {
          ie = ti/nb;
          ib = ti%nb;
          btracks = bTk(trk,ie,ib);
          obtracks = bTk(outtrk,ie,ib);
          (*obtracks) = (*btracks);
      }
      for (int layer=0;layer<nlayer;++layer){	
        if(threadIdx.x == 0) {
            bhits = bHit(hit,ie,ib,layer);
        }
        __syncthreads();

        propagateToZ(&(*obtracks).cov, &(*obtracks).par, &(*obtracks).q, &(*bhits).pos, 
                     &(*obtracks).cov, &(*obtracks).par, &errorProp, &temp);
        //KalmanUpdate(&(*obtracks).cov,&(*obtracks).par,&(*bhits).cov,&(*bhits).pos);
        KalmanUpdate_v2(&(*obtracks).cov, &(*obtracks).par, &(*bhits).cov, &(*bhits).pos, &resErr_loc, &kGain, &res_loc, &newErr);
      }
  }
}

void prefetch_device(MPTRK* trk, MPHIT* hit, hipStream_t* streams, int stream_chunk, int stream_remainder, int device) {
    for (int s = 0; s<num_streams;s++){
#ifdef USE_ASYNC
      hipMemPrefetchAsync(trk+(s*stream_chunk),stream_chunk*sizeof(MPTRK), device,streams[s]);
#else
      hipMemPrefetchAsync(trk+(s*stream_chunk),stream_chunk*sizeof(MPTRK), device,0);
#endif
      
#ifdef USE_ASYNC
      hipMemPrefetchAsync(hit+(s*stream_chunk*nlayer),nlayer*stream_chunk*sizeof(MPHIT), device,streams[s]);
#else
      hipMemPrefetchAsync(hit+(s*stream_chunk*nlayer),nlayer*stream_chunk*sizeof(MPHIT), device,0);
#endif
    }  
    if(stream_remainder != 0){
#ifdef USE_ASYNC
      hipMemPrefetchAsync(trk+(num_streams*stream_chunk),stream_remainder*sizeof(MPTRK), device,streams[num_streams]);
#else
      hipMemPrefetchAsync(trk+(num_streams*stream_chunk),stream_remainder*sizeof(MPTRK), device,0);
#endif
      
#ifdef USE_ASYNC
      hipMemPrefetchAsync(hit+(num_streams*stream_chunk*nlayer),nlayer*stream_remainder*sizeof(MPHIT), device,streams[num_streams]);
#else
      hipMemPrefetchAsync(hit+(num_streams*stream_chunk*nlayer),nlayer*stream_remainder*sizeof(MPHIT), device,0);
#endif
    }
}

void prefetch_host(MPTRK* outtrk, hipStream_t* streams, int stream_chunk, int stream_remainder) {
    for (int s = 0; s<num_streams;s++){
#ifdef USE_ASYNC
      hipMemPrefetchAsync(outtrk+(s*stream_chunk),stream_chunk*sizeof(MPTRK), hipCpuDeviceId,streams[s]);
#else
      hipMemPrefetchAsync(outtrk+(s*stream_chunk),stream_chunk*sizeof(MPTRK), hipCpuDeviceId,0);
#endif
    }
    if(stream_remainder != 0){
#ifdef USE_ASYNC
      hipMemPrefetchAsync(outtrk+(num_streams*stream_chunk),stream_remainder*sizeof(MPTRK), hipCpuDeviceId,streams[num_streams]);
#else
      hipMemPrefetchAsync(outtrk+(num_streams*stream_chunk),stream_remainder*sizeof(MPTRK), hipCpuDeviceId,0);
#endif
    }
}

int main (int argc, char* argv[]) {

#ifdef USE_ASYNC
  printf("RUNNING CUDA Async Version!!\n");
#else
  printf("RUNNING CUDA Sync Version!!\n");
#endif
#ifdef include_data
  printf("Measure Both Memory Transfer Times and Compute Times!\n");
#else
  printf("Measure Compute Times Only!\n");
#endif

#include "input_track.h"

   struct AHIT inputhits[26] = {inputhit25,inputhit24,inputhit23,inputhit22,inputhit21,inputhit20,inputhit19,inputhit18,inputhit17,
				inputhit16,inputhit15,inputhit14,inputhit13,inputhit12,inputhit11,inputhit10,inputhit09,inputhit08,
				inputhit07,inputhit06,inputhit05,inputhit04,inputhit03,inputhit02,inputhit01,inputhit00};

   printf("track in pos: x=%f, y=%f, z=%f, r=%f, pt=%f, phi=%f, theta=%f \n", inputtrk.par[0], inputtrk.par[1], inputtrk.par[2],
	  sqrtf(inputtrk.par[0]*inputtrk.par[0] + inputtrk.par[1]*inputtrk.par[1]),
	  1./inputtrk.par[3], inputtrk.par[4], inputtrk.par[5]);

   printf("track in cov: %.2e, %.2e, %.2e \n", inputtrk.cov[SymOffsets66(PosInMtrx(0,0,6))],
                                               inputtrk.cov[SymOffsets66(PosInMtrx(1,1,6))],
	                                       inputtrk.cov[SymOffsets66(PosInMtrx(2,2,6))]);
   for (size_t lay=0; lay<nlayer; lay++){
     printf("hit in layer=%lu, pos: x=%f, y=%f, z=%f, r=%f \n", lay, inputhits[lay].pos[0], inputhits[lay].pos[1], inputhits[lay].pos[2], sqrtf(inputhits[lay].pos[0]*inputhits[lay].pos[0] + inputhits[lay].pos[1]*inputhits[lay].pos[1]));
   }

   printf("produce nevts=%i ntrks=%i smearing by=%f \n", nevts, ntrks, smear);
   printf("NITER=%d\n", NITER);
 
  long setup_start, setup_stop;
  struct timeval timecheck;

  gettimeofday(&timecheck, NULL);
  setup_start = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;      
#ifdef FIXED_RSEED
  //[DEBUG by Seyong on Dec. 28, 2020] add an explicit srand(1) call to generate fixed inputs for better debugging.
  srand(1);
#endif
  hipDeviceSetCacheConfig(hipFuncCachePreferShared);
//  hipFuncSetCacheConfig(reinterpret_cast<const void*>(GPUsequence),hipFuncCachePreferL1);
//  hipFuncSetCacheConfig(reinterpret_cast<const void*>(GPUsequenceR),hipFuncCachePreferL1);
  MPTRK* trk = prepareTracks(inputtrk);
  MPHIT* hit = prepareHits(inputhits);
  MPTRK* outtrk;
  hipMallocManaged((void**)&outtrk,nevts*nb*sizeof(MPTRK)); 
  dim3 grid(blockspergrid,1,1);
  dim3 block(threadsperblockx,threadsperblocky,1); 
  int device = -1;
  hipGetDevice(&device);
  int stream_chunk = ((int)(nevts/num_streams))*nb;//*sizeof(MPTRK);
  int stream_remainder = ((int)(nevts%num_streams))*nb;//*sizeof(MPTRK);
  int stream_range;
  if (stream_remainder == 0){ stream_range =num_streams;}
  else{stream_range = num_streams+1;}
  hipStream_t streams[stream_range];
  for (int s = 0; s<stream_range;s++){
    //hipStreamCreateWithFlags(&streams[s],hipStreamNonBlocking);
    hipStreamCreate(&streams[s]);
  }
#ifndef include_data
	prefetch_device(trk, hit, streams, stream_chunk, stream_remainder, device);
#ifdef USE_ASYNC
    hipDeviceSynchronize(); 
#endif
#endif

  gettimeofday(&timecheck, NULL);
  setup_stop = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;

  printf("done preparing!\n");

  printf("Size of struct MPTRK trk[] = %ld\n", nevts*nb*sizeof(struct MPTRK));
  printf("Size of struct MPTRK outtrk[] = %ld\n", nevts*nb*sizeof(struct MPTRK));
  printf("Size of struct struct MPHIT hit[] = %ld\n", nevts*nb*sizeof(struct MPHIT));
  
  auto wall_start = std::chrono::high_resolution_clock::now();

  int itr;
  for(itr=0; itr<NITER; itr++){
#ifdef include_data
	prefetch_device(trk, hit, streams, stream_chunk, stream_remainder, device);
#endif

    for (int s = 0; s<num_streams;++s){
      //printf("stream = %d, grid (%d, %d, %d), block(%d, %d, %d), stream_chunk = %d\n",s, grid.x, grid.y, grid.z, block.x, block.y, block.z, stream_chunk);
#ifdef USE_ASYNC
  	  GPUsequence<<<grid,block,0,streams[s]>>>(trk+(s*stream_chunk),hit+(s*stream_chunk*nlayer),outtrk+(s*stream_chunk),s);
#else
  	  GPUsequence<<<grid,block,0,0>>>(trk+(s*stream_chunk),hit+(s*stream_chunk*nlayer),outtrk+(s*stream_chunk),s);
#endif
    }  
    if(stream_remainder != 0){
#ifdef USE_ASYNC
  	  GPUsequenceR<<<grid,block,0,streams[num_streams]>>>(trk+(num_streams*stream_chunk),hit+(num_streams*stream_chunk*nlayer),outtrk+(num_streams*stream_chunk),num_streams);
#else
  	  GPUsequenceR<<<grid,block,0,0>>>(trk+(num_streams*stream_chunk),hit+(num_streams*stream_chunk*nlayer),outtrk+(num_streams*stream_chunk),num_streams);
#endif
    }
#ifdef include_data
	prefetch_host(outtrk, streams, stream_chunk, stream_remainder);
#endif
  } //end itr loop
  
  hipDeviceSynchronize(); 
  auto wall_stop = std::chrono::high_resolution_clock::now();
#ifndef include_data
	prefetch_host(outtrk, streams, stream_chunk, stream_remainder);
#ifdef USE_ASYNC
    hipDeviceSynchronize(); 
#endif
#endif

  for (int s = 0; s<stream_range;s++){
    hipStreamDestroy(streams[s]);
  }
 
   auto wall_diff = wall_stop - wall_start;
   auto wall_time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(wall_diff).count()) / 1e6;
   printf("setup time time=%f (s)\n", (setup_stop-setup_start)*0.001);
   printf("done ntracks=%i tot time=%f (s) time/trk=%e (s)\n", nevts*ntrks*int(NITER), wall_time, wall_time/(nevts*ntrks*int(NITER)));
   printf("formatted %i %i %i %i %i %f 0 %f %i\n",int(NITER),nevts, ntrks, bsize, nb, wall_time, (setup_stop-setup_start)*0.001, num_streams);
#ifdef DUMP_OUTPUT
   FILE *fp_x;
   FILE *fp_y;
   FILE *fp_z;
   fp_x = fopen("output_x.txt", "w");
   fp_y = fopen("output_y.txt", "w");
   fp_z = fopen("output_z.txt", "w");
#endif

   int nnans = 0, nfail = 0;
   double avgx = 0, avgy = 0, avgz = 0;
   double avgpt = 0, avgphi = 0, avgtheta = 0;
   double avgdx = 0, avgdy = 0, avgdz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       float pt_ = 1./ipt(outtrk,ie,it);
       float phi_ = phi(outtrk,ie,it);
       float theta_ = theta(outtrk,ie,it);
       float hx_ = x(hit,ie,it);
       float hy_ = y(hit,ie,it);
       float hz_ = z(hit,ie,it);
       float hr_ = sqrtf(hx_*hx_ + hy_*hy_);
       if (isnan(x_) ||
	   isnan(y_) ||
	   isnan(z_) ||
	   isnan(pt_) ||
	   isnan(phi_) ||
	   isnan(theta_)
	   ) {
	 nnans++;
	 continue;
       }
       if (fabs( (x_-hx_)/hx_ )>1. ||
           fabs( (y_-hy_)/hy_ )>1. ||
           fabs( (z_-hz_)/hz_ )>1. ||
           fabs( (pt_-12.)/12.)>1.
           ) {
	 nfail++;
	 continue;
       }
#ifdef DUMP_OUTPUT
       fprintf(fp_x, "%f\n", x_);
       fprintf(fp_y, "%f\n", y_);
       fprintf(fp_z, "%f\n", z_);
#endif
       avgpt += pt_;
       avgphi += phi_;
       avgtheta += theta_;
       avgx += x_;
       avgy += y_;
       avgz += z_;
       avgdx += (x_-hx_)/x_;
       avgdy += (y_-hy_)/y_;
       avgdz += (z_-hz_)/z_;
     }
   }
#ifdef DUMP_OUTPUT
   fclose(fp_x);
   fclose(fp_y);
   fclose(fp_z);
   fp_x = fopen("input_x.txt", "w");
   fp_y = fopen("input_y.txt", "w");
   fp_z = fopen("input_z.txt", "w");
#endif
   avgpt = avgpt/double(nevts*ntrks);
   avgphi = avgphi/double(nevts*ntrks);
   avgtheta = avgtheta/double(nevts*ntrks);
   avgx = avgx/double(nevts*ntrks);
   avgy = avgy/double(nevts*ntrks);
   avgz = avgz/double(nevts*ntrks);
   avgdx = avgdx/double(nevts*ntrks);
   avgdy = avgdy/double(nevts*ntrks);
   avgdz = avgdz/double(nevts*ntrks);

   double stdx = 0, stdy = 0, stdz = 0;
   double stddx = 0, stddy = 0, stddz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       float hx_ = x(hit,ie,it);
       float hy_ = y(hit,ie,it);
       float hz_ = z(hit,ie,it);
       float pt_ = 1./ipt(outtrk,ie,it);
       float hr_ = sqrtf(hx_*hx_ + hy_*hy_);
       if (isnan(x_) ||
	   isnan(y_) ||
	   isnan(z_)
	   ) {
	 continue;
       }
       if (fabs( (x_-hx_)/hx_ )>1. ||
           fabs( (y_-hy_)/hy_ )>1. ||
           fabs( (z_-hz_)/hz_ )>1. ||
           fabs( (pt_-12.)/12.)>1.
           ) {
         continue;
       }
       stdx += (x_-avgx)*(x_-avgx);
       stdy += (y_-avgy)*(y_-avgy);
       stdz += (z_-avgz)*(z_-avgz);
       stddx += ((x_-hx_)/x_-avgdx)*((x_-hx_)/x_-avgdx);
       stddy += ((y_-hy_)/y_-avgdy)*((y_-hy_)/y_-avgdy);
       stddz += ((z_-hz_)/z_-avgdz)*((z_-hz_)/z_-avgdz);
#ifdef DUMP_OUTPUT
       x_ = x(trk,ie,it);
       y_ = y(trk,ie,it);
       z_ = z(trk,ie,it);
       fprintf(fp_x, "%f\n", x_);
       fprintf(fp_y, "%f\n", y_);
       fprintf(fp_z, "%f\n", z_);
#endif
     }
   }
#ifdef DUMP_OUTPUT
   fclose(fp_x);
   fclose(fp_y);
   fclose(fp_z);
#endif

   stdx = sqrtf(stdx/double(nevts*ntrks));
   stdy = sqrtf(stdy/double(nevts*ntrks));
   stdz = sqrtf(stdz/double(nevts*ntrks));
   stddx = sqrtf(stddx/double(nevts*ntrks));
   stddy = sqrtf(stddy/double(nevts*ntrks));
   stddz = sqrtf(stddz/double(nevts*ntrks));

   printf("track x avg=%f std/avg=%f\n", avgx, fabs(stdx/avgx));
   printf("track y avg=%f std/avg=%f\n", avgy, fabs(stdy/avgy));
   printf("track z avg=%f std/avg=%f\n", avgz, fabs(stdz/avgz));
   printf("track dx/x avg=%f std=%f\n", avgdx, stddx);
   printf("track dy/y avg=%f std=%f\n", avgdy, stddy);
   printf("track dz/z avg=%f std=%f\n", avgdz, stddz);
   printf("track pt avg=%f\n", avgpt);
   printf("track phi avg=%f\n", avgphi);
   printf("track theta avg=%f\n", avgtheta);
   printf("number of tracks with nans=%i\n", nnans);
   printf("number of tracks failed=%i\n", nfail);
	
   //free(trk);
   //free(hit);
   //free(outtrk);
   hipFree(trk);
   hipFree(hit);
   hipFree(outtrk);
   
return 0;
}

