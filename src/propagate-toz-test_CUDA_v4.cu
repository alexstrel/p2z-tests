/*
icc propagate-toz-test.C -o propagate-toz-test.exe -fopenmp -O3
*/
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <iostream>
#include <chrono>
#include <iomanip>

//#define DUMP_OUTPUT
#define FIXED_RSEED
//#define USE_ASYNC
#ifndef USE_ASYNC
#define num_streams 1
#endif

#ifndef nevts
#define nevts 100
#endif
#ifndef bsize
#define bsize 32
#endif
#ifndef ntrks
#define ntrks 9600 //122880
#endif

#define nb    (ntrks/bsize)
#define smear 0.1

#ifndef NITER
#define NITER 5
#endif
#ifndef nlayer
#define nlayer 20
#endif
#ifndef num_streams
#define num_streams 10
#endif

#ifndef threadsperblockx
#define threadsperblockx bsize
#endif
#define threadsperblocky 1
#ifndef blockspergrid
#define blockspergrid nevts*nb/num_streams
#endif

#define HOSTDEV __host__ __device__

#define loadData(dst, src, tid, itrsize) \
  _Pragma("unroll")                      \
  for(int ip=0; ip<itrsize; ++ip) {      \
    dst[ip] = src[ip*bsize + tid];       \
  }                               

#define saveData(dst, src, tid, itrsize) \
  _Pragma("unroll")                      \
  for(int ip=0; ip<itrsize; ++ip) {      \
    dst[ip*bsize + tid] = src[ip];       \
  }                               

#define iparX     0
#define iparY     1
#define iparZ     2
#define iparIpt   3
#define iparPhi   4
#define iparTheta 5

HOSTDEV size_t PosInMtrx(size_t i, size_t j, size_t D) {
  return i*D+j;
}

HOSTDEV size_t SymOffsets33(size_t i) {
  const size_t offs[9] = {0, 1, 3, 1, 2, 4, 3, 4, 5};
  return offs[i];
}

HOSTDEV size_t SymOffsets66(size_t i) {
  const size_t offs[36] = {0, 1, 3, 6, 10, 15, 1, 2, 4, 7, 11, 16, 3, 4, 5, 8, 12, 17, 6, 7, 8, 9, 13, 18, 10, 11, 12, 13, 14, 19, 15, 16, 17, 18, 19, 20};
  return offs[i];
}


struct ATRK {
  float par[6];
  float cov[21];
  int q;
//  int hitidx[22];
};

struct AHIT {
  float pos[3];
  float cov[6];
};

struct MP1I {
  int data[1*bsize];
};

struct MP22I {
  int data[22*bsize];
};

struct MP3F {
  float data[3*bsize];
};

struct MP6F {
  float data[6*bsize];
};

struct MP3x3 {
  float data[9*bsize];
};
struct MP3x6 {
  float data[18*bsize];
};

struct MP3x3SF {
  float data[6*bsize];
};

struct MP6x6SF {
  float data[21*bsize];
};

struct MP6x6F {
  float data[36*bsize];
};

struct MPTRK {
  MP6F    par;
  MP6x6SF cov;
  MP1I    q;
//  MP22I   hitidx;
};

struct MPHIT {
  MP3F    pos;
  MP3x3SF cov;
};

struct MP1I_ {
  int data[1];
};

struct MP22I_ {
  int data[22];
};

struct MP3F_ {
  float data[3];
};

struct MP6F_ {
  float data[6];
};

struct MP3x3_ {
  float data[9];
};
struct MP3x6_ {
  float data[18];
};

struct MP3x3SF_ {
  float data[6];
};

struct MP6x6SF_ {
  float data[21];
};

struct MP6x6F_ {
  float data[36];
};

struct MPTRK_ {
  MP6F_    par;
  MP6x6SF_ cov;
  MP1I_    q;
//  MP22I_   hitidx;
};

struct MPHIT_ {
  MP3F_    pos;
  MP3x3SF_ cov;
};


float randn(float mu, float sigma) {
  float U1, U2, W, mult;
  static float X1, X2;
  static int call = 0;
  if (call == 1) {
    call = !call;
    return (mu + sigma * (float) X2);
  } do {
    U1 = -1 + ((float) rand () / RAND_MAX) * 2;
    U2 = -1 + ((float) rand () / RAND_MAX) * 2;
    W = pow (U1, 2) + pow (U2, 2);
  }
  while (W >= 1 || W == 0);
  mult = sqrt ((-2 * log (W)) / W);
  X1 = U1 * mult;
  X2 = U2 * mult;
  call = !call;
  return (mu + sigma * (float) X1);
}

MPTRK* prepareTracks(ATRK inputtrk) {
  MPTRK* result;
  hipHostMalloc((void**)&result,nevts*nb*sizeof(MPTRK), hipHostMallocDefault);
  for (size_t ie=0;ie<nevts;++ie) {
    for (size_t ib=0;ib<nb;++ib) {
      for (size_t it=0;it<bsize;++it) {
        //par
        for (size_t ip=0;ip<6;++ip) {
          result[ib + nb*ie].par.data[it + ip*bsize] = (1+smear*randn(0,1))*inputtrk.par[ip];
        }
        //cov
        for (size_t ip=0;ip<21;++ip) {
          result[ib + nb*ie].cov.data[it + ip*bsize] = (1+smear*randn(0,1))*inputtrk.cov[ip];
        }
        //q
        result[ib + nb*ie].q.data[it] = inputtrk.q-2*ceil(-0.5 + (float)rand() / RAND_MAX);//fixme check
      }
    }
  }
  return result;
}

MPHIT* prepareHits(AHIT inputhit) {
  MPHIT* result;
  hipHostMalloc((void**)&result,nlayer*nevts*nb*sizeof(MPHIT), hipHostMallocDefault);
  for (size_t lay=0;lay<nlayer;++lay) {
    for (size_t ie=0;ie<nevts;++ie) {
      for (size_t ib=0;ib<nb;++ib) {
        for (size_t it=0;it<bsize;++it) {
          //pos
          for (size_t ip=0;ip<3;++ip) {
            result[lay+nlayer*(ib + nb*ie)].pos.data[it + ip*bsize] = (1+smear*randn(0,1))*inputhit.pos[ip];
          }
          //cov
          for (size_t ip=0;ip<6;++ip) {
            result[lay+nlayer*(ib + nb*ie)].cov.data[it + ip*bsize] = (1+smear*randn(0,1))*inputhit.cov[ip];
          }
        }
      }
    }
  }
  return result;
}


HOSTDEV MPTRK* bTk(MPTRK* tracks, size_t ev, size_t ib) {
  return &(tracks[ib + nb*ev]);
}

HOSTDEV const MPTRK* bTk(const MPTRK* tracks, size_t ev, size_t ib) {
  return &(tracks[ib + nb*ev]);
}


HOSTDEV float q(const MP1I* bq, size_t it){
  return (*bq).data[it];
}

HOSTDEV float par(const MP6F* bpars, size_t it, size_t ipar){
  return (*bpars).data[it + ipar*bsize];
}
HOSTDEV float x    (const MP6F* bpars, size_t it){ return par(bpars, it, 0); }
HOSTDEV float y    (const MP6F* bpars, size_t it){ return par(bpars, it, 1); }
HOSTDEV float z    (const MP6F* bpars, size_t it){ return par(bpars, it, 2); }
HOSTDEV float ipt  (const MP6F* bpars, size_t it){ return par(bpars, it, 3); }
HOSTDEV float phi  (const MP6F* bpars, size_t it){ return par(bpars, it, 4); }
HOSTDEV float theta(const MP6F* bpars, size_t it){ return par(bpars, it, 5); }

HOSTDEV float par(const MPTRK* btracks, size_t it, size_t ipar){
  return par(&(*btracks).par,it,ipar);
}
HOSTDEV float x    (const MPTRK* btracks, size_t it){ return par(btracks, it, 0); }
HOSTDEV float y    (const MPTRK* btracks, size_t it){ return par(btracks, it, 1); }
HOSTDEV float z    (const MPTRK* btracks, size_t it){ return par(btracks, it, 2); }
HOSTDEV float ipt  (const MPTRK* btracks, size_t it){ return par(btracks, it, 3); }
HOSTDEV float phi  (const MPTRK* btracks, size_t it){ return par(btracks, it, 4); }
HOSTDEV float theta(const MPTRK* btracks, size_t it){ return par(btracks, it, 5); }

HOSTDEV float par(const MPTRK* tracks, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  const MPTRK* btracks = bTk(tracks, ev, ib);
  size_t it = tk % bsize;
  return par(btracks, it, ipar);
}

HOSTDEV float x    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 0); }
HOSTDEV float y    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 1); }
HOSTDEV float z    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 2); }
HOSTDEV float ipt  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 3); }
HOSTDEV float phi  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 4); }
HOSTDEV float theta(const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 5); }

HOSTDEV void setpar(MP6F* bpars, size_t it, size_t ipar, float val){
  (*bpars).data[it + ipar*bsize] = val;
}
HOSTDEV void setx    (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 0, val); }
HOSTDEV void sety    (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 1, val); }
HOSTDEV void setz    (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 2, val); }
HOSTDEV void setipt  (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 3, val); }
HOSTDEV void setphi  (MP6F* bpars, size_t it, float val){ setpar(bpars, it, 4, val); }
HOSTDEV void settheta(MP6F* bpars, size_t it, float val){ setpar(bpars, it, 5, val); }

HOSTDEV void setpar(MPTRK* btracks, size_t it, size_t ipar, float val){
  setpar(&(*btracks).par,it,ipar,val);
}
HOSTDEV void setx    (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 0, val); }
HOSTDEV void sety    (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 1, val); }
HOSTDEV void setz    (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 2, val); }
HOSTDEV void setipt  (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 3, val); }
HOSTDEV void setphi  (MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 4, val); }
HOSTDEV void settheta(MPTRK* btracks, size_t it, float val){ setpar(btracks, it, 5, val); }

HOSTDEV MPHIT* bHit(MPHIT* hits, size_t ev, size_t ib) {
  return &(hits[ib + nb*ev]);
}
HOSTDEV const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib) {
  return &(hits[ib + nb*ev]);
}
HOSTDEV const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib,int lay) {
  return &(hits[lay + (ib*nlayer) +(ev*nlayer*nb)]);
}

HOSTDEV float pos(const MP3F* hpos, size_t it, size_t ipar){
  return (*hpos).data[it + ipar*bsize];
}
HOSTDEV float x(const MP3F* hpos, size_t it)    { return pos(hpos, it, 0); }
HOSTDEV float y(const MP3F* hpos, size_t it)    { return pos(hpos, it, 1); }
HOSTDEV float z(const MP3F* hpos, size_t it)    { return pos(hpos, it, 2); }

HOSTDEV float pos(const MPHIT* hits, size_t it, size_t ipar){
  return pos(&(*hits).pos,it,ipar);
}
HOSTDEV float x(const MPHIT* hits, size_t it)    { return pos(hits, it, 0); }
HOSTDEV float y(const MPHIT* hits, size_t it)    { return pos(hits, it, 1); }
HOSTDEV float z(const MPHIT* hits, size_t it)    { return pos(hits, it, 2); }

HOSTDEV float pos(const MPHIT* hits, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  //[DEBUG by Seyong on Dec. 28, 2020] add 4th argument(nlayer-1) to bHit() below.
  const MPHIT* bhits = bHit(hits, ev, ib, nlayer-1);
  size_t it = tk % bsize;
  return pos(bhits,it,ipar);
}
HOSTDEV float x(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 0); }
HOSTDEV float y(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 1); }
HOSTDEV float z(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 2); }



#define N bsize
__forceinline__ __device__ void MultHelixPropEndcap(const MP6x6F_* A, const MP6x6SF_* B, MP6x6F_* C) {
  const float *a = A->data; //ASSUME_ALIGNED(a, 64);
  const float *b = B->data; //ASSUME_ALIGNED(b, 64);
  float *c = C->data;       //ASSUME_ALIGNED(c, 64);
  {
    c[ 0] = b[ 0] + a[ 2]*b[ 3] + a[ 3]*b[ 6] + a[ 4]*b[10] + a[ 5]*b[15];
    c[ 1] = b[ 1] + a[ 2]*b[ 4] + a[ 3]*b[ 7] + a[ 4]*b[11] + a[ 5]*b[16];
    c[ 2] = b[ 3] + a[ 2]*b[ 5] + a[ 3]*b[ 8] + a[ 4]*b[12] + a[ 5]*b[17];
    c[ 3] = b[ 6] + a[ 2]*b[ 8] + a[ 3]*b[ 9] + a[ 4]*b[13] + a[ 5]*b[18];
    c[ 4] = b[10] + a[ 2]*b[12] + a[ 3]*b[13] + a[ 4]*b[14] + a[ 5]*b[19];
    c[ 5] = b[15] + a[ 2]*b[17] + a[ 3]*b[18] + a[ 4]*b[19] + a[ 5]*b[20];
    c[ 6] = b[ 1] + a[ 8]*b[ 3] + a[ 9]*b[ 6] + a[10]*b[10] + a[11]*b[15];
    c[ 7] = b[ 2] + a[ 8]*b[ 4] + a[ 9]*b[ 7] + a[10]*b[11] + a[11]*b[16];
    c[ 8] = b[ 4] + a[ 8]*b[ 5] + a[ 9]*b[ 8] + a[10]*b[12] + a[11]*b[17];
    c[ 9] = b[ 7] + a[ 8]*b[ 8] + a[ 9]*b[ 9] + a[10]*b[13] + a[11]*b[18];
    c[10] = b[11] + a[ 8]*b[12] + a[ 9]*b[13] + a[10]*b[14] + a[11]*b[19];
    c[11] = b[16] + a[ 8]*b[17] + a[ 9]*b[18] + a[10]*b[19] + a[11]*b[20];
    c[12] = 0;
    c[13] = 0;
    c[14] = 0;
    c[15] = 0;
    c[16] = 0;
    c[17] = 0;
    c[18] = b[ 6];
    c[19] = b[ 7];
    c[20] = b[ 8];
    c[21] = b[ 9];
    c[22] = b[13];
    c[23] = b[18];
    c[24] = a[26]*b[ 3] + a[27]*b[ 6] + b[10] + a[29]*b[15];
    c[25] = a[26]*b[ 4] + a[27]*b[ 7] + b[11] + a[29]*b[16];
    c[26] = a[26]*b[ 5] + a[27]*b[ 8] + b[12] + a[29]*b[17];
    c[27] = a[26]*b[ 8] + a[27]*b[ 9] + b[13] + a[29]*b[18];
    c[28] = a[26]*b[12] + a[27]*b[13] + b[14] + a[29]*b[19];
    c[29] = a[26]*b[17] + a[27]*b[18] + b[19] + a[29]*b[20];
    c[30] = b[15];
    c[31] = b[16];
    c[32] = b[17];
    c[33] = b[18];
    c[34] = b[19];
    c[35] = b[20];
  }
}

__forceinline__ __device__ void MultHelixPropTranspEndcap(MP6x6F_* A, MP6x6F_* B, MP6x6SF_* C) {
  const float *a = A->data; //ASSUME_ALIGNED(a, 64);
  const float *b = B->data; //ASSUME_ALIGNED(b, 64);
  float *c = C->data;       //ASSUME_ALIGNED(c, 64);
  {
    c[ 0] = b[ 0] + b[ 2]*a[ 2] + b[ 3]*a[ 3] + b[ 4]*a[ 4] + b[ 5]*a[ 5];
    c[ 1] = b[ 6] + b[ 8]*a[ 2] + b[ 9]*a[ 3] + b[10]*a[ 4] + b[11]*a[ 5];
    c[ 2] = b[ 7] + b[ 8]*a[ 8] + b[ 9]*a[ 9] + b[10]*a[10] + b[11]*a[11];
    c[ 3] = b[12] + b[14]*a[ 2] + b[15]*a[ 3] + b[16]*a[ 4] + b[17]*a[ 5];
    c[ 4] = b[13] + b[14]*a[ 8] + b[15]*a[ 9] + b[16]*a[10] + b[17]*a[11];
    c[ 5] = 0;
    c[ 6] = b[18] + b[20]*a[ 2] + b[21]*a[ 3] + b[22]*a[ 4] + b[23]*a[ 5];
    c[ 7] = b[19] + b[20]*a[ 8] + b[21]*a[ 9] + b[22]*a[10] + b[23]*a[11];
    c[ 8] = 0;
    c[ 9] = b[21];
    c[10] = b[24] + b[26]*a[ 2] + b[27]*a[ 3] + b[28]*a[ 4] + b[29]*a[ 5];
    c[11] = b[25] + b[26]*a[ 8] + b[27]*a[ 9] + b[28]*a[10] + b[29]*a[11];
    c[12] = 0;
    c[13] = b[27];
    c[14] = b[26]*a[26] + b[27]*a[27] + b[28] + b[29]*a[29];
    c[15] = b[30] + b[32]*a[ 2] + b[33]*a[ 3] + b[34]*a[ 4] + b[35]*a[ 5];
    c[16] = b[31] + b[32]*a[ 8] + b[33]*a[ 9] + b[34]*a[10] + b[35]*a[11];
    c[17] = 0;
    c[18] = b[33];
    c[19] = b[32]*a[26] + b[33]*a[27] + b[34] + b[35]*a[29];
    c[20] = b[35];
  }
}

__forceinline__ __device__ void KalmanGainInv(const MP6x6SF_* A, const MP3x3SF_* B, MP3x3_* C) {
  // k = P Ht(HPHt + R)^-1
  // HpHt -> cov of x,y,z. take upper 3x3 matrix of P
  // This calculates the inverse of HpHt +R
  const float *a = A->data; //ASSUME_ALIGNED(a, 64);
  const float *b = B->data; //ASSUME_ALIGNED(b, 64);
  float *c = C->data;       //ASSUME_ALIGNED(c, 64);
  {
    double det =
      ((a[0]+b[0])*(((a[ 6]+b[ 3]) *(a[11]+b[5])) - ((a[7]+b[4]) *(a[7]+b[4])))) -
      ((a[1]+b[1])*(((a[ 1]+b[ 1]) *(a[11]+b[5])) - ((a[7]+b[4]) *(a[2]+b[2])))) +
      ((a[2]+b[2])*(((a[ 1]+b[ 1]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[6]+b[3]))));
    double invdet = 1.0/det;

    c[ 0] =  invdet*(((a[ 6]+b[ 3]) *(a[11]+b[5])) - ((a[7]+b[4]) *(a[7]+b[4])));
    c[ 1] =  -invdet*(((a[ 1]+b[ 1]) *(a[11]+b[5])) - ((a[2]+b[2]) *(a[7]+b[4])));
    c[ 2] =  invdet*(((a[ 1]+b[ 1]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[7]+b[4])));
    c[ 3] =  -invdet*(((a[ 1]+b[ 1]) *(a[11]+b[5])) - ((a[7]+b[4]) *(a[2]+b[2])));
    c[ 4] =  invdet*(((a[ 0]+b[ 0]) *(a[11]+b[5])) - ((a[2]+b[2]) *(a[2]+b[2])));
    c[ 5] =  -invdet*(((a[ 0]+b[ 0]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[1]+b[1])));
    c[ 6] =  invdet*(((a[ 1]+b[ 1]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[6]+b[3])));
    c[ 7] =  -invdet*(((a[ 0]+b[ 0]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[1]+b[1])));
    c[ 8] =  invdet*(((a[ 0]+b[ 0]) *(a[6]+b[3])) - ((a[1]+b[1]) *(a[1]+b[1])));
  }
}

__forceinline__ __device__ void KalmanGain(const MP6x6SF_* A, const MP3x3_* B, MP3x6_* C) {
  // k = P Ht(HPHt + R)^-1
  // HpHt -> cov of x,y,z. take upper 3x3 matrix of P
  // This calculates the kalman gain 
  const float *a = A->data; //ASSUME_ALIGNED(a, 64);
  const float *b = B->data; //ASSUME_ALIGNED(b, 64);
  float *c = C->data;       //ASSUME_ALIGNED(c, 64);
  {
    c[ 0] = a[0]*b[0] + a[1]*b[3] + a[2]*b[6];
    c[ 1] = a[0]*b[1] + a[1]*b[4] + a[2]*b[7];
    c[ 2] = a[0]*b[2] + a[1]*b[5] + a[2]*b[8];
    c[ 3] = a[1]*b[0] + a[6]*b[3] + a[7]*b[6];
    c[ 4] = a[1]*b[1] + a[6]*b[4] + a[7]*b[7];
    c[ 5] = a[1]*b[2] + a[6]*b[5] + a[7]*b[8];
    c[ 6] = a[2]*b[0] + a[7]*b[3] + a[11]*b[6];
    c[ 7] = a[2]*b[1] + a[7]*b[4] + a[11]*b[7];
    c[ 8] = a[2]*b[2] + a[7]*b[5] + a[11]*b[8];
    c[ 9] = a[3]*b[0] + a[8]*b[3] + a[12]*b[6];
    c[ 10] = a[3]*b[1] + a[8]*b[4] + a[12]*b[7];
    c[ 11] = a[3]*b[2] + a[8]*b[5] + a[12]*b[8];
    c[ 12] = a[4]*b[0] + a[9]*b[3] + a[13]*b[6];
    c[ 13] = a[4]*b[1] + a[9]*b[4] + a[13]*b[7];
    c[ 14] = a[4]*b[2] + a[9]*b[5] + a[13]*b[8];
    c[ 15] = a[5]*b[0] + a[10]*b[3] + a[14]*b[6];
    c[ 16] = a[5]*b[1] + a[10]*b[4] + a[14]*b[7];
    c[ 17] = a[5]*b[2] + a[10]*b[5] + a[14]*b[8];
  }
}

__forceinline__ __device__ void KalmanUpdate(MP6x6SF_* trkErr, MP6F_* inPar, const MP3x3SF_* hitErr, const MP3F_* msP){
  MP3x3_ inverse_temp;
  MP3x6_ kGain;
  MP6x6SF_ newErr;
  KalmanGainInv(trkErr,hitErr,&inverse_temp);
  KalmanGain(trkErr,&inverse_temp,&kGain);

  {
    float *inParData = inPar->data;
    float *trkErrData = trkErr->data;
    const float xin = inParData[iparX];
    const float yin = inParData[iparY];
    const float zin = inParData[iparZ];
    const float ptin = 1.0f/inParData[iparIpt]; // is this pt or ipt? 
    const float phiin = inParData[iparPhi];
    const float thetain = inParData[iparTheta];
    const float xout = msP->data[iparX];
    const float yout = msP->data[iparY];
    //const float zout = msP->data[iparZ];
  
    float xnew = xin + (kGain.data[0]*(xout-xin)) +(kGain.data[1]*(yout-yin));
    float ynew = yin + (kGain.data[3]*(xout-xin)) +(kGain.data[4]*(yout-yin));
    float znew = zin + (kGain.data[6]*(xout-xin)) +(kGain.data[7]*(yout-yin));
    float ptnew = ptin + (kGain.data[9]*(xout-xin)) +(kGain.data[10]*(yout-yin));
    float phinew = phiin + (kGain.data[12]*(xout-xin)) +(kGain.data[13]*(yout-yin));
    float thetanew = thetain + (kGain.data[15]*(xout-xin)) +(kGain.data[16]*(yout-yin));
  
    newErr.data[0] = trkErrData[0] - (kGain.data[0]*trkErrData[0]+kGain.data[1]*trkErrData[1]+kGain.data[2]*trkErrData[2]);
    newErr.data[1] = trkErrData[1] - (kGain.data[0]*trkErrData[1]+kGain.data[1]*trkErrData[6]+kGain.data[2]*trkErrData[7]);
    newErr.data[2] = trkErrData[2] - (kGain.data[0]*trkErrData[2]+kGain.data[1]*trkErrData[7]+kGain.data[2]*trkErrData[11]);
    newErr.data[3] = trkErrData[3] - (kGain.data[0]*trkErrData[3]+kGain.data[1]*trkErrData[8]+kGain.data[2]*trkErrData[12]);
    newErr.data[4] = trkErrData[4] - (kGain.data[0]*trkErrData[4]+kGain.data[1]*trkErrData[9]+kGain.data[2]*trkErrData[13]);
    newErr.data[5] = trkErrData[5] - (kGain.data[0]*trkErrData[5]+kGain.data[1]*trkErrData[10]+kGain.data[2]*trkErrData[14]);
  
    newErr.data[6] = trkErrData[6] - (kGain.data[3]*trkErrData[1]+kGain.data[4]*trkErrData[6]+kGain.data[5]*trkErrData[7]);
    newErr.data[7] = trkErrData[7] - (kGain.data[3]*trkErrData[2]+kGain.data[4]*trkErrData[7]+kGain.data[5]*trkErrData[11]);
    newErr.data[8] = trkErrData[8] - (kGain.data[3]*trkErrData[3]+kGain.data[4]*trkErrData[8]+kGain.data[5]*trkErrData[12]);
    newErr.data[9] = trkErrData[9] - (kGain.data[3]*trkErrData[4]+kGain.data[4]*trkErrData[9]+kGain.data[5]*trkErrData[13]);
    newErr.data[10] = trkErrData[10] - (kGain.data[3]*trkErrData[5]+kGain.data[4]*trkErrData[10]+kGain.data[5]*trkErrData[14]);
  
    newErr.data[11] = trkErrData[11] - (kGain.data[6]*trkErrData[2]+kGain.data[7]*trkErrData[7]+kGain.data[8]*trkErrData[11]);
    newErr.data[12] = trkErrData[12] - (kGain.data[6]*trkErrData[3]+kGain.data[7]*trkErrData[8]+kGain.data[8]*trkErrData[12]);
    newErr.data[13] = trkErrData[13] - (kGain.data[6]*trkErrData[4]+kGain.data[7]*trkErrData[9]+kGain.data[8]*trkErrData[13]);
    newErr.data[14] = trkErrData[14] - (kGain.data[6]*trkErrData[5]+kGain.data[7]*trkErrData[10]+kGain.data[8]*trkErrData[14]);
  
    newErr.data[15] = trkErrData[15] - (kGain.data[9]*trkErrData[3]+kGain.data[10]*trkErrData[8]+kGain.data[11]*trkErrData[12]);
    newErr.data[16] = trkErrData[16] - (kGain.data[9]*trkErrData[4]+kGain.data[10]*trkErrData[9]+kGain.data[11]*trkErrData[13]);
    newErr.data[17] = trkErrData[17] - (kGain.data[9]*trkErrData[5]+kGain.data[10]*trkErrData[10]+kGain.data[11]*trkErrData[14]);
  
    newErr.data[18] = trkErrData[18] - (kGain.data[12]*trkErrData[4]+kGain.data[13]*trkErrData[9]+kGain.data[14]*trkErrData[13]);
    newErr.data[19] = trkErrData[19] - (kGain.data[12]*trkErrData[5]+kGain.data[13]*trkErrData[10]+kGain.data[14]*trkErrData[14]);
  
    newErr.data[20] = trkErrData[20] - (kGain.data[15]*trkErrData[5]+kGain.data[16]*trkErrData[10]+kGain.data[17]*trkErrData[14]);
  
    inParData[iparX] = xnew;
    inParData[iparY] = ynew;
    inParData[iparZ] = znew;
    inParData[iparIpt] = ptnew;
    inParData[iparPhi] = phinew;
    inParData[iparTheta] = thetanew;
    #pragma unroll
    for (int i = 0; i < 21; i++){
      trkErrData[ i] = trkErrData[ i] - newErr.data[ i];
    }
  }
}

__device__ __constant__ float kfact = 100/3.8;
__device__ __forceinline__ void propagateToZ(const MP6x6SF_* inErr, const MP6F_* inPar, const MP1I_* inChg,const MP3F_* msP, 
			  MP6x6SF_* outErr, MP6F_* outPar) {
    struct MP6x6F_ errorProp, temp; 
  //for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x){
  {
    const float *inParData = inPar->data;
    float *outParData = outPar->data;
    const float zout = msP->data[iparZ];
    const float k = inChg->data[0]*kfact;//*100/3.8;
    const float deltaZ = zout - inParData[iparZ];
    const float ipt_ = inParData[iparIpt];
    const float pt = 1.0f/ipt_;
    const float phi_ = inParData[iparPhi];
    const float cosP = cosf(phi_);
    const float sinP = sinf(phi_);
    const float theta_ = inParData[iparTheta];
    const float cosT = cosf(theta_);
    const float sinT = sinf(theta_);
    const float pxin = cosP*pt;
    const float pyin = sinP*pt;
    const float icosT = 1.0f/cosT;
    const float icosTk = icosT/k;
    const float alpha = deltaZ*sinT*ipt_*icosTk;
    const float sina = sinf(alpha); // this can be approximated;
    const float cosa = cosf(alpha); // this can be approximated;
    outParData[iparX] = inParData[iparX] + k*(pxin*sina - pyin*(1.0f-cosa));
    outParData[iparY] = inParData[iparY] + k*(pyin*sina + pxin*(1.0f-cosa));
    outParData[iparZ] = zout;
    outParData[iparIpt] = ipt_;
    outParData[iparPhi] = phi_+alpha;
    outParData[iparTheta] = theta_;
    
    const float sCosPsina = sinf(cosP*sina);
    const float cCosPsina = cosf(cosP*sina);
    
    //for (size_t i=0;i<6;++i) errorProp.data[PosInMtrx(i,i,6) + it] = 1.f;
    errorProp.data[PosInMtrx(0,0,6)] = 1.0f;
    errorProp.data[PosInMtrx(1,1,6)] = 1.0f;
    errorProp.data[PosInMtrx(2,2,6)] = 1.0f;
    errorProp.data[PosInMtrx(3,3,6)] = 1.0f;
    errorProp.data[PosInMtrx(4,4,6)] = 1.0f;
    errorProp.data[PosInMtrx(5,5,6)] = 1.0f;
    //[Dec. 21, 2022] Added to have the same pattern as the cudauvm version.
    errorProp.data[PosInMtrx(0,1,6)] = 0.0f;
    errorProp.data[PosInMtrx(0,2,6)] = cosP*sinT*(sinP*cosa*sCosPsina-cosa)*icosT;
    errorProp.data[PosInMtrx(0,3,6)] = cosP*sinT*deltaZ*cosa*(1.0f-sinP*sCosPsina)*(icosT*pt)-k*(cosP*sina-sinP*(1.0f-cCosPsina))*(pt*pt);
    errorProp.data[PosInMtrx(0,4,6)] = (k*pt)*(-sinP*sina+sinP*sinP*sina*sCosPsina-cosP*(1.0f-cCosPsina));
    errorProp.data[PosInMtrx(0,5,6)] = cosP*deltaZ*cosa*(1.0f-sinP*sCosPsina)*(icosT*icosT);
    errorProp.data[PosInMtrx(1,2,6)] = cosa*sinT*(cosP*cosP*sCosPsina-sinP)*icosT;
    errorProp.data[PosInMtrx(1,3,6)] = sinT*deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*pt)-k*(sinP*sina+cosP*(1.0f-cCosPsina))*(pt*pt);
    errorProp.data[PosInMtrx(1,4,6)] = (k*pt)*(-sinP*(1.0f-cCosPsina)-sinP*cosP*sina*sCosPsina+cosP*sina);
    errorProp.data[PosInMtrx(1,5,6)] = deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*icosT);
    errorProp.data[PosInMtrx(4,2,6)] = -ipt_*sinT*(icosTk);
    errorProp.data[PosInMtrx(4,3,6)] = sinT*deltaZ*(icosTk);
    errorProp.data[PosInMtrx(4,5,6)] = ipt_*deltaZ*(icosT*icosTk);
  }
  MultHelixPropEndcap(&errorProp, inErr, &temp);
  MultHelixPropTranspEndcap(&errorProp, &temp, outErr);
}

__device__ __constant__ int ie_range = (int) nevts/num_streams; 
__device__ __constant__ int ie_rangeR = (int) nevts%num_streams; 
__global__ void GPUsequence(MPTRK* trk, MPHIT* hit, MPTRK* outtrk, const int stream){
  for (int ti = blockIdx.x; ti<ie_range*nb; ti+=gridDim.x){
      struct MPTRK_ obtracks;
      struct MPTRK_ btracks;
      float *dstPtr = btracks.par.data;
      float *srcPtr = trk[ti].par.data;
      loadData(dstPtr,srcPtr,threadIdx.x,6);
      dstPtr = btracks.cov.data;
      srcPtr = trk[ti].cov.data;
      loadData(dstPtr,srcPtr,threadIdx.x,21);
      int *dstPtrI = btracks.q.data;
      int *srcPtrI = trk[ti].q.data;
      loadData(dstPtrI,srcPtrI,threadIdx.x,1);

#pragma unroll
      for (int layer=0;layer<nlayer;++layer){	
        struct MPHIT_ bhits;
        float *dstPtr2 = bhits.pos.data;
        float *srcPtr2 = hit[layer+ti*nlayer].pos.data;
        loadData(dstPtr2,srcPtr2,threadIdx.x,3);
        dstPtr2 = bhits.cov.data;
        srcPtr2 = hit[layer+ti*nlayer].cov.data;
        loadData(dstPtr2,srcPtr2,threadIdx.x,6);
     
        propagateToZ(&(btracks.cov), &(btracks.par), &(btracks.q), &(bhits.pos), 
                     &(obtracks.cov), &(obtracks.par));
        KalmanUpdate(&(obtracks.cov),&(obtracks.par),&(bhits.cov),&(bhits.pos));
      }
      dstPtr = outtrk[ti].par.data;
      srcPtr = obtracks.par.data;
      saveData(dstPtr,srcPtr,threadIdx.x,6);
      dstPtr = outtrk[ti].cov.data;
      srcPtr = obtracks.cov.data;
      saveData(dstPtr,srcPtr,threadIdx.x,21);
      dstPtrI = outtrk[ti].q.data;
      srcPtrI = obtracks.q.data;
      saveData(dstPtrI,srcPtrI,threadIdx.x,1);
  }
}

__global__ void GPUsequenceR(MPTRK* trk, MPHIT* hit, MPTRK* outtrk, const int stream){
  for (int ti = blockIdx.x; ti<ie_rangeR*nb; ti+=gridDim.x){
      struct MPTRK_ obtracks;
      struct MPTRK_ btracks;
      float *dstPtr = btracks.par.data;
      float *srcPtr = trk[ti].par.data;
      loadData(dstPtr,srcPtr,threadIdx.x,6);
      dstPtr = btracks.cov.data;
      srcPtr = trk[ti].cov.data;
      loadData(dstPtr,srcPtr,threadIdx.x,21);
      int *dstPtrI = btracks.q.data;
      int *srcPtrI = trk[ti].q.data;
      loadData(dstPtrI,srcPtrI,threadIdx.x,1);

#pragma unroll
      for (int layer=0;layer<nlayer;++layer){	
        struct MPHIT_ bhits;
        float *dstPtr2 = bhits.pos.data;
        float *srcPtr2 = hit[layer+ti*nlayer].pos.data;
        loadData(dstPtr2,srcPtr2,threadIdx.x,3);
        dstPtr2 = bhits.cov.data;
        srcPtr2 = hit[layer+ti*nlayer].cov.data;
        loadData(dstPtr2,srcPtr2,threadIdx.x,6);
     
        propagateToZ(&(btracks.cov), &(btracks.par), &(btracks.q), &(bhits.pos), 
                     &(obtracks.cov), &(obtracks.par));
        KalmanUpdate(&(obtracks.cov),&(obtracks.par),&(bhits.cov),&(bhits.pos));
      }
      dstPtr = outtrk[ti].par.data;
      srcPtr = obtracks.par.data;
      saveData(dstPtr,srcPtr,threadIdx.x,6);
      dstPtr = outtrk[ti].cov.data;
      srcPtr = obtracks.cov.data;
      saveData(dstPtr,srcPtr,threadIdx.x,21);
      dstPtrI = outtrk[ti].q.data;
      srcPtrI = obtracks.q.data;
      saveData(dstPtrI,srcPtrI,threadIdx.x,1);
  }
}

void memcpy_host2dev(MPTRK* trk_dev, MPTRK* trk, MPHIT* hit_dev, MPHIT* hit, hipStream_t* streams, int stream_chunk, int stream_remainder) {
    for (int s = 0; s<num_streams;s++){
#ifdef USE_ASYNC
      cudaMemcpyAsync(trk_dev+(s*stream_chunk), trk+(s*stream_chunk), stream_chunk*sizeof(MPTRK), cudaMemcpyHostToDevice, streams[s]);
#else
      hipMemcpy(trk_dev+(s*stream_chunk), trk+(s*stream_chunk), stream_chunk*sizeof(MPTRK), hipMemcpyHostToDevice);
#endif
      
#ifdef USE_ASYNC
      cudaMemcpyAsync(hit_dev+(s*stream_chunk*nlayer),hit+(s*stream_chunk*nlayer),nlayer*stream_chunk*sizeof(MPHIT), cudaMemcpyHostToDevice, streams[s]);
#else
      hipMemcpy(hit_dev+(s*stream_chunk*nlayer),hit+(s*stream_chunk*nlayer),nlayer*stream_chunk*sizeof(MPHIT), hipMemcpyHostToDevice);
#endif
    }  
    if(stream_remainder != 0){
#ifdef USE_ASYNC
      cudaMemcpyAsync(trk_dev+(num_streams*stream_chunk), trk+(num_streams*stream_chunk), stream_remainder*sizeof(MPTRK), cudaMemcpyHostToDevice, streams[num_streams]);
#else
      hipMemcpy(trk_dev+(num_streams*stream_chunk), trk+(num_streams*stream_chunk), stream_remainder*sizeof(MPTRK), hipMemcpyHostToDevice);
#endif
      
#ifdef USE_ASYNC
      cudaMemcpyAsync(hit_dev+(num_streams*stream_chunk*nlayer),hit+(num_streams*stream_chunk*nlayer),nlayer*stream_remainder*sizeof(MPHIT), cudaMemcpyHostToDevice, streams[num_streams]);
#else
      hipMemcpy(hit_dev+(num_streams*stream_chunk*nlayer),hit+(num_streams*stream_chunk*nlayer),nlayer*stream_remainder*sizeof(MPHIT), hipMemcpyHostToDevice);
#endif
    }
}

void memcpy_dev2host(MPTRK* outtrk, MPTRK* outtrk_dev, hipStream_t* streams, int stream_chunk, int stream_remainder) {
    for (int s = 0; s<num_streams;s++){
#ifdef USE_ASYNC
      cudaMemcpyAsync(outtrk+(s*stream_chunk), outtrk_dev+(s*stream_chunk), stream_chunk*sizeof(MPTRK), cudaMemcpyDeviceToHost, streams[s]);
#else
      hipMemcpy(outtrk+(s*stream_chunk), outtrk_dev+(s*stream_chunk), stream_chunk*sizeof(MPTRK), hipMemcpyDeviceToHost);
#endif
    }
    if(stream_remainder != 0){
#ifdef USE_ASYNC
      cudaMemcpyAsync(outtrk+(num_streams*stream_chunk), outtrk_dev+(num_streams*stream_chunk), stream_remainder*sizeof(MPTRK), cudaMemcpyDeviceToHost, streams[num_streams]);
#else
      hipMemcpy(outtrk+(num_streams*stream_chunk), outtrk_dev+(num_streams*stream_chunk), stream_remainder*sizeof(MPTRK), hipMemcpyDeviceToHost);
#endif
    }
}

int main (int argc, char* argv[]) {

#ifdef USE_ASYNC
  printf("RUNNING CUDA Async Version!!\n");
#else
  printf("RUNNING CUDA Sync Version!!\n");
#endif
#ifdef include_data
  printf("Measure Both Memory Transfer Times and Compute Times!\n");
#else
  printf("Measure Compute Times Only!\n");
#endif
  printf("Streams: %d, blocks: %d, threads(x,y): (%d,%d)\n",num_streams,blockspergrid,threadsperblockx,threadsperblocky);
  int itr;
  ATRK inputtrk = {
     {-12.806846618652344, -7.723824977874756, 38.13014221191406,0.23732035065189902, -2.613372802734375, 0.35594117641448975},
     {6.290299552347278e-07,4.1375109560704004e-08,7.526661534029699e-07,2.0973730840978533e-07,1.5431574240665213e-07,9.626245400795597e-08,-2.804026640189443e-06,
      6.219111130687595e-06,2.649119409845118e-07,0.00253512163402557,-2.419662877381737e-07,4.3124190760040646e-07,3.1068903991780678e-09,0.000923913115050627,
      0.00040678296006807003,-7.755406890332818e-07,1.68539375883925e-06,6.676875566525437e-08,0.0008420574605423793,7.356584799406111e-05,0.0002306247719158348},
     1
  };

  AHIT inputhit = {
     {-20.7824649810791, -12.24150276184082, 57.8067626953125},
     {2.545517190810642e-06,-2.6680759219743777e-06,2.8030024168401724e-06,0.00014160551654640585,0.00012282167153898627,11.385087966918945}
  };
  printf("track in pos: %f, %f, %f \n", inputtrk.par[0], inputtrk.par[1], inputtrk.par[2]);
  printf("track in cov: %.2e, %.2e, %.2e \n", inputtrk.cov[SymOffsets66(PosInMtrx(0,0,6))],
                                              inputtrk.cov[SymOffsets66(PosInMtrx(1,1,6))],
                                              inputtrk.cov[SymOffsets66(PosInMtrx(2,2,6))]);
  printf("hit in pos: %f %f %f \n", inputhit.pos[0], inputhit.pos[1], inputhit.pos[2]);

  printf("produce nevts=%i ntrks=%i smearing by=%f \n", nevts, ntrks, smear);
  printf("NITER=%d\n", NITER);
 
  long setup_start, setup_stop;
  struct timeval timecheck;

  gettimeofday(&timecheck, NULL);
  setup_start = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;      
#ifdef FIXED_RSEED
  //[DEBUG by Seyong on Dec. 28, 2020] add an explicit srand(1) call to generate fixed inputs for better debugging.
  srand(1);
#endif
  hipDeviceSetCacheConfig(hipFuncCachePreferShared);
//  cudaFuncSetCacheConfig(GPUsequence,cudaFuncCachePreferL1);
//  cudaFuncSetCacheConfig(GPUsequenceR,cudaFuncCachePreferL1);
  MPTRK* trk = prepareTracks(inputtrk);
  MPHIT* hit = prepareHits(inputhit);
  //cudaHostRegister((void**)&trk,nevts*nb*sizeof(MPTRK),cudaHostRegisterDefault);
  //cudaHostRegister((void**)&hit,nlayer*nevts*nb*sizeof(MPHIT),cudaHostRegisterDefault);
  MPTRK* trk_dev;
  MPHIT* hit_dev;
  //MPTRK* outtrk= (MPTRK*) malloc(nevts*nb*sizeof(MPTRK)); 
  MPTRK* outtrk;
  hipHostMalloc((void**)&outtrk,nevts*nb*sizeof(MPTRK), hipHostMallocDefault); 
  MPTRK* outtrk_dev;
  hipMalloc((MPTRK**)&trk_dev,nevts*nb*sizeof(MPTRK));
  hipMalloc((MPHIT**)&hit_dev,nlayer*nevts*nb*sizeof(MPHIT));
  hipMalloc((MPTRK**)&outtrk_dev,nevts*nb*sizeof(MPTRK));
  dim3 grid(blockspergrid,1,1);
  dim3 block(threadsperblockx,threadsperblocky,1); 
  int device = -1;
  hipGetDevice(&device);
  int stream_chunk = ((int)(nevts/num_streams))*nb;//*sizeof(MPTRK);
  int stream_remainder = ((int)(nevts%num_streams))*nb;//*sizeof(MPTRK);
  int stream_range;
  if (stream_remainder == 0){ stream_range =num_streams;}
  else{stream_range = num_streams+1;}
  hipStream_t streams[stream_range];
  for (int s = 0; s<stream_range;s++){
    //cudaStreamCreateWithFlags(&streams[s],cudaStreamNonBlocking);
    hipStreamCreate(&streams[s]);
  }
#ifndef include_data
	memcpy_host2dev(trk_dev, trk, hit_dev, hit, streams, stream_chunk, stream_remainder);
#ifdef USE_ASYNC
	cudaDeviceSynchronize(); 
#endif
#endif

  gettimeofday(&timecheck, NULL);
  setup_stop = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;

  printf("done preparing!\n");

  printf("Size of struct MPTRK trk[] = %ld\n", nevts*nb*sizeof(struct MPTRK));
  printf("Size of struct MPTRK outtrk[] = %ld\n", nevts*nb*sizeof(struct MPTRK));
  printf("Size of struct struct MPHIT hit[] = %ld\n", nevts*nb*sizeof(struct MPHIT));
  
  auto wall_start = std::chrono::high_resolution_clock::now();

  for(itr=0; itr<NITER; itr++){
#ifdef include_data
	memcpy_host2dev(trk_dev, trk, hit_dev, hit, streams, stream_chunk, stream_remainder);
#endif

    for (int s = 0; s<num_streams;++s){
      //printf("stream = %d, grid (%d, %d, %d), block(%d, %d, %d), stream_chunk = %d\n",s, grid.x, grid.y, grid.z, block.x, block.y, block.z, stream_chunk);
#ifdef USE_ASYNC
  	  GPUsequence<<<grid,block,0,streams[s]>>>(trk_dev+(s*stream_chunk),hit_dev+(s*stream_chunk*nlayer),outtrk_dev+(s*stream_chunk),s);
#else
  	  GPUsequence<<<grid,block,0,0>>>(trk_dev+(s*stream_chunk),hit_dev+(s*stream_chunk*nlayer),outtrk_dev+(s*stream_chunk),s);
#endif
    }  
    if(stream_remainder != 0){
#ifdef USE_ASYNC
  	  GPUsequenceR<<<grid,block,0,streams[num_streams]>>>(trk_dev+(num_streams*stream_chunk),hit_dev+(num_streams*stream_chunk*nlayer),outtrk_dev+(num_streams*stream_chunk),num_streams);
#else
  	  GPUsequenceR<<<grid,block,0,0>>>(trk_dev+(num_streams*stream_chunk),hit_dev+(num_streams*stream_chunk*nlayer),outtrk_dev+(num_streams*stream_chunk),num_streams);
#endif
    }
#ifdef include_data
	memcpy_dev2host(outtrk, outtrk_dev, streams, stream_chunk, stream_remainder);
#endif
  } //end itr loop
  
#ifdef USE_ASYNC
	cudaDeviceSynchronize(); 
#endif
  auto wall_stop = std::chrono::high_resolution_clock::now();
#ifndef include_data
	memcpy_dev2host(outtrk, outtrk_dev, streams, stream_chunk, stream_remainder);
#ifdef USE_ASYNC
	cudaDeviceSynchronize(); 
#endif
#endif

  for (int s = 0; s<stream_range;s++){
    hipStreamDestroy(streams[s]);
  }
 
   auto wall_diff = wall_stop - wall_start;
   auto wall_time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(wall_diff).count()) / 1e6;
   printf("setup time time=%f (s)\n", (setup_stop-setup_start)*0.001);
   printf("done ntracks=%i tot time=%f (s) time/trk=%e (s)\n", nevts*ntrks*int(NITER), wall_time, wall_time/(nevts*ntrks*int(NITER)));
   printf("formatted %i %i %i %i %i %f 0 %f %i\n",int(NITER),nevts, ntrks, bsize, nb, wall_time, (setup_stop-setup_start)*0.001, num_streams);
#ifdef DUMP_OUTPUT
   FILE *fp_x;
   FILE *fp_y;
   FILE *fp_z;
   fp_x = fopen("output_x.txt", "w");
   fp_y = fopen("output_y.txt", "w");
   fp_z = fopen("output_z.txt", "w");
#endif



   double avgx = 0, avgy = 0, avgz = 0;
   double avgpt = 0, avgphi = 0, avgtheta = 0;
   double avgdx = 0, avgdy = 0, avgdz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float xl = x(outtrk,ie,it);
       float yl = y(outtrk,ie,it);
       float zl = z(outtrk,ie,it);
       float pt_ = 1./ipt(outtrk,ie,it);
       float phi_ = phi(outtrk,ie,it);
       float theta_ = theta(outtrk,ie,it);
#ifdef DUMP_OUTPUT
       fprintf(fp_x, "ie=%lu, it=%lu, %f\n", ie, it, xl);
       fprintf(fp_y, "%f\n", yl);
       fprintf(fp_z, "%f\n", zl);
#endif
       //if(xl ==0 || yl==0||zl==0){
       //printf("x: %f,y: %f,z: %f, ie: %d, it: %f\n",xl,yl,zl,ie,it);
       //continue;
       //}
       avgpt += pt_;
       avgphi += phi_;
       avgtheta += theta_;
       avgx += xl;
       avgy += yl;
       avgz += zl;
       float hxl = x(hit,ie,it);
       float hyl = y(hit,ie,it);
       float hzl = z(hit,ie,it);
       //if(xl ==0 || yl==0 || zl==0){continue;}
       avgdx += (xl-hxl)/xl;
       avgdy += (yl-hyl)/yl;
       avgdz += (zl-hzl)/zl;
     }
   }
#ifdef DUMP_OUTPUT
   fclose(fp_x);
   fclose(fp_y);
   fclose(fp_z);
   fp_x = fopen("input_x.txt", "w");
   fp_y = fopen("input_y.txt", "w");
   fp_z = fopen("input_z.txt", "w");
#endif
   avgpt = avgpt/double(nevts*ntrks);
   avgphi = avgphi/double(nevts*ntrks);
   avgtheta = avgtheta/double(nevts*ntrks);
   avgx = avgx/double(nevts*ntrks);
   avgy = avgy/double(nevts*ntrks);
   avgz = avgz/double(nevts*ntrks);
   avgdx = avgdx/double(nevts*ntrks);
   avgdy = avgdy/double(nevts*ntrks);
   avgdz = avgdz/double(nevts*ntrks);

   double stdx = 0, stdy = 0, stdz = 0;
   double stddx = 0, stddy = 0, stddz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float xl = x(outtrk,ie,it);
       float yl = y(outtrk,ie,it);
       float zl = z(outtrk,ie,it);
       stdx += (xl-avgx)*(xl-avgx);
       stdy += (yl-avgy)*(yl-avgy);
       stdz += (zl-avgz)*(zl-avgz);
       float hxl = x(hit,ie,it);
       float hyl = y(hit,ie,it);
       float hzl = z(hit,ie,it);
       stddx += ((xl-hxl)/xl-avgdx)*((xl-hxl)/xl-avgdx);
       stddy += ((yl-hyl)/yl-avgdy)*((yl-hyl)/yl-avgdy);
       stddz += ((zl-hzl)/zl-avgdz)*((zl-hzl)/zl-avgdz);
#ifdef DUMP_OUTPUT
       xl = x(trk,ie,it);
       yl = y(trk,ie,it);
       zl = z(trk,ie,it);
       fprintf(fp_x, "%f\n", xl);
       fprintf(fp_y, "%f\n", yl);
       fprintf(fp_z, "%f\n", zl);
#endif
     }
   }
#ifdef DUMP_OUTPUT
   fclose(fp_x);
   fclose(fp_y);
   fclose(fp_z);
#endif

   stdx = sqrtf(stdx/double(nevts*ntrks));
   stdy = sqrtf(stdy/double(nevts*ntrks));
   stdz = sqrtf(stdz/double(nevts*ntrks));
   stddx = sqrtf(stddx/double(nevts*ntrks));
   stddy = sqrtf(stddy/double(nevts*ntrks));
   stddz = sqrtf(stddz/double(nevts*ntrks));

   printf("track x avg=%f std/avg=%f\n", avgx, fabs(stdx/avgx));
   printf("track y avg=%f std/avg=%f\n", avgy, fabs(stdy/avgy));
   printf("track z avg=%f std/avg=%f\n", avgz, fabs(stdz/avgz));
   printf("track dx/x avg=%f std=%f\n", avgdx, stddx);
   printf("track dy/y avg=%f std=%f\n", avgdy, stddy);
   printf("track dz/z avg=%f std=%f\n", avgdz, stddz);
   printf("track pt avg=%f\n", avgpt);
   printf("track phi avg=%f\n", avgphi);
   printf("track theta avg=%f\n", avgtheta);
	
   hipHostFree(trk);
   hipHostFree(hit);
   hipHostFree(outtrk);
   //free(trk);
   //free(hit);
   //free(outtrk);
   hipFree(trk_dev);
   hipFree(hit_dev);
   hipFree(outtrk_dev);
   
return 0;
}

