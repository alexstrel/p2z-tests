#include "hip/hip_runtime.h"
/*
icc propagate-toz-test.C -o propagate-toz-test.exe -fopenmp -O3
*/
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <vector>
#include <Eigen/Dense>
#include <Eigen/Core>
#include <iostream>
#include <chrono>
#include <iomanip>

#define FIXED_RSEED

#ifndef bsize
#define bsize 1
#endif
#ifndef ntrks
#define ntrks 9600
#endif

#define nb    (ntrks/bsize)
#ifndef nevts
#define nevts 100
#endif
#define smear 0.1

#ifndef NITER
#define NITER 5
#endif
#ifndef nlayer
#define nlayer 20
#endif
#ifndef num_streams
#define num_streams 10
#endif

#ifndef threadsperblockx
#define threadsperblockx 32
#endif
#define threadsperblocky 512/threadsperblockx
#ifndef blockspergrid
#define blockspergrid 10
#endif

#define HOSTDEV __host__ __device__

using namespace Eigen;
//using Eigen::VectorXt;
//typedef Matrix<size_t, Dynamic, Dynamic> MatrixXt;
//typedef Matrix<size_t, Dynamic, 1> VectorXt;
//typedef Matrix<float, Dynamic, 1> VectorXf;


HOSTDEV size_t PosInMtrx(size_t i, size_t j, size_t D) {
  return i*D+j;
}

HOSTDEV size_t SymOffsets33(size_t i) {
  VectorXd offs(9);
  offs << 0, 1, 3, 1, 2, 4, 3, 4, 5;
  return offs(i);
}

HOSTDEV size_t SymOffsets66(size_t i) {
  VectorXf offs(36);
  offs << 0, 1, 3, 6, 10, 15, 1, 2, 4, 7, 11, 16, 3, 4, 5, 8, 12, 17, 6, 7, 8, 9, 13, 18, 10, 11, 12, 13, 14, 19, 15, 16, 17, 18, 19, 20;
  return offs(i);
}

struct ATRK {
  Matrix<float,6,1> par;
  Matrix<float,21,1> cov;
  int q;
//  Matrix<float,22,1> hitidx;
};

struct AHIT {
  Matrix<float,3,1> pos;
  Matrix<float,6,1> cov;
};

struct MP1I {
  Matrix<int,1,1> data[bsize];
};
struct MP22I {
  Matrix<int,22,1> data[bsize];
};

struct MP3F {
  Vector3f data[bsize];
};

struct MP6F {
  Matrix<float,6,1> data[bsize];
};

struct MP3x3 {
  Matrix<float,3,3> data[bsize];
};
struct MP6x3 {
  Matrix<float,6,3> data[bsize];
};

struct MP3x3SF {
  Matrix3f data[bsize];
};

struct MP6x6SF {
  Matrix<float,6,6> data[bsize];
};

struct MP6x6F {
  Matrix<float,6,6> data[bsize];
};

struct MPTRK {
  MP6F    par;
  MP6x6SF cov;
  MP1I    q;
//  MP22I   hitidx;
};                                                                                                                   

struct MPHIT {
  MP3F    pos;
  MP3x3SF cov;
};



float randn(float mu, float sigma) {
  float U1, U2, W, mult;
  static float X1, X2;
  static int call = 0;
  if (call == 1) {
    call = !call;
    return (mu + sigma * (float) X2);
  } do {
    U1 = -1 + ((float) rand () / RAND_MAX) * 2;
    U2 = -1 + ((float) rand () / RAND_MAX) * 2;
    W = pow (U1, 2) + pow (U2, 2);
  }
  while (W >= 1 || W == 0);
  mult = sqrt ((-2 * log (W)) / W);
  X1 = U1 * mult;
  X2 = U2 * mult;
  call = !call;
  return (mu + sigma * (float) X1);
}

MPTRK* prepareTracks(ATRK inputtrk) {
  //MPTRK* result = (MPTRK*) malloc(nevts*nb*sizeof(MPTRK));
  MPTRK* result;
  hipHostMalloc((void**)&result,nevts*nb*sizeof(MPTRK));
  for (size_t ie=0;ie<nevts;++ie) {
    for (size_t ib=0;ib<nb;++ib) {
      for (size_t it=0;it<bsize;++it) {
        //par
        for (size_t ip=0;ip<6;++ip) {
          result[ib + nb*ie].par.data[it](ip) = (1+smear*randn(0,1))*inputtrk.par[ip];
        }
        //cov
        for (size_t ip=0;ip<21;++ip) {
          result[ib + nb*ie].cov.data[it](ip) = (1+smear*randn(0,1))*inputtrk.cov[ip];
        }
        //q
        result[ib + nb*ie].q.data[it](0) = inputtrk.q-2*ceil(-0.5 + (float)rand() / RAND_MAX);//fixme check
      }
    }
  }
  return result;
}

MPHIT* prepareHits(AHIT inputhit) {
  //MPHIT* result = (MPHIT*) malloc(nlayer*nevts*nb*sizeof(MPHIT));
  MPHIT* result;
  hipHostMalloc((void**)&result,nlayer*nevts*nb*sizeof(MPHIT));
  for (int lay=0;lay<nlayer;++lay) {
    for (size_t ie=0;ie<nevts;++ie) {
      for (size_t ib=0;ib<nb;++ib) {
        for (size_t it=0;it<bsize;++it) {
          //pos
          for (size_t ip=0;ip<3;++ip) {
            result[lay+nlayer*(ib + nb*ie)].pos.data[it](ip) = (1+smear*randn(0,1))*inputhit.pos[ip];
          }
          //cov
          for (size_t ip=0;ip<6;++ip) {
            result[lay+nlayer*(ib + nb*ie)].cov.data[it](ip) = (1+smear*randn(0,1))*inputhit.cov[ip];
          }
        }
      }
    }
  }
  return result;
}


HOSTDEV MPTRK* bTk(MPTRK* tracks, size_t ev, size_t ib) {
  return &(tracks[ib + nb*ev]);
}

HOSTDEV const MPTRK* bTk(const MPTRK* tracks, size_t ev, size_t ib) {
  return &(tracks[ib + nb*ev]);
}


HOSTDEV float q(const MP1I* bq, size_t it){
  return (*bq).data[it](0);
}

HOSTDEV float par(const MP6F* bpars, size_t it, size_t ipar){
  return (*bpars).data[it](ipar);
}
HOSTDEV float x    (const MP6F* bpars, size_t it){ return par(bpars, it, 0); }
HOSTDEV float y    (const MP6F* bpars, size_t it){ return par(bpars, it, 1); }
HOSTDEV float z    (const MP6F* bpars, size_t it){ return par(bpars, it, 2); }
HOSTDEV float ipt  (const MP6F* bpars, size_t it){ return par(bpars, it, 3); }
HOSTDEV float phi  (const MP6F* bpars, size_t it){ return par(bpars, it, 4); }
HOSTDEV float theta(const MP6F* bpars, size_t it){ return par(bpars, it, 5); }

HOSTDEV float par(const MPTRK* btracks, size_t it, size_t ipar){
  return par(&(*btracks).par,it,ipar);
}
HOSTDEV float x    (const MPTRK* btracks, size_t it){ return par(btracks, it, 0); }
HOSTDEV float y    (const MPTRK* btracks, size_t it){ return par(btracks, it, 1); }
HOSTDEV float z    (const MPTRK* btracks, size_t it){ return par(btracks, it, 2); }
HOSTDEV float ipt  (const MPTRK* btracks, size_t it){ return par(btracks, it, 3); }
HOSTDEV float phi  (const MPTRK* btracks, size_t it){ return par(btracks, it, 4); }
HOSTDEV float theta(const MPTRK* btracks, size_t it){ return par(btracks, it, 5); }

HOSTDEV float par(const MPTRK* tracks, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  const MPTRK* btracks = bTk(tracks, ev, ib);
  size_t it = tk % bsize;
  return par(btracks, it, ipar);
}

HOSTDEV float x    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 0); }
HOSTDEV float y    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 1); }
HOSTDEV float z    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 2); }
HOSTDEV float ipt  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 3); }
HOSTDEV float phi  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 4); }
HOSTDEV float theta(const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 5); }

HOSTDEV void setpar(MP6F* bpars, size_t it, size_t ipar, float val){
  (*bpars).data[it](ipar) = val;
}
HOSTDEV void setx    (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 0, val); }
HOSTDEV void sety    (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 1, val); }
HOSTDEV void setz    (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 2, val); }
HOSTDEV void setipt  (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 3, val); }
HOSTDEV void setphi  (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 4, val); }
HOSTDEV void settheta(MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 5, val); }

HOSTDEV void setpar(MPTRK* btracks, size_t it, size_t ipar, float val){
  return setpar(&(*btracks).par,it,ipar,val);
}
HOSTDEV void setx    (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 0, val); }
HOSTDEV void sety    (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 1, val); }
HOSTDEV void setz    (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 2, val); }
HOSTDEV void setipt  (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 3, val); }
HOSTDEV void setphi  (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 4, val); }
HOSTDEV void settheta(MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 5, val); }

HOSTDEV MPHIT* bHit(MPHIT* hits, size_t ev, size_t ib, int lay) {
  return &(hits[lay+nlayer*(ib + nb*ev)]);
}
HOSTDEV const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib,int lay) {
  return &(hits[lay+nlayer*(ib + nb*ev)]);
}
HOSTDEV MPHIT* bHit(MPHIT* hits, size_t ev, size_t ib) {
  return &(hits[ib + nb*ev]);
}
HOSTDEV const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib) {
  return &(hits[ib + nb*ev]);
}

HOSTDEV float pos(const MP3F* hpos, size_t it, size_t ipar){
  return (*hpos).data[it](ipar);
}
HOSTDEV float x(const MP3F* hpos, size_t it)    { return pos(hpos, it, 0); }
HOSTDEV float y(const MP3F* hpos, size_t it)    { return pos(hpos, it, 1); }
HOSTDEV float z(const MP3F* hpos, size_t it)    { return pos(hpos, it, 2); }

HOSTDEV float pos(const MPHIT* hits, size_t it, size_t ipar){
  return pos(&(*hits).pos,it,ipar);
}
HOSTDEV float x(const MPHIT* hits, size_t it)    { return pos(hits, it, 0); }
HOSTDEV float y(const MPHIT* hits, size_t it)    { return pos(hits, it, 1); }
HOSTDEV float z(const MPHIT* hits, size_t it)    { return pos(hits, it, 2); }

HOSTDEV float pos(const MPHIT* hits, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  //[DEBUG by Seyong on Dec. 28, 2020] add 4th argument(nlayer-1) to bHit() below.
  const MPHIT* bhits = bHit(hits, ev, ib, nlayer-1);
  size_t it = tk % bsize;
  return pos(bhits,it,ipar);
}
HOSTDEV float x(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 0); }
HOSTDEV float y(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 1); }
HOSTDEV float z(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 2); }



#define N bsize
__forceinline__ __device__ void MultHelixPropEndcap(const MP6x6F* A, const MP6x6SF* B, MP6x6F* C) {
  const Matrix<float,6,6> *a = (*A).data; //ASSUME_ALIGNED(a, 64);
  const Matrix<float,6,6> *b = (*B).data; //ASSUME_ALIGNED(b, 64);
  Matrix<float,6,6> *c = (*C).data;       //ASSUME_ALIGNED(c, 64);
  for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x){
    c[it]/*.noalias()*/ = a[it]*b[it];
  }
}

__forceinline__ __device__ void MultHelixPropTranspEndcap(MP6x6F* A, MP6x6F* B, MP6x6SF* C) {
  const Matrix<float,6,6> *a = (*A).data; //ASSUME_ALIGNED(a, 64);
  const Matrix<float,6,6> *b = (*B).data; //ASSUME_ALIGNED(b, 64);
  Matrix<float,6,6> *c = (*C).data;       //ASSUME_ALIGNED(c, 64);
  for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x){
    c[it].noalias()= a[it]*b[it].transpose();
  }
}

__device__ __forceinline__ void KalmanGain(MP6x6SF* A, const MP3x3SF* B, MP6x3* C) {
  // k = P Ht(HPHt + R)^-1
  // HpHt -> cov of x,y,z. take upper 3x3 matrix of P
  // This calculates the inverse of HpHt +R
  const Matrix<float,6,6> *a = (*A).data; //ASSUME_ALIGNED(a, 64);
  const Matrix<float,3,3> *b = (*B).data; //ASSUME_ALIGNED(b, 64);
  Matrix<float,6,3> *c = (*C).data;       //ASSUME_ALIGNED(c, 64);
  Matrix<float,3,3> inter;
  for (int n = 0; n < N; ++n)
  {
    inter = ((a[n].block<3,3>(0,0)+b[n]).inverse()) ;
    c[n] = a[n].block<6,3>(0,0) * inter;
  }
}

__device__ __forceinline__ void KalmanUpdate(MP6x6SF* trkErr, MP6F* inPar, const MP3x3SF* hitErr, const MP3F* msP){
    MP6x3 kGain;
    //KalmanGain(trkErr,hitErr,&kGain);
//#pragma omp simd
  for (size_t it=0;it<bsize;++it) {
    kGain.data[it] = trkErr->data[it].block<6,3>(0,0) * ((trkErr->data[it].block<3,3>(0,0)+hitErr->data[it]).inverse()); // computes the full kalman gain without the funtion
    //if(isnan(kGain.data[it].sum())) {
    //Matrix<float,6,1> zeros;
    //zeros << 0,0,0,0,0,0;
    //inPar->data[it] = zeros;
    //continue;}
    //inPar->data[it] = test;//inPar->data[it] + (kGain.data[it]*(msP->data[it]- ((inPar->data[it]).block<3,1>(0,0))));
   // if(it==0){
   // std::cout<<(kGain.data[it]*(msP->data[it]- ((inPar->data[it]).block<3,1>(0,0))))<<"done"<<std::endl;
   // }
    inPar->data[it] = inPar->data[it] + (kGain.data[it]*(msP->data[it]- ((inPar->data[it]).block<3,1>(0,0))));
    trkErr->data[it] = trkErr->data[it] - (kGain.data[it]*((trkErr->data[it]).block<3,6>(0,0)));
  }

}

__device__ __constant__ float kfact = 100/3.8;
__device__ __forceinline__ void propagateToZ(const MP6x6SF* inErr, const MP6F* inPar,
			  const MP1I* inChg,const MP3F* msP, MP6x6SF* outErr, MP6F* outPar) {
  MP6x6F errorProp, temp;
  for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x){
    const float zout = z(msP,it);
    const float k = q(inChg,it)*kfact;//100/3.8;
    const float deltaZ = zout - z(inPar,it);
    const float pt = 1./ipt(inPar,it);
    const float cosP = cosf(phi(inPar,it));
    const float sinP = sinf(phi(inPar,it));
    const float cosT = cosf(theta(inPar,it));
    const float sinT = sinf(theta(inPar,it));
    const float pxin = cosP*pt;
    const float pyin = sinP*pt;
    const float icosT = 1.0/cosT;
    const float icosTk = icosT/k;
    const float alpha = deltaZ*sinT*ipt(inPar,it)*icosTk;
    //const float alpha = deltaZ*sinT*ipt(inPar,it)/(cosT*k);
    const float sina = sinf(alpha); // this can be approximated;
    const float cosa = cosf(alpha); // this can be approximated;
    outPar->data[it](0,0) = x(inPar,it) + k*(pxin*sina - pyin*(1.-cosa));
    outPar->data[it](1,0) = y(inPar,it) + k*(pyin*sina + pxin*(1.-cosa));
    outPar->data[it](2,0) = zout;
    outPar->data[it](3,0) = ipt(inPar,it);
    outPar->data[it](4,0) = phi(inPar,it)+alpha;
    outPar->data[it](5,0) = theta(inPar,it);
    
    const float sCosPsina = sinf(cosP*sina);
    const float cCosPsina = cosf(cosP*sina);
    
    for (size_t i=0;i<6;++i) errorProp.data[it](i,i) = 1.;
    errorProp.data[it](0,2) = cosP*sinT*(sinP*cosa*sCosPsina-cosa)*icosT;
    errorProp.data[it](0,3) = cosP*sinT*deltaZ*cosa*(1.-sinP*sCosPsina)*(icosT*pt)-k*(cosP*sina-sinP*(1.-cCosPsina))*(pt*pt);
    errorProp.data[it](0,4) = (k*pt)*(-sinP*sina+sinP*sinP*sina*sCosPsina-cosP*(1.-cCosPsina));
    errorProp.data[it](0,5) = cosP*deltaZ*cosa*(1.-sinP*sCosPsina)*(icosT*icosT);
    errorProp.data[it](1,2) = cosa*sinT*(cosP*cosP*sCosPsina-sinP)*icosT;
    errorProp.data[it](1,3) = sinT*deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*pt)-k*(sinP*sina+cosP*(1.-cCosPsina))*(pt*pt);
    errorProp.data[it](1,4) = (k*pt)*(-sinP*(1.-cCosPsina)-sinP*cosP*sina*sCosPsina+cosP*sina);
    errorProp.data[it](1,5) = deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*icosT);
    errorProp.data[it](4,2) = -ipt(inPar,it)*sinT*(icosTk);
    errorProp.data[it](4,3) = sinT*deltaZ*(icosTk);
    errorProp.data[it](4,5) = ipt(inPar,it)*deltaZ*icosT*icosTk;
    //errorProp.data[it](4,5) = ipt(inPar,it)*deltaZ*(icosT*icosTk);

   // errorProp.data[it](0,2) = cosP*sinT*(sinP*cosa*sCosPsina-cosa)/cosT;
   // errorProp.data[it](0,3) = cosP*sinT*deltaZ*cosa*(1.-sinP*sCosPsina)/(cosT*ipt(inPar,it))-k*(cosP*sina-sinP*(1.-cCosPsina))/(ipt(inPar,it)*ipt(inPar,it));
   // errorProp.data[it](0,4) = (k/ipt(inPar,it))*(-sinP*sina+sinP*sinP*sina*sCosPsina-cosP*(1.-cCosPsina));
   // errorProp.data[it](0,5) = cosP*deltaZ*cosa*(1.-sinP*sCosPsina)/(cosT*cosT);
   // errorProp.data[it](1,2) = cosa*sinT*(cosP*cosP*sCosPsina-sinP)/cosT;
   // errorProp.data[it](1,3) = sinT*deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)/(cosT*ipt(inPar,it))-k*(sinP*sina+cosP*(1.-cCosPsina))/(ipt(inPar,it)*ipt(inPar,it));
   // errorProp.data[it](1,4) = (k/ipt(inPar,it))*(-sinP*(1.-cCosPsina)-sinP*cosP*sina*sCosPsina+cosP*sina);
   // errorProp.data[it](1,5) = deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)/(cosT*cosT);
   // errorProp.data[it](4,2) = -ipt(inPar,it)*sinT/(cosT*k);
   // errorProp.data[it](4,3) = sinT*deltaZ/(cosT*k);
    //errorProp.data[it](4,5) = ipt(inPar,it)*deltaZ/(cosT*cosT*k);

  }
  __syncthreads();
  MultHelixPropEndcap(&errorProp, inErr, &temp);
  __syncthreads();
  MultHelixPropTranspEndcap(&errorProp, &temp, outErr);
}



__global__ void GPUsequence(MPTRK* trk, MPHIT* hit, MPTRK* outtrk, const int stream){
  int ie_range;
  if(stream == num_streams){ie_range = (int)(nevts%num_streams);}
  else{ie_range= (int)(nevts/num_streams);}
  for (size_t ie = blockIdx.x; ie<ie_range; ie+=gridDim.x){
    for(size_t ib = threadIdx.y; ib <nb; ib+=blockDim.y){
      const MPTRK* btracks = bTk(trk,ie,ib);
      MPTRK* obtracks = bTk(outtrk,ie,ib);
      for(int layer=0; layer<nlayer;++layer){
        const MPHIT* bhits = bHit(hit,ie,ib,layer);	
        propagateToZ(&(*btracks).cov, &(*btracks).par, &(*btracks).q, &(*bhits).pos, 
                     &(*obtracks).cov, &(*obtracks).par);
        __syncthreads();
        KalmanUpdate(&(*obtracks).cov,&(*obtracks).par,&(*bhits).cov,&(*bhits).pos);
      }
    }
  }
}


void transfer(MPTRK* trk, MPHIT* hit, MPTRK* trk_dev, MPHIT* hit_dev){

   
  hipMemcpy(trk_dev, trk, nevts*nb*sizeof(MPTRK), hipMemcpyHostToDevice);
  hipMemcpy(&trk_dev->par, &trk->par, sizeof(MP6F), hipMemcpyHostToDevice);
  hipMemcpy(&((trk_dev->par).data), &((trk->par).data), bsize*sizeof(Matrix<float,6,1>), hipMemcpyHostToDevice);
  hipMemcpy(&trk_dev->cov, &trk->cov, sizeof(MP6x6SF), hipMemcpyHostToDevice);
  hipMemcpy(&((trk_dev->cov).data), &((trk->cov).data), bsize*sizeof(Matrix<float,6,6>), hipMemcpyHostToDevice);
  hipMemcpy(&trk_dev->q, &trk->q, sizeof(MP1I), hipMemcpyHostToDevice);
  hipMemcpy(&((trk_dev->q).data), &((trk->q).data), bsize*sizeof(Matrix<int,1,1>), hipMemcpyHostToDevice);

  hipMemcpy(hit_dev,hit,nlayer*nevts*nb*sizeof(MPHIT), hipMemcpyHostToDevice);
  hipMemcpy(&hit_dev->pos,&hit->pos,sizeof(MP3F), hipMemcpyHostToDevice);
  hipMemcpy(&(hit_dev->pos).data,&(hit->pos).data,bsize*sizeof(Matrix<float,3,1>), hipMemcpyHostToDevice);
  hipMemcpy(&hit_dev->cov,&hit->cov,sizeof(MP3x3SF), hipMemcpyHostToDevice);
  hipMemcpy(&(hit_dev->cov).data,&(hit->cov).data,bsize*sizeof(Matrix<float,3,3>), hipMemcpyHostToDevice);
}
void transfer_back(MPTRK* trk, MPTRK* trk_host){
  hipMemcpy(trk_host, trk, nevts*nb*sizeof(MPTRK), hipMemcpyDeviceToHost);
  hipMemcpy(&trk_host->par, &trk->par, sizeof(MP6F), hipMemcpyDeviceToHost);
  hipMemcpy(&((trk_host->par).data), &((trk->par).data), bsize*sizeof(Matrix<float,6,1>), hipMemcpyDeviceToHost);
  hipMemcpy(&trk_host->cov, &trk->cov, sizeof(MP6x6SF), hipMemcpyDeviceToHost);
  hipMemcpy(&((trk_host->cov).data), &((trk->cov).data), bsize*sizeof(Matrix<float,6,6>), hipMemcpyDeviceToHost);
  hipMemcpy(&trk_host->q, &trk->q, sizeof(MP1I), hipMemcpyDeviceToHost);
  hipMemcpy(&((trk_host->q).data), &((trk->q).data), bsize*sizeof(Matrix<int,1,1>), hipMemcpyDeviceToHost);
}



int main (int argc, char* argv[]) {

  printf("RUNNING CUDA!!\n");
  ATRK inputtrk = {
     {-12.806846618652344, -7.723824977874756, 38.13014221191406,0.23732035065189902, -2.613372802734375, 0.35594117641448975},
     {6.290299552347278e-07,4.1375109560704004e-08,7.526661534029699e-07,2.0973730840978533e-07,1.5431574240665213e-07,9.626245400795597e-08,-2.804026640189443e-06,
      6.219111130687595e-06,2.649119409845118e-07,0.00253512163402557,-2.419662877381737e-07,4.3124190760040646e-07,3.1068903991780678e-09,0.000923913115050627,
      0.00040678296006807003,-7.755406890332818e-07,1.68539375883925e-06,6.676875566525437e-08,0.0008420574605423793,7.356584799406111e-05,0.0002306247719158348},
     1
  };

  AHIT inputhit = {
     {-20.7824649810791, -12.24150276184082, 57.8067626953125},
     {2.545517190810642e-06,-2.6680759219743777e-06,2.8030024168401724e-06,0.00014160551654640585,0.00012282167153898627,11.385087966918945}
  };
  printf("track in pos: %f, %f, %f \n", inputtrk.par[0], inputtrk.par[1], inputtrk.par[2]);
  printf("track in cov: %.2e, %.2e, %.2e \n", inputtrk.cov[SymOffsets66(PosInMtrx(0,0,6))],
                                              inputtrk.cov[SymOffsets66(PosInMtrx(1,1,6))],
                                              inputtrk.cov[SymOffsets66(PosInMtrx(2,2,6))]);
  printf("hit in pos: %f %f %f \n", inputhit.pos[0], inputhit.pos[1], inputhit.pos[2]);

  printf("produce nevts=%i ntrks=%i smearing by=%f \n", nevts, ntrks, smear);
  printf("NITER=%d\n", NITER);
 
  long setup_start, setup_stop;
  struct timeval timecheck;
 
  gettimeofday(&timecheck, NULL);
  setup_start = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
#ifdef FIXED_RSEED
   //[DEBUG by Seyong on Dec. 28, 2020] add an explicit srand(1) call to generate fixed inputs for better debugging.
   srand(1);
#endif
  MPTRK* trk = prepareTracks(inputtrk);
  MPHIT* hit = prepareHits(inputhit);
  MPTRK* trk_dev;
  MPHIT* hit_dev;
  MPTRK* outtrk = (MPTRK*) malloc(nevts*nb*sizeof(MPTRK));
  MPTRK* outtrk_dev;
  hipMalloc((MPTRK**)&trk_dev,nevts*nb*sizeof(MPTRK));  
  hipMalloc((MPTRK**)&hit_dev,nlayer*nevts*nb*sizeof(MPHIT));
  hipMalloc((MPTRK**)&outtrk_dev,nevts*nb*sizeof(MPTRK));  

  dim3 grid(blockspergrid,1,1);
  dim3 block(threadsperblockx,threadsperblocky,1); 
  int device = -1;
  hipGetDevice(&device);
  int stream_chunk = ((int)(nevts/num_streams))*nb;//*sizeof(MPTRK);
  int stream_remainder = ((int)(nevts%num_streams))*nb;//*sizeof(MPTRK);
  int stream_range;
  if (stream_remainder == 0){ stream_range =num_streams;}
  else{stream_range = num_streams+1;}
  hipStream_t streams[stream_range];
  for (int s = 0; s<stream_range;s++){
    hipStreamCreateWithFlags(&streams[s],hipStreamNonBlocking);
  }

  gettimeofday(&timecheck, NULL);
  setup_stop = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
 

  printf("done preparing!\n");
  printf("dev: %f\n",trk->par.data[0](0));
  
  printf("Size of struct MPTRK trk[] = %ld\n", nevts*nb*sizeof(struct MPTRK));
  printf("Size of struct MPTRK outtrk[] = %ld\n", nevts*nb*sizeof(struct MPTRK));
  printf("Size of struct struct MPHIT hit[] = %ld\n", nlayer*nevts*nb*sizeof(struct MPHIT));

  auto wall_start = std::chrono::high_resolution_clock::now();

  for(int itr=0; itr<NITER; itr++){
    for (int s = 0; s<num_streams;s++){
      hipMemcpyAsync(trk_dev+(s*stream_chunk), trk+(s*stream_chunk), stream_chunk*sizeof(MPTRK), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(trk_dev+(s*stream_chunk))->par, &(trk+(s*stream_chunk))->par, sizeof(MP6F), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(((trk_dev+(s*stream_chunk))->par).data), &(((trk+(s*stream_chunk))->par).data), 6*bsize*sizeof(float), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(trk_dev+(s*stream_chunk))->cov, &(trk+(s*stream_chunk))->cov, sizeof(MP6x6SF), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(((trk_dev+(s*stream_chunk))->cov).data), &(((trk+(s*stream_chunk))->cov).data), 36*bsize*sizeof(float), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(trk_dev+(s*stream_chunk))->q, &(trk+(s*stream_chunk))->q, sizeof(MP1I), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(((trk_dev+(s*stream_chunk))->q).data), &(((trk+(s*stream_chunk))->q).data), 1*bsize*sizeof(int), hipMemcpyHostToDevice, streams[s]);
      
      hipMemcpyAsync(hit_dev+(s*stream_chunk*nlayer),hit+(s*stream_chunk),nlayer*stream_chunk*sizeof(MPHIT), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(hit_dev+(s*stream_chunk*nlayer))->pos,&(hit+(s*stream_chunk*nlayer))->pos,sizeof(MP3F), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&((hit_dev+(s*stream_chunk*nlayer))->pos).data,&((hit+(s*stream_chunk*nlayer))->pos).data,3*bsize*sizeof(float), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(hit_dev+(s*stream_chunk*nlayer))->cov,&(hit+(s*stream_chunk*nlayer))->cov,sizeof(MP3x3SF), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&((hit_dev+(s*stream_chunk*nlayer))->cov).data,&((hit+(s*stream_chunk*nlayer))->cov).data,6*bsize*sizeof(float), hipMemcpyHostToDevice, streams[s]);
    }
    if(stream_remainder != 0){
      hipMemcpyAsync(trk_dev+(num_streams*stream_chunk), trk+(num_streams*stream_chunk), stream_remainder*sizeof(MPTRK), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(trk_dev+(num_streams*stream_chunk))->par, &(trk+(num_streams*stream_chunk))->par, sizeof(MP6F), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(((trk_dev+(num_streams*stream_chunk))->par).data), &(((trk+(num_streams*stream_chunk))->par).data), 6*bsize*sizeof(float), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(trk_dev+(num_streams*stream_chunk))->cov, &(trk+(num_streams*stream_chunk))->cov, sizeof(MP6x6SF), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(((trk_dev+(num_streams*stream_chunk))->cov).data), &(((trk+(num_streams*stream_chunk))->cov).data), 36*bsize*sizeof(float), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(trk_dev+(num_streams*stream_chunk))->q, &(trk+(num_streams*stream_chunk))->q, sizeof(MP1I), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(((trk_dev+(num_streams*stream_chunk))->q).data), &(((trk+(num_streams*stream_chunk))->q).data), 1*bsize*sizeof(int), hipMemcpyHostToDevice, streams[num_streams]);
      
      hipMemcpyAsync(hit_dev+(num_streams*stream_chunk*nlayer),hit+(num_streams*stream_chunk*nlayer),nlayer*stream_remainder*sizeof(MPHIT), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(hit_dev+(num_streams*stream_chunk*nlayer))->pos,&(hit+(num_streams*stream_chunk*nlayer))->pos,sizeof(MP3F), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&((hit_dev+(num_streams*stream_chunk*nlayer))->pos).data,&((hit+(num_streams*stream_chunk*nlayer))->pos).data,3*bsize*sizeof(float), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(hit_dev+(num_streams*stream_chunk*nlayer))->cov,&(hit+(num_streams*stream_chunk*nlayer))->cov,sizeof(MP3x3SF), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&((hit_dev+(num_streams*stream_chunk*nlayer))->cov).data,&((hit+(num_streams*stream_chunk*nlayer))->cov).data,6*bsize*sizeof(float), hipMemcpyHostToDevice, streams[num_streams]);
    }

    for (int s = 0; s<num_streams;s++){
      GPUsequence<<<grid,block,0,streams[s]>>>(trk_dev+(s*stream_chunk),hit_dev+(s*stream_chunk*nlayer),outtrk_dev+(s*stream_chunk),s);
    }  
    if(stream_remainder != 0){
      GPUsequence<<<grid,block,0,streams[num_streams]>>>(trk_dev+(num_streams*stream_chunk),hit_dev+(num_streams*stream_chunk*nlayer),outtrk_dev+(num_streams*stream_chunk),num_streams); 
    }
  
    for (int s = 0; s<num_streams;s++){
      hipMemcpyAsync(outtrk+(s*stream_chunk), outtrk_dev+(s*stream_chunk), stream_chunk*sizeof(MPTRK), hipMemcpyDeviceToHost, streams[s]);
      hipMemcpyAsync(&(outtrk+(s*stream_chunk))->par, &(outtrk_dev+(s*stream_chunk))->par, sizeof(MP6F), hipMemcpyDeviceToHost, streams[s]);
      hipMemcpyAsync(&(((outtrk+(s*stream_chunk))->par).data), &(((outtrk_dev+(s*stream_chunk))->par).data), 6*bsize*sizeof(float), hipMemcpyDeviceToHost, streams[s]);
      hipMemcpyAsync(&(outtrk+(s*stream_chunk))->cov, &(outtrk_dev+(s*stream_chunk))->cov, sizeof(MP6x6SF), hipMemcpyDeviceToHost, streams[s]);
      hipMemcpyAsync(&(((outtrk+(s*stream_chunk))->cov).data), &(((outtrk_dev+(s*stream_chunk))->cov).data), 36*bsize*sizeof(float), hipMemcpyDeviceToHost, streams[s]);
      hipMemcpyAsync(&(outtrk+(s*stream_chunk))->q, &(outtrk_dev+(s*stream_chunk))->q, sizeof(MP1I), hipMemcpyDeviceToHost, streams[s]);
      hipMemcpyAsync(&(((outtrk+(s*stream_chunk))->q).data), &(((outtrk_dev+(s*stream_chunk))->q).data), 1*bsize*sizeof(int), hipMemcpyDeviceToHost, streams[s]);
    }
    if(stream_remainder != 0){
      hipMemcpyAsync(outtrk+(num_streams*stream_chunk), outtrk_dev+(num_streams*stream_chunk), stream_remainder*sizeof(MPTRK), hipMemcpyDeviceToHost, streams[num_streams]);
      hipMemcpyAsync(&(outtrk+(num_streams*stream_chunk))->par, &(outtrk_dev+(num_streams*stream_chunk))->par, sizeof(MP6F), hipMemcpyDeviceToHost, streams[num_streams]);
      hipMemcpyAsync(&(((outtrk+(num_streams*stream_chunk))->par).data), &(((outtrk_dev+(num_streams*stream_chunk))->par).data), 6*bsize*sizeof(float), hipMemcpyDeviceToHost, streams[num_streams]);
      hipMemcpyAsync(&(outtrk+(num_streams*stream_chunk))->cov, &(outtrk_dev+(num_streams*stream_chunk))->cov, sizeof(MP6x6SF), hipMemcpyDeviceToHost, streams[num_streams]);
      hipMemcpyAsync(&(((outtrk+(num_streams*stream_chunk))->cov).data), &(((outtrk_dev+(num_streams*stream_chunk))->cov).data), 36*bsize*sizeof(float), hipMemcpyDeviceToHost, streams[num_streams]);
      hipMemcpyAsync(&(outtrk+(num_streams*stream_chunk))->q, &(outtrk_dev+(num_streams*stream_chunk))->q, sizeof(MP1I), hipMemcpyDeviceToHost, streams[num_streams]);
      hipMemcpyAsync(&(((outtrk+(num_streams*stream_chunk))->q).data), &(((outtrk_dev+(num_streams*stream_chunk))->q).data), 1*bsize*sizeof(int), hipMemcpyDeviceToHost, streams[num_streams]);
    }
  } //end itr loop

  hipDeviceSynchronize(); // shaves a few seconds
  auto wall_stop = std::chrono::high_resolution_clock::now();

  
  for (int s = 0; s<stream_range;s++){
    hipStreamDestroy(streams[s]);
  }
 
   auto wall_diff = wall_stop - wall_start;
   auto wall_time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(wall_diff).count()) / 1e6;
   printf("setup time time=%f (s)\n", (setup_stop-setup_start)*0.001);
   printf("done ntracks=%i tot time=%f (s) time/trk=%e (s)\n", nevts*ntrks*int(NITER), wall_time, wall_time/(nevts*ntrks*int(NITER)));
   printf("formatted %i %i %i %i %i %f 0 %f %i\n",int(NITER),nevts, ntrks, bsize, nb, wall_time, (setup_stop-setup_start)*0.001, num_streams);

   int bad_count =0;
   float avgx = 0, avgy = 0, avgz = 0;
   float avgpt = 0, avgphi = 0, avgtheta = 0;
   float avgdx = 0, avgdy = 0, avgdz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       float pt_ = 1./ipt(outtrk,ie,it);
       float phi_ = phi(outtrk,ie,it);
       float theta_ = theta(outtrk,ie,it);
       if( isnan(x_) && isnan(y_) && isnan(z_)){bad_count++;continue;} // counts and skips over bad values
       avgpt += pt_;
       avgphi += phi_;
       avgtheta += theta_;
       avgx += x_;
       avgy += y_;
       avgz += z_;
       float hx_ = x(hit,ie,it);
       float hy_ = y(hit,ie,it);
       float hz_ = z(hit,ie,it);
       avgdx += (x_-hx_)/x_;
       avgdy += (y_-hy_)/y_;
       avgdz += (z_-hz_)/z_;
     }
   }
   avgpt = avgpt/float(nevts*ntrks);
   avgphi = avgphi/float(nevts*ntrks);
   avgtheta = avgtheta/float(nevts*ntrks);
   avgx = avgx/float(nevts*ntrks);
   avgy = avgy/float(nevts*ntrks);
   avgz = avgz/float(nevts*ntrks);
   avgdx = avgdx/float(nevts*ntrks);
   avgdy = avgdy/float(nevts*ntrks);
   avgdz = avgdz/float(nevts*ntrks);

   float stdx = 0, stdy = 0, stdz = 0;
   float stddx = 0, stddy = 0, stddz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       if( isnan(x_) && isnan(y_) && isnan(z_)){continue;}
       stdx += (x_-avgx)*(x_-avgx);
       stdy += (y_-avgy)*(y_-avgy);
       stdz += (z_-avgz)*(z_-avgz);
       float hx_ = x(hit,ie,it);
       float hy_ = y(hit,ie,it);
       float hz_ = z(hit,ie,it);
       stddx += ((x_-hx_)/x_-avgdx)*((x_-hx_)/x_-avgdx);
       stddy += ((y_-hy_)/y_-avgdy)*((y_-hy_)/y_-avgdy);
       stddz += ((z_-hz_)/z_-avgdz)*((z_-hz_)/z_-avgdz);
     }
   }

   stdx = sqrtf(stdx/float(nevts*ntrks));
   stdy = sqrtf(stdy/float(nevts*ntrks));
   stdz = sqrtf(stdz/float(nevts*ntrks));
   stddx = sqrtf(stddx/float(nevts*ntrks));
   stddy = sqrtf(stddy/float(nevts*ntrks));
   stddz = sqrtf(stddz/float(nevts*ntrks));

   printf("track x avg=%f std/avg=%f\n", avgx, fabs(stdx/avgx));
   printf("track y avg=%f std/avg=%f\n", avgy, fabs(stdy/avgy));
   printf("track z avg=%f std/avg=%f\n", avgz, fabs(stdz/avgz));
   printf("track dx/x avg=%f std=%f\n", avgdx, stddx);
   printf("track dy/y avg=%f std=%f\n", avgdy, stddy);
   printf("track dz/z avg=%f std=%f\n", avgdz, stddz);
   printf("track pt avg=%f\n", avgpt);
   printf("track phi avg=%f\n", avgphi);
   printf("track theta avg=%f\n", avgtheta);
   printf("bad track evaluation=%d/%d (%f%%)\n", bad_count,nevts*ntrks,100*(float)bad_count/(nevts*ntrks));
	
   hipFree(trk);
   hipFree(hit);
   hipFree(outtrk);
   
return 0;
}

