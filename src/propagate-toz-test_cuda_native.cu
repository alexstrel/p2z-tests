#include "hip/hip_runtime.h"
/*
nvc++ -O2 -std=c++17 -stdpar=gpu -gpu=cc75 -gpu=managed -gpu=fma -gpu=fastmath -gpu=autocollapse -gpu=loadcache:L1 -gpu=unroll  src/propagate-tor-test_pstl.cpp   -o ./propagate_nvcpp_pstl
nvc++ -O2 -std=c++17 -stdpar=multicore src/propagate-tor-test_pstl.cpp   -o ./propagate_nvcpp_pstl 
g++ -O3 -I. -fopenmp -mavx512f -std=c++17 src/propagate-tor-test_pstl.cpp -lm -lgomp -Lpath-to-tbb-lib -ltbb  -o ./propagate_gcc_pstl
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <iostream>
#include <chrono>
#include <iomanip>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <cassert>

#include <algorithm>
#include <vector>
#include <memory>
#include <numeric>
#include <random>


#ifndef ntrks
#define ntrks 9600
#endif

//#define ntrks    (ntrks/bsize)

#ifndef nevts
#define nevts 100
#endif
#define smear 0.00001

#ifndef NITER
#define NITER 5
#endif
#ifndef nlayer
#define nlayer 20
#endif

#ifndef num_streams
#define num_streams 1
#endif

#ifndef threadsperblock
#define threadsperblock 32
#endif

#ifdef include_data
constexpr bool include_data_transfer = true;
#else
constexpr bool include_data_transfer = false;
#endif

static int nstreams  = num_streams;//we have only one stream, though

constexpr int host_id = -1; /*hipCpuDeviceId*/

namespace impl {

  /**
     Simple array object which mimics std::array
  */
  template <typename T, int n> struct array {
    using value_type = T;
    T data[n];

    constexpr T &operator[](int i) { return data[i]; }
    constexpr const T &operator[](int i) const { return data[i]; }
    constexpr int size() const { return n; }

    array() = default;
    array(const array<T, n> &) = default;
    array(array<T, n> &&) = default;

    array<T, n> &operator=(const array<T, n> &) = default;
    array<T, n> &operator=(array<T, n> &&) = default;
  };
  
  template<typename Tp>
  struct UVMAllocator {
    public:

      typedef Tp value_type;

      UVMAllocator () {};

      UVMAllocator(const UVMAllocator&) { }
       
      template<typename Tp1> constexpr UVMAllocator(const UVMAllocator<Tp1>&) { }

      ~UVMAllocator() { }

      Tp* address(Tp& x) const { return &x; }

      std::size_t  max_size() const throw() { return size_t(-1) / sizeof(Tp); }

      [[nodiscard]] Tp* allocate(std::size_t n){

        Tp* ptr = nullptr;

        auto err = hipMallocManaged((void **)&ptr,n*sizeof(Tp));

        if( err != hipSuccess ) {
          ptr = (Tp *) NULL;
          std::cerr << " hipMallocManaged failed for " << n*sizeof(Tp) << " bytes " <<hipGetErrorString(err)<< std::endl;
          assert(0);
        }

        return ptr;
      }
      void deallocate( Tp* p, std::size_t n) noexcept {
        hipFree((void *)p);
        return;
      }
    };
    
   template <typename IntType>
   class counting_iterator {
       static_assert(std::numeric_limits<IntType>::is_integer, "Cannot instantiate counting_iterator with a non-integer type");
     public:
       using value_type = IntType;
       using difference_type = typename std::make_signed<IntType>::type;
       using pointer = IntType*;
       using reference = IntType&;
       using iterator_category = std::random_access_iterator_tag;

       counting_iterator() : value(0) { }
       explicit counting_iterator(IntType v) : value(v) { }

       value_type operator*() const { return value; }
       value_type operator[](difference_type n) const { return value + n; }

       counting_iterator& operator++() { ++value; return *this; }
       counting_iterator operator++(int) {
         counting_iterator result{value};
         ++value;
         return result;
       }  
       counting_iterator& operator--() { --value; return *this; }
       counting_iterator operator--(int) {
         counting_iterator result{value};
         --value;
         return result;
       }
       counting_iterator& operator+=(difference_type n) { value += n; return *this; }
       counting_iterator& operator-=(difference_type n) { value -= n; return *this; }

       friend counting_iterator operator+(counting_iterator const& i, difference_type n)          { return counting_iterator(i.value + n);  }
       friend counting_iterator operator+(difference_type n, counting_iterator const& i)          { return counting_iterator(i.value + n);  }
       friend difference_type   operator-(counting_iterator const& x, counting_iterator const& y) { return x.value - y.value;  }
       friend counting_iterator operator-(counting_iterator const& i, difference_type n)          { return counting_iterator(i.value - n);  }

       friend bool operator==(counting_iterator const& x, counting_iterator const& y) { return x.value == y.value;  }
       friend bool operator!=(counting_iterator const& x, counting_iterator const& y) { return x.value != y.value;  }
       friend bool operator<(counting_iterator const& x, counting_iterator const& y)  { return x.value < y.value; }
       friend bool operator<=(counting_iterator const& x, counting_iterator const& y) { return x.value <= y.value; }
       friend bool operator>(counting_iterator const& x, counting_iterator const& y)  { return x.value > y.value; }
       friend bool operator>=(counting_iterator const& x, counting_iterator const& y) { return x.value >= y.value; }

     private:
       IntType value;
   };

} //impl

//Collection of API functions:
int p2z_get_compute_device_id(){
  int dev = -1;
  hipGetDevice(&dev);
  hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  return dev;
}

void p2z_check_error(){
  //
  auto error = hipGetLastError();
  if(error != hipSuccess) std::cout << "Error detected, error " << error << std::endl;
  //
  return;
}

decltype(auto) p2z_get_streams(const int n){
  std::vector<hipStream_t> streams;
  streams.reserve(n);
  for (int i = 0; i < n; i++) {
    hipStream_t stream;
    hipStreamCreate(&stream);
    streams.push_back(stream);
  }
  return streams;
}

template <typename data_tp, typename Allocator, typename stream_t, bool is_sync = false>
void p2z_prefetch(std::vector<data_tp, Allocator> &v, int devId, stream_t stream) {
  hipMemPrefetchAsync(v.data(), v.size() * sizeof(data_tp), devId, stream);
  //
  if constexpr (is_sync) {hipStreamSynchronize(stream);}

  return;
}

void p2z_wait() {
  hipDeviceSynchronize();
  return;
}

const std::array<int, 36> SymOffsets66{0, 1, 3, 6, 10, 15, 1, 2, 4, 7, 11, 16, 3, 4, 5, 8, 12, 17, 6, 7, 8, 9, 13, 18, 10, 11, 12, 13, 14, 19, 15, 16, 17, 18, 19, 20};

struct ATRK {
  std::array<float,6> par;
  std::array<float,21> cov;
  int q;
};

struct AHIT {
  std::array<float,3> pos;
  std::array<float,6> cov;
};

constexpr int iparX     = 0;
constexpr int iparY     = 1;
constexpr int iparZ     = 2;
constexpr int iparIpt   = 3;
constexpr int iparPhi   = 4;
constexpr int iparTheta = 5;

template <typename T, int N>
struct MPNX {
   impl::array<T,N> data;
   //basic accessors
   __device__ __host__ inline const T& operator[](const int idx) const {return data[idx];}
   __device__ __host__ inline T& operator[](const int idx) {return data[idx];}

   __device__ __host__ void copy(const MPNX& src) {
#pragma unroll
     for (size_t ip=0;ip<N;++ip){
       this->data[ip] = src.data[ip];
     }
     return;
   }

};

using MP1I    = MPNX<int,   1 >;
using MP1F    = MPNX<float, 1 >;
using MP2F    = MPNX<float, 3 >;
using MP3F    = MPNX<float, 3 >;
using MP6F    = MPNX<float, 6 >;
using MP2x2SF = MPNX<float, 3 >;
using MP3x3SF = MPNX<float, 6 >;
using MP6x6SF = MPNX<float, 21>;
using MP6x6F  = MPNX<float, 36>;
using MP3x3   = MPNX<float, 9 >;
using MP3x6   = MPNX<float, 18>;

struct MPTRK {
  MP6F    par;
  MP6x6SF cov;
  MP1I    q;

  //  MP22I   hitidx;
  __device__ __host__ MPTRK& operator=(const MPTRK &src){
    par.copy(src.par);
    cov.copy(src.cov);
    q.copy(src.q);
    return *this;
  }

};

struct MPHIT {
  MP3F    pos;
  MP3x3SF cov;
  //
  __device__ __host__  MPHIT& operator=(const MPHIT &src){
    pos.copy(src.pos);
    cov.copy(src.cov);
    return *this;
  }
};

using MPTRKAllocator = impl::UVMAllocator<MPTRK>;
using MPHITAllocator = impl::UVMAllocator<MPHIT>;

///////////////////////////////////////
//Gen. utils

float randn(float mu, float sigma) {
  float U1, U2, W, mult;
  static float X1, X2;
  static int call = 0;
  if (call == 1) {
    call = !call;
    return (mu + sigma * (float) X2);
  } do {
    U1 = -1 + ((float) rand () / RAND_MAX) * 2;
    U2 = -1 + ((float) rand () / RAND_MAX) * 2;
    W = pow (U1, 2) + pow (U2, 2);
  }
  while (W >= 1 || W == 0); 
  mult = sqrt ((-2 * log (W)) / W);
  X1 = U1 * mult;
  X2 = U2 * mult; 
  call = !call; 
  return (mu + sigma * (float) X1);
}


template<typename MPTRKAllocator>
void prepareTracks(std::vector<MPTRK, MPTRKAllocator> &trcks, ATRK &inputtrk) {
  //
  for (int ie=0;ie<nevts;++ie) {
    for (int ib=0;ib<ntrks;++ib) {
      {
	      //par
	      for (int ip=0;ip<6;++ip) {
	        trcks[ib + ntrks*ie].par.data[ip] = (1+smear*randn(0,1))*inputtrk.par[ip];
	      }
	      //cov, scale by factor 100
	      for (int ip=0;ip<21;++ip) {
	        trcks[ib + ntrks*ie].cov.data[ip] = (1+smear*randn(0,1))*inputtrk.cov[ip]*100;
	      }
	      //q
	      trcks[ib + ntrks*ie].q.data[0] = inputtrk.q;//can't really smear this or fit will be wrong
      }
    }
  }
  //
  return;
}

template<typename MPHITAllocator>
void prepareHits(std::vector<MPHIT, MPHITAllocator> &hits, std::vector<AHIT>& inputhits) {
  // store in element order for bunches of bsize matrices (a la matriplex)
  for (int lay=0;lay<nlayer;++lay) {

    int mylay = lay;
    if (lay>=inputhits.size()) {
      // int wraplay = inputhits.size()/lay;
      exit(1);
    }
    AHIT& inputhit = inputhits[mylay];

    for (int ie=0;ie<nevts;++ie) {
      for (int ib=0;ib<ntrks;++ib) {
        {
        	//pos
        	for (int ip=0;ip<3;++ip) {
        	  hits[lay+nlayer*(ib + ntrks*ie)].pos.data[ip] = (1+smear*randn(0,1))*inputhit.pos[ip];
        	}
        	//cov
        	for (int ip=0;ip<6;++ip) {
        	  hits[lay+nlayer*(ib + ntrks*ie)].cov.data[ip] = (1+smear*randn(0,1))*inputhit.cov[ip];
        	}
        }
      }
    }
  }
  return;
}


//////////////////////////////////////////////////////////////////////////////////////
// Aux utils 
MPTRK* bTk(MPTRK* tracks, int ev, int ib) {
  return &(tracks[ib + ntrks*ev]);
}

const MPTRK* bTk(const MPTRK* tracks, int ev, int ib) {
  return &(tracks[ib + ntrks*ev]);
}

float q(const MP1I* bq, int it){
  return (*bq).data[0];
}
//
float par(const MP6F* bpars, int it, int ipar){
  return (*bpars).data[it + ipar];
}
float x    (const MP6F* bpars, int it){ return par(bpars, it, 0); }
float y    (const MP6F* bpars, int it){ return par(bpars, it, 1); }
float z    (const MP6F* bpars, int it){ return par(bpars, it, 2); }
float ipt  (const MP6F* bpars, int it){ return par(bpars, it, 3); }
float phi  (const MP6F* bpars, int it){ return par(bpars, it, 4); }
float theta(const MP6F* bpars, int it){ return par(bpars, it, 5); }
//
float par(const MPTRK* btracks, int it, int ipar){
  return par(&(*btracks).par,it,ipar);
}
float x    (const MPTRK* btracks, int it){ return par(btracks, it, 0); }
float y    (const MPTRK* btracks, int it){ return par(btracks, it, 1); }
float z    (const MPTRK* btracks, int it){ return par(btracks, it, 2); }
float ipt  (const MPTRK* btracks, int it){ return par(btracks, it, 3); }
float phi  (const MPTRK* btracks, int it){ return par(btracks, it, 4); }
float theta(const MPTRK* btracks, int it){ return par(btracks, it, 5); }
//
float par(const MPTRK* tracks, int ev, int tk, int ipar){
  int ib = tk;
  const MPTRK* btracks = bTk(tracks, ev, ib);
  int it = 0;
  return par(btracks, it, ipar);
}
float x    (const MPTRK* tracks, int ev, int tk){ return par(tracks, ev, tk, 0); }
float y    (const MPTRK* tracks, int ev, int tk){ return par(tracks, ev, tk, 1); }
float z    (const MPTRK* tracks, int ev, int tk){ return par(tracks, ev, tk, 2); }
float ipt  (const MPTRK* tracks, int ev, int tk){ return par(tracks, ev, tk, 3); }
float phi  (const MPTRK* tracks, int ev, int tk){ return par(tracks, ev, tk, 4); }
float theta(const MPTRK* tracks, int ev, int tk){ return par(tracks, ev, tk, 5); }
//

const MPHIT* bHit(const MPHIT* hits, int ev, int ib) {
  return &(hits[ib + ntrks*ev]);
}
const MPHIT* bHit(const MPHIT* hits, int ev, int ib,int lay) {
return &(hits[lay + (ib*nlayer) +(ev*nlayer*ntrks)]);
}
//
float Pos(const MP3F* hpos, int it, int ipar){
  return (*hpos).data[it + ipar];
}
float x(const MP3F* hpos, int it)    { return Pos(hpos, it, 0); }
float y(const MP3F* hpos, int it)    { return Pos(hpos, it, 1); }
float z(const MP3F* hpos, int it)    { return Pos(hpos, it, 2); }
//
float Pos(const MPHIT* hits, int it, int ipar){
  return Pos(&(*hits).pos,it,ipar);
}
float x(const MPHIT* hits, int it)    { return Pos(hits, it, 0); }
float y(const MPHIT* hits, int it)    { return Pos(hits, it, 1); }
float z(const MPHIT* hits, int it)    { return Pos(hits, it, 2); }
//
float Pos(const MPHIT* hits, int ev, int tk, int ipar){
  int ib = tk;
  const MPHIT* bhits = bHit(hits, ev, ib);
  int it = 0;
  return Pos(bhits,it,ipar);
}
float x(const MPHIT* hits, int ev, int tk)    { return Pos(hits, ev, tk, 0); }
float y(const MPHIT* hits, int ev, int tk)    { return Pos(hits, ev, tk, 1); }
float z(const MPHIT* hits, int ev, int tk)    { return Pos(hits, ev, tk, 2); }


////////////////////////////////////////////////////////////////////////
///MAIN compute kernels

__device__ inline void MultHelixPropEndcap(const MP6x6F &a, const MP6x6SF &b, MP6x6F &c) {
  {
    c[ 0] = b[ 0] + a[ 2]*b[ 3] + a[ 3]*b[ 6] + a[ 4]*b[10] + a[ 5]*b[15];
    c[ 1] = b[ 1] + a[ 2]*b[ 4] + a[ 3]*b[ 7] + a[ 4]*b[11] + a[ 5]*b[16];
    c[ 2] = b[ 3] + a[ 2]*b[ 5] + a[ 3]*b[ 8] + a[ 4]*b[12] + a[ 5]*b[17];
    c[ 3] = b[ 6] + a[ 2]*b[ 8] + a[ 3]*b[ 9] + a[ 4]*b[13] + a[ 5]*b[18];
    c[ 4] = b[10] + a[ 2]*b[12] + a[ 3]*b[13] + a[ 4]*b[14] + a[ 5]*b[19];
    c[ 5] = b[15] + a[ 2]*b[17] + a[ 3]*b[18] + a[ 4]*b[19] + a[ 5]*b[20];
    c[ 6] = b[ 1] + a[ 8]*b[ 3] + a[ 9]*b[ 6] + a[10]*b[10] + a[11]*b[15];
    c[ 7] = b[ 2] + a[ 8]*b[ 4] + a[ 9]*b[ 7] + a[10]*b[11] + a[11]*b[16];
    c[ 8] = b[ 4] + a[ 8]*b[ 5] + a[ 9]*b[ 8] + a[10]*b[12] + a[11]*b[17];
    c[ 9] = b[ 7] + a[ 8]*b[ 8] + a[ 9]*b[ 9] + a[10]*b[13] + a[11]*b[18];
    c[10] = b[11] + a[ 8]*b[12] + a[ 9]*b[13] + a[10]*b[14] + a[11]*b[19];
    c[11] = b[16] + a[ 8]*b[17] + a[ 9]*b[18] + a[10]*b[19] + a[11]*b[20];
    c[12] = 0.f;
    c[13] = 0.f;
    c[14] = 0.f;
    c[15] = 0.f;
    c[16] = 0.f;
    c[17] = 0.f;
    c[18] = b[ 6];
    c[19] = b[ 7];
    c[20] = b[ 8];
    c[21] = b[ 9];
    c[22] = b[13];
    c[23] = b[18];
    c[24] = a[26]*b[ 3] + a[27]*b[ 6] + b[10] + a[29]*b[15];
    c[25] = a[26]*b[ 4] + a[27]*b[ 7] + b[11] + a[29]*b[16];
    c[26] = a[26]*b[ 5] + a[27]*b[ 8] + b[12] + a[29]*b[17];
    c[27] = a[26]*b[ 8] + a[27]*b[ 9] + b[13] + a[29]*b[18];
    c[28] = a[26]*b[12] + a[27]*b[13] + b[14] + a[29]*b[19];
    c[29] = a[26]*b[17] + a[27]*b[18] + b[19] + a[29]*b[20];
    c[30] = b[15];
    c[31] = b[16];
    c[32] = b[17];
    c[33] = b[18];
    c[34] = b[19];
    c[35] = b[20];
  }
  return;
}

__device__ inline void MultHelixPropTranspEndcap(const MP6x6F &a, const MP6x6F &b, MP6x6SF &c) {
  {
    c[ 0] = b[ 0] + b[ 2]*a[ 2] + b[ 3]*a[ 3] + b[ 4]*a[ 4] + b[ 5]*a[ 5];
    c[ 1] = b[ 6] + b[ 8]*a[ 2] + b[ 9]*a[ 3] + b[10]*a[ 4] + b[11]*a[ 5];
    c[ 2] = b[ 7] + b[ 8]*a[ 8] + b[ 9]*a[ 9] + b[10]*a[10] + b[11]*a[11];
    c[ 3] = b[12] + b[14]*a[ 2] + b[15]*a[ 3] + b[16]*a[ 4] + b[17]*a[ 5];
    c[ 4] = b[13] + b[14]*a[ 8] + b[15]*a[ 9] + b[16]*a[10] + b[17]*a[11];
    c[ 5] = 0.f;
    c[ 6] = b[18] + b[20]*a[ 2] + b[21]*a[ 3] + b[22]*a[ 4] + b[23]*a[ 5];
    c[ 7] = b[19] + b[20]*a[ 8] + b[21]*a[ 9] + b[22]*a[10] + b[23]*a[11];
    c[ 8] = 0.f;
    c[ 9] = b[21];
    c[10] = b[24] + b[26]*a[ 2] + b[27]*a[ 3] + b[28]*a[ 4] + b[29]*a[ 5];
    c[11] = b[25] + b[26]*a[ 8] + b[27]*a[ 9] + b[28]*a[10] + b[29]*a[11];
    c[12] = 0.f;
    c[13] = b[27];
    c[14] = b[26]*a[26] + b[27]*a[27] + b[28] + b[29]*a[29];
    c[15] = b[30] + b[32]*a[ 2] + b[33]*a[ 3] + b[34]*a[ 4] + b[35]*a[ 5];
    c[16] = b[31] + b[32]*a[ 8] + b[33]*a[ 9] + b[34]*a[10] + b[35]*a[11];
    c[17] = 0.f;
    c[18] = b[33];
    c[19] = b[32]*a[26] + b[33]*a[27] + b[34] + b[35]*a[29];
    c[20] = b[35];
  }
  return;
}

__device__ inline void KalmanGainInv(const MP6x6SF &a, const MP3x3SF &b, MP3x3 &c) {

  {
    double det =
      ((a[0]+b[0])*(((a[ 6]+b[ 3]) *(a[11]+b[5])) - ((a[7]+b[4]) *(a[7]+b[4])))) -
      ((a[1]+b[1])*(((a[ 1]+b[ 1]) *(a[11]+b[5])) - ((a[7]+b[4]) *(a[2]+b[2])))) +
      ((a[2]+b[2])*(((a[ 1]+b[ 1]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[6]+b[3]))));
    double invdet = 1.0/det;

    c[ 0] =   invdet*(((a[ 6]+b[ 3]) *(a[11]+b[5])) - ((a[7]+b[4]) *(a[7]+b[4])));
    c[ 1] =  -invdet*(((a[ 1]+b[ 1]) *(a[11]+b[5])) - ((a[2]+b[2]) *(a[7]+b[4])));
    c[ 2] =   invdet*(((a[ 1]+b[ 1]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[7]+b[4])));
    c[ 3] =  -invdet*(((a[ 1]+b[ 1]) *(a[11]+b[5])) - ((a[7]+b[4]) *(a[2]+b[2])));
    c[ 4] =   invdet*(((a[ 0]+b[ 0]) *(a[11]+b[5])) - ((a[2]+b[2]) *(a[2]+b[2])));
    c[ 5] =  -invdet*(((a[ 0]+b[ 0]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[1]+b[1])));
    c[ 6] =   invdet*(((a[ 1]+b[ 1]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[6]+b[3])));
    c[ 7] =  -invdet*(((a[ 0]+b[ 0]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[1]+b[1])));
    c[ 8] =   invdet*(((a[ 0]+b[ 0]) *(a[6]+b[3])) - ((a[1]+b[1]) *(a[1]+b[1])));
  }
  
  return;
}

__device__ inline void KalmanGain(const MP6x6SF &a, const MP3x3 &b, MP3x6 &c) {

  {
    c[ 0] = a[0]*b[0] + a[ 1]*b[3] + a[2]*b[6];
    c[ 1] = a[0]*b[1] + a[ 1]*b[4] + a[2]*b[7];
    c[ 2] = a[0]*b[2] + a[ 1]*b[5] + a[2]*b[8];
    c[ 3] = a[1]*b[0] + a[ 6]*b[3] + a[7]*b[6];
    c[ 4] = a[1]*b[1] + a[ 6]*b[4] + a[7]*b[7];
    c[ 5] = a[1]*b[2] + a[ 6]*b[5] + a[7]*b[8];
    c[ 6] = a[2]*b[0] + a[ 7]*b[3] + a[11]*b[6];
    c[ 7] = a[2]*b[1] + a[ 7]*b[4] + a[11]*b[7];
    c[ 8] = a[2]*b[2] + a[ 7]*b[5] + a[11]*b[8];
    c[ 9] = a[3]*b[0] + a[ 8]*b[3] + a[12]*b[6];
    c[10] = a[3]*b[1] + a[ 8]*b[4] + a[12]*b[7];
    c[11] = a[3]*b[2] + a[ 8]*b[5] + a[12]*b[8];
    c[12] = a[4]*b[0] + a[ 9]*b[3] + a[13]*b[6];
    c[13] = a[4]*b[1] + a[ 9]*b[4] + a[13]*b[7];
    c[14] = a[4]*b[2] + a[ 9]*b[5] + a[13]*b[8];
    c[15] = a[5]*b[0] + a[10]*b[3] + a[14]*b[6];
    c[16] = a[5]*b[1] + a[10]*b[4] + a[14]*b[7];
    c[17] = a[5]*b[2] + a[10]*b[5] + a[14]*b[8];
  }
  
  return;
}

__device__ void KalmanUpdate(MP6x6SF &trkErr, MP6F &inPar, const MP3x3SF &hitErr, const MP3F &msP){

  MP3x3 inverse_temp;
  MP3x6 kGain;
  MP6x6SF newErr;
  
  KalmanGainInv(trkErr, hitErr, inverse_temp);
  KalmanGain(trkErr, inverse_temp, kGain);

  {
    const auto xin     = inPar[iparX];
    const auto yin     = inPar[iparY];
    const auto zin     = inPar[iparZ];
    const auto ptin    = 1.f/ inPar[iparIpt];
    const auto phiin   = inPar[iparPhi];
    const auto thetain = inPar[iparTheta];
    const auto xout    = msP[iparX];
    const auto yout    = msP[iparY];
    //const auto zout    = msP[iparZ];

    auto xnew     = xin + (kGain[0]*(xout-xin)) +(kGain[1]*(yout-yin)); 
    auto ynew     = yin + (kGain[3]*(xout-xin)) +(kGain[4]*(yout-yin)); 
    auto znew     = zin + (kGain[6]*(xout-xin)) +(kGain[7]*(yout-yin)); 
    auto ptnew    = ptin + (kGain[9]*(xout-xin)) +(kGain[10]*(yout-yin)); 
    auto phinew   = phiin + (kGain[12]*(xout-xin)) +(kGain[13]*(yout-yin)); 
    auto thetanew = thetain + (kGain[15]*(xout-xin)) +(kGain[16]*(yout-yin)); 

    newErr[ 0] = trkErr[ 0] - (kGain[ 0]*trkErr[0]+kGain[1]*trkErr[1]+kGain[2]*trkErr[2]);
    newErr[ 1] = trkErr[ 1] - (kGain[ 0]*trkErr[1]+kGain[1]*trkErr[6]+kGain[2]*trkErr[7]);
    newErr[ 2] = trkErr[ 2] - (kGain[ 0]*trkErr[2]+kGain[1]*trkErr[7]+kGain[2]*trkErr[11]);
    newErr[ 3] = trkErr[ 3] - (kGain[ 0]*trkErr[3]+kGain[1]*trkErr[8]+kGain[2]*trkErr[12]);
    newErr[ 4] = trkErr[ 4] - (kGain[ 0]*trkErr[4]+kGain[1]*trkErr[9]+kGain[2]*trkErr[13]);
    newErr[ 5] = trkErr[ 5] - (kGain[ 0]*trkErr[5]+kGain[1]*trkErr[10]+kGain[2]*trkErr[14]);

    newErr[ 6] = trkErr[ 6] - (kGain[ 3]*trkErr[1]+kGain[4]*trkErr[6]+kGain[5]*trkErr[7]);
    newErr[ 7] = trkErr[ 7] - (kGain[ 3]*trkErr[2]+kGain[4]*trkErr[7]+kGain[5]*trkErr[11]);
    newErr[ 8] = trkErr[ 8] - (kGain[ 3]*trkErr[3]+kGain[4]*trkErr[8]+kGain[5]*trkErr[12]);
    newErr[ 9] = trkErr[ 9] - (kGain[ 3]*trkErr[4]+kGain[4]*trkErr[9]+kGain[5]*trkErr[13]);
    newErr[10] = trkErr[10] - (kGain[ 3]*trkErr[5]+kGain[4]*trkErr[10]+kGain[5]*trkErr[14]);

    newErr[11] = trkErr[11] - (kGain[ 6]*trkErr[2]+kGain[7]*trkErr[7]+kGain[8]*trkErr[11]);
    newErr[12] = trkErr[12] - (kGain[ 6]*trkErr[3]+kGain[7]*trkErr[8]+kGain[8]*trkErr[12]);
    newErr[13] = trkErr[13] - (kGain[ 6]*trkErr[4]+kGain[7]*trkErr[9]+kGain[8]*trkErr[13]);
    newErr[14] = trkErr[14] - (kGain[ 6]*trkErr[5]+kGain[7]*trkErr[10]+kGain[8]*trkErr[14]);

    newErr[15] = trkErr[15] - (kGain[ 9]*trkErr[3]+kGain[10]*trkErr[8]+kGain[11]*trkErr[12]);
    newErr[16] = trkErr[16] - (kGain[ 9]*trkErr[4]+kGain[10]*trkErr[9]+kGain[11]*trkErr[13]);
    newErr[17] = trkErr[17] - (kGain[ 9]*trkErr[5]+kGain[10]*trkErr[10]+kGain[11]*trkErr[14]);

    newErr[18] = trkErr[18] - (kGain[12]*trkErr[4]+kGain[13]*trkErr[9]+kGain[14]*trkErr[13]);
    newErr[19] = trkErr[19] - (kGain[12]*trkErr[5]+kGain[13]*trkErr[10]+kGain[14]*trkErr[14]);

    newErr[20] = trkErr[20] - (kGain[15]*trkErr[5]+kGain[16]*trkErr[10]+kGain[17]*trkErr[14]);
    
    inPar[iparX]     = xnew;
    inPar[iparY]     = ynew;
    inPar[iparZ]     = znew;
    inPar[iparIpt]   = ptnew;
    inPar[iparPhi]   = phinew;
    inPar[iparTheta] = thetanew;
    
 #pragma unroll
    for (int i = 0; i < 21; i++){
      trkErr[ i] = trkErr[ i] - newErr[ i];
    }

  }
  
  return;
}              

//constexpr auto kfact= 100/(-0.299792458*3.8112);
constexpr auto kfact= 100/3.8;

__device__ void propagateToZ(const MP6x6SF &inErr, const MP6F &inPar, const MP1I &inChg, 
                  const MP3F &msP, MP6x6SF &outErr, MP6F &outPar) {
  
  MP6x6F errorProp;
  MP6x6F temp;

  auto PosInMtrx = [=] (int i, int j, int D) constexpr {return (i*D+j);};
//#pragma omp simd
  {	
    const auto zout = msP[iparZ];
    //note: in principle charge is not needed and could be the sign of ipt
    const auto k = inChg[0]*kfact;
    const auto deltaZ = zout - inPar[iparZ];
    const auto ipt  = inPar[iparIpt];
    const auto pt   = 1.f/ipt;
    const auto phi  = inPar[iparPhi];
    const auto cosP = cosf(phi);
    const auto sinP = sinf(phi);
    const auto theta= inPar[iparTheta];
    const auto cosT = cosf(theta);
    const auto sinT = sinf(theta);
    const auto pxin = cosP*pt;
    const auto pyin = sinP*pt;
    const auto icosT  = 1.f/cosT;
    const auto icosTk = icosT/k;
    const auto alpha  = deltaZ*sinT*ipt*icosTk;
    //const auto alpha = deltaZ*sinT*ipt(inPar]/(cosT*k);
    const auto sina = sinf(alpha); // this can be approximated;
    const auto cosa = cosf(alpha); // this can be approximated;
    //
    outPar[iparX]     = inPar[iparX] + k*(pxin*sina - pyin*(1.f-cosa));
    outPar[iparY]     = inPar[iparY] + k*(pyin*sina + pxin*(1.f-cosa));
    outPar[iparZ]     = zout;
    outPar[iparIpt]   = ipt;
    outPar[iparPhi]   = phi +alpha;
    outPar[iparTheta] = theta;
    
    const auto sCosPsina = sinf(cosP*sina);
    const auto cCosPsina = cosf(cosP*sina);
    
    //for (size_t i=0;i<6;++i) errorProp[bsize*PosInMtrx(i,i,6) + it] = 1.;
    errorProp[PosInMtrx(0,0,6)] = 1.0f;
    errorProp[PosInMtrx(1,1,6)] = 1.0f;
    errorProp[PosInMtrx(2,2,6)] = 1.0f;
    errorProp[PosInMtrx(3,3,6)] = 1.0f;
    errorProp[PosInMtrx(4,4,6)] = 1.0f;
    errorProp[PosInMtrx(5,5,6)] = 1.0f;
    //
    errorProp[PosInMtrx(0,1,6)] = 0.f;
    errorProp[PosInMtrx(0,2,6)] = cosP*sinT*(sinP*cosa*sCosPsina-cosa)*icosT;
    errorProp[PosInMtrx(0,3,6)] = cosP*sinT*deltaZ*cosa*(1.f-sinP*sCosPsina)*(icosT*pt)-k*(cosP*sina-sinP*(1.f-cCosPsina))*(pt*pt);
    errorProp[PosInMtrx(0,4,6)] = (k*pt)*(-sinP*sina+sinP*sinP*sina*sCosPsina-cosP*(1.f-cCosPsina));
    errorProp[PosInMtrx(0,5,6)] = cosP*deltaZ*cosa*(1.f-sinP*sCosPsina)*(icosT*icosT);
    errorProp[PosInMtrx(1,2,6)] = cosa*sinT*(cosP*cosP*sCosPsina-sinP)*icosT;
    errorProp[PosInMtrx(1,3,6)] = sinT*deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*pt)-k*(sinP*sina+cosP*(1.f-cCosPsina))*(pt*pt);
    errorProp[PosInMtrx(1,4,6)] = (k*pt)*(-sinP*(1.f-cCosPsina)-sinP*cosP*sina*sCosPsina+cosP*sina);
    errorProp[PosInMtrx(1,5,6)] = deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*icosT);
    errorProp[PosInMtrx(4,2,6)] = -ipt*sinT*(icosTk);//!
    errorProp[PosInMtrx(4,3,6)] = sinT*deltaZ*(icosTk);
    errorProp[PosInMtrx(4,5,6)] = ipt*deltaZ*(icosT*icosTk);//!
  }
  
  MultHelixPropEndcap(errorProp, inErr, temp);
  MultHelixPropTranspEndcap(errorProp, temp, outErr);
  
  return;
}


template <bool grid_stride = true>
__global__ void launch_p2z_kernels(MPTRK *obtracks_, MPTRK *btracks_, MPHIT *bhits_, const int length){
   auto i = threadIdx.x + blockIdx.x * blockDim.x;

   while (i < length) {
     //
     MPTRK obtracks;
     //
     const MPTRK btracks = btracks_[i];
     //
     for(int layer=0; layer<nlayer; ++layer) {  
       //
       const MPHIT bhits = bhits_[layer+nlayer*i];
       //
       propagateToZ(btracks.cov, btracks.par, btracks.q, bhits.pos, obtracks.cov, obtracks.par);
       KalmanUpdate(obtracks.cov, obtracks.par, bhits.cov, bhits.pos);
       //
     }
     //
     obtracks_[i] = obtracks;
     
     if constexpr (grid_stride) i += gridDim.x * blockDim.x;
     else break;
  }
  return;
}


int main (int argc, char* argv[]) {

   #include "input_track.h"

   std::vector<AHIT> inputhits{inputhit21,inputhit20,inputhit19,inputhit18,inputhit17,inputhit16,inputhit15,inputhit14,
                               inputhit13,inputhit12,inputhit11,inputhit10,inputhit09,inputhit08,inputhit07,inputhit06,
                               inputhit05,inputhit04,inputhit03,inputhit02,inputhit01,inputhit00};

   printf("track in pos: x=%f, y=%f, z=%f, r=%f, pt=%f, phi=%f, theta=%f \n", inputtrk.par[0], inputtrk.par[1], inputtrk.par[2],
	  sqrtf(inputtrk.par[0]*inputtrk.par[0] + inputtrk.par[1]*inputtrk.par[1]),
	  1./inputtrk.par[3], inputtrk.par[4], inputtrk.par[5]);
   printf("track in cov: xx=%.2e, yy=%.2e, zz=%.2e \n", inputtrk.cov[SymOffsets66[0]],
	                                       inputtrk.cov[SymOffsets66[(1*6+1)]],
	                                       inputtrk.cov[SymOffsets66[(2*6+2)]]);
   for (int lay=0; lay<nlayer; lay++){
     printf("hit in layer=%lu, pos: x=%f, y=%f, z=%f, r=%f \n", lay, inputhits[lay].pos[0], inputhits[lay].pos[1], inputhits[lay].pos[2], sqrtf(inputhits[lay].pos[0]*inputhits[lay].pos[0] + inputhits[lay].pos[1]*inputhits[lay].pos[1]));
   }
   
   printf("produce nevts=%i ntrks=%i smearing by=%f \n", nevts, ntrks, smear);
   printf("NITER=%d\n", NITER);

   long setup_start, setup_stop;
   struct timeval timecheck;

   auto dev_id = p2z_get_compute_device_id();
   auto streams= p2z_get_streams(nstreams);

   auto stream = streams[0];//with UVM, we use only one compute stream    
   //
   std::vector<MPTRK, MPTRKAllocator> outtrcks(nevts*ntrks);
   // migrate output object to dev memory:
   p2z_prefetch<MPTRK, MPTRKAllocator>(outtrcks, dev_id, stream);

   std::vector<MPTRK, MPTRKAllocator > trcks(nevts*ntrks); 
   prepareTracks<MPTRKAllocator>(trcks, inputtrk);
   //
   std::vector<MPHIT, MPHITAllocator> hits(nlayer*nevts*ntrks);
   prepareHits<MPHITAllocator>(hits, inputhits);
   //
   //
   if constexpr (include_data_transfer == false) {
     p2z_prefetch<MPTRK, MPTRKAllocator>(trcks, dev_id, stream);
     p2z_prefetch<MPHIT, MPHITAllocator>(hits,  dev_id, stream);
   }

   // synchronize to ensure that all needed data is on the device:
   p2z_wait();

   gettimeofday(&timecheck, NULL);
   setup_stop = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;

   printf("done preparing!\n");

   printf("Size of struct MPTRK trk[] = %ld\n", nevts*ntrks*sizeof(MPTRK));
   printf("Size of struct MPTRK outtrk[] = %ld\n", nevts*ntrks*sizeof(MPTRK));
   printf("Size of struct struct MPHIT hit[] = %ld\n", nevts*ntrks*sizeof(MPHIT));

   const int phys_length      = nevts*ntrks;
   const int outer_loop_range = phys_length;
   //
   dim3 blocks(threadsperblock, 1, 1);
   dim3 grid(((outer_loop_range + threadsperblock - 1)/ threadsperblock),1,1);

   double wall_time = 0.0;

   for(int itr=0; itr<NITER; itr++) {
     auto wall_start = std::chrono::high_resolution_clock::now();
     //
     if constexpr (include_data_transfer) {
       p2z_prefetch<MPTRK, MPTRKAllocator>(trcks, dev_id, stream);
       p2z_prefetch<MPHIT, MPHITAllocator>(hits,  dev_id, stream);
     }

     launch_p2z_kernels<<<grid, blocks, 0, stream>>>(outtrcks.data(), trcks.data(), hits.data(), phys_length);

     if constexpr (include_data_transfer) {
       p2z_prefetch<MPTRK, MPTRKAllocator>(outtrcks, host_id, stream);
     }
     //
     p2z_wait();
     //
     auto wall_stop = std::chrono::high_resolution_clock::now();
     //
     auto wall_diff = wall_stop - wall_start;
     //
     wall_time += static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(wall_diff).count()) / 1e6;
     // reset initial states (don't need if we won't measure data migrations):
     if constexpr (include_data_transfer) {

       p2z_prefetch<MPTRK, MPTRKAllocator>(trcks, host_id, stream);
       p2z_prefetch<MPHIT, MPHITAllocator>(hits,  host_id, stream);
       //
       p2z_prefetch<MPTRK, MPTRKAllocator, decltype(stream), true>(outtrcks, dev_id, stream);
     }
   } //end of itr loop

   printf("setup time time=%f (s)\n", (setup_stop-setup_start)*0.001);
   printf("done ntracks=%i tot time=%f (s) time/trk=%e (s)\n", nevts*ntrks*int(NITER), wall_time, wall_time/(nevts*ntrks*int(NITER)));
   printf("formatted %i %i %i %i %i %f 0 %f %i\n",int(NITER),nevts, ntrks, 1, ntrks, wall_time, (setup_stop-setup_start)*0.001, -1);

   auto outtrk = outtrcks.data();

   int nnans = 0, nfail = 0;
   float avgx = 0, avgy = 0, avgz = 0, avgr = 0;
   float avgpt = 0, avgphi = 0, avgtheta = 0;
   float avgdx = 0, avgdy = 0, avgdz = 0, avgdr = 0;

   for (int ie=0;ie<nevts;++ie) {
     for (int it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       float r_ = sqrtf(x_*x_ + y_*y_);
       float pt_ = std::abs(1./ipt(outtrk,ie,it));
       float phi_ = phi(outtrk,ie,it);
       float theta_ = theta(outtrk,ie,it);
       float hx_ = inputhits[nlayer-1].pos[0];
       float hy_ = inputhits[nlayer-1].pos[1];
       float hz_ = inputhits[nlayer-1].pos[2];
       float hr_ = sqrtf(hx_*hx_ + hy_*hy_);
 
       if (std::isfinite(x_)==false ||
          std::isfinite(y_)==false ||
          std::isfinite(z_)==false ||
          std::isfinite(pt_)==false ||
          std::isfinite(phi_)==false ||
          std::isfinite(theta_)==false
          ) {
        nnans++;
        continue;
       }
       if (fabs( (x_-hx_)/hx_ )>1. ||
	   fabs( (y_-hy_)/hy_ )>1. ||
	   fabs( (z_-hz_)/hz_ )>1.) {
	 nfail++;
	 continue;
       }

       avgpt += pt_;
       avgphi += phi_;
       avgtheta += theta_;
       avgx += x_;
       avgy += y_;
       avgz += z_;
       avgr += r_;
       avgdx += (x_-hx_)/x_;
       avgdy += (y_-hy_)/y_;
       avgdz += (z_-hz_)/z_;
       avgdr += (r_-hr_)/r_;
       //if((it+ie*ntrks) < 64) printf("iTrk = %i,  track (x,y,z,r)=(%.6f,%.6f,%.6f,%.6f) \n", it+ie*ntrks, x_,y_,z_,r_);
     }
   }

   avgpt = avgpt/float(nevts*ntrks);
   avgphi = avgphi/float(nevts*ntrks);
   avgtheta = avgtheta/float(nevts*ntrks);
   avgx = avgx/float(nevts*ntrks);
   avgy = avgy/float(nevts*ntrks);
   avgz = avgz/float(nevts*ntrks);
   avgr = avgr/float(nevts*ntrks);
   avgdx = avgdx/float(nevts*ntrks);
   avgdy = avgdy/float(nevts*ntrks);
   avgdz = avgdz/float(nevts*ntrks);
   avgdr = avgdr/float(nevts*ntrks);

   float stdx = 0, stdy = 0, stdz = 0, stdr = 0;
   float stddx = 0, stddy = 0, stddz = 0, stddr = 0;
   for (int ie=0;ie<nevts;++ie) {
     for (int it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       float r_ = sqrtf(x_*x_ + y_*y_);
       float hx_ = inputhits[nlayer-1].pos[0];
       float hy_ = inputhits[nlayer-1].pos[1];
       float hz_ = inputhits[nlayer-1].pos[2];
       float hr_ = sqrtf(hx_*hx_ + hy_*hy_);
       if (std::isfinite(x_)==false ||
          std::isfinite(y_)==false ||
          std::isfinite(z_)==false
          ) {
        continue;
       }
       if (fabs( (x_-hx_)/hx_ )>1. ||
	   fabs( (y_-hy_)/hy_ )>1. ||
	   fabs( (z_-hz_)/hz_ )>1.) {
	 continue;
       }
       stdx += (x_-avgx)*(x_-avgx);
       stdy += (y_-avgy)*(y_-avgy);
       stdz += (z_-avgz)*(z_-avgz);
       stdr += (r_-avgr)*(r_-avgr);
       stddx += ((x_-hx_)/x_-avgdx)*((x_-hx_)/x_-avgdx);
       stddy += ((y_-hy_)/y_-avgdy)*((y_-hy_)/y_-avgdy);
       stddz += ((z_-hz_)/z_-avgdz)*((z_-hz_)/z_-avgdz);
       stddr += ((r_-hr_)/r_-avgdr)*((r_-hr_)/r_-avgdr);
     }
   }

   stdx = sqrtf(stdx/float(nevts*ntrks));
   stdy = sqrtf(stdy/float(nevts*ntrks));
   stdz = sqrtf(stdz/float(nevts*ntrks));
   stdr = sqrtf(stdr/float(nevts*ntrks));
   stddx = sqrtf(stddx/float(nevts*ntrks));
   stddy = sqrtf(stddy/float(nevts*ntrks));
   stddz = sqrtf(stddz/float(nevts*ntrks));
   stddr = sqrtf(stddr/float(nevts*ntrks));

   printf("track x avg=%f std/avg=%f\n", avgx, fabs(stdx/avgx));
   printf("track y avg=%f std/avg=%f\n", avgy, fabs(stdy/avgy));
   printf("track z avg=%f std/avg=%f\n", avgz, fabs(stdz/avgz));
   printf("track r avg=%f std/avg=%f\n", avgr, fabs(stdr/avgz));
   printf("track dx/x avg=%f std=%f\n", avgdx, stddx);
   printf("track dy/y avg=%f std=%f\n", avgdy, stddy);
   printf("track dz/z avg=%f std=%f\n", avgdz, stddz);
   printf("track dr/r avg=%f std=%f\n", avgdr, stddr);
   printf("track pt avg=%f\n", avgpt);
   printf("track phi avg=%f\n", avgphi);
   printf("track theta avg=%f\n", avgtheta);
   printf("number of tracks with nans=%i\n", nnans);
   printf("number of tracks failed=%i\n", nfail);

   return 0;
}
