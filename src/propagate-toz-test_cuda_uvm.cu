#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <iostream>
#include <chrono>
#include <iomanip>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <cassert>

#include <algorithm>
#include <vector>
#include <memory>
#include <numeric>
#include <random>


#ifndef ntrks
#define ntrks 9600
#endif

#ifndef bsize
#define bsize 32
#endif

#define nb    (ntrks/bsize)

#ifndef nevts
#define nevts 100
#endif
#define smear 0.1

#ifndef NITER
#define NITER 5
#endif
#ifndef nlayer
#define nlayer 20
#endif

#ifndef num_streams
#define num_streams 1
#endif

#ifndef threadsperblock
#define threadsperblock 32
#endif

#ifdef include_data
constexpr bool include_data_transfer = true;
#else
constexpr bool include_data_transfer = false;
#endif

static int nstreams  = num_streams;//we have only one stream, though

constexpr int host_id = -1; /*hipCpuDeviceId*/

namespace impl {

  template<typename Tp>
  struct UVMAllocator {
    public:

      typedef Tp value_type;

      UVMAllocator () {};

      UVMAllocator(const UVMAllocator&) { }
       
      template<typename Tp1> constexpr UVMAllocator(const UVMAllocator<Tp1>&) { }

      ~UVMAllocator() { }

      Tp* address(Tp& x) const { return &x; }

      std::size_t  max_size() const throw() { return size_t(-1) / sizeof(Tp); }

      [[nodiscard]] Tp* allocate(std::size_t n){

        Tp* ptr = nullptr;

        auto err = hipMallocManaged((void **)&ptr,n*sizeof(Tp));

        if( err != hipSuccess ) {
          ptr = (Tp *) NULL;
          std::cerr << " hipMallocManaged failed for " << n*sizeof(Tp) << " bytes " <<hipGetErrorString(err)<< std::endl;
          assert(0);
        }

        return ptr;
      }
      void deallocate( Tp* p, std::size_t n) noexcept {
        hipFree((void *)p);
        return;
      }
    };
} //impl

//Collection of API functions:
int p2z_get_compute_device_id(){
  int dev = -1;
  hipGetDevice(&dev);
  hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  return dev;
}

void p2z_check_error(){
  //
  auto error = hipGetLastError();
  if(error != hipSuccess) std::cout << "Error detected, error " << error << std::endl;
  //
  return;
}

decltype(auto) p2z_get_streams(const int n){
  std::vector<hipStream_t> streams;
  streams.reserve(n);
  for (int i = 0; i < n; i++) {
    hipStream_t stream;
    hipStreamCreate(&stream);
    streams.push_back(stream);
  }
  return streams;
}

template <typename data_tp, typename Allocator, typename stream_t, bool is_sync = false>
void p2z_prefetch(std::vector<data_tp, Allocator> &v, int devId, stream_t stream) {
  hipMemPrefetchAsync(v.data(), v.size() * sizeof(data_tp), devId, stream);
  //
  if constexpr (is_sync) {hipStreamSynchronize(stream);}

  return;
}

void p2z_wait() {
  hipDeviceSynchronize();
  return;
}

const std::array<int, 36> SymOffsets66{0, 1, 3, 6, 10, 15, 1, 2, 4, 7, 11, 16, 3, 4, 5, 8, 12, 17, 6, 7, 8, 9, 13, 18, 10, 11, 12, 13, 14, 19, 15, 16, 17, 18, 19, 20};

struct ATRK {
  std::array<float,6> par;
  std::array<float,21> cov;
  int q;
};

struct AHIT {
  std::array<float,3> pos;
  std::array<float,6> cov;
};

constexpr int iparX     = 0;
constexpr int iparY     = 1;
constexpr int iparZ     = 2;
constexpr int iparIpt   = 3;
constexpr int iparPhi   = 4;
constexpr int iparTheta = 5;

template <typename T, int N, int bSize = 1>
struct MPNX {
   T data[N*bSize];

   MPNX() = default;
   MPNX(const MPNX<T, N, bSize> &) = default;
   MPNX(MPNX<T, N, bSize> &&)      = default;

   //basic accessors   
   constexpr T &operator[](int i) { return data[i]; }
   constexpr const T &operator[](int i) const { return data[i]; }
   constexpr int size() const { return N*bSize; }   
   //
   
   __device__ __host__ inline void load(MPNX<T, N, 1>& dst, const int b) const {
#pragma unroll
     for (int ip=0;ip<N;++ip) { //block load   	
    	dst.data[ip] = data[ip*bSize + b]; 
     }
     
     return;
   }

   __device__ __host__ inline void save(const MPNX<T, N, 1>& src, const int b) {
#pragma unroll
     for (int ip=0;ip<N;++ip) {    	
    	 data[ip*bSize + b] = src.data[ip]; 
     }
     
     return;
   }  
   
   auto operator=(const MPNX&) -> MPNX& = default;
   auto operator=(MPNX&&     ) -> MPNX& = default; 
};

// external data formats:
using MP1I    = MPNX<int,   1 , bsize>;
using MP1F    = MPNX<float, 1 , bsize>;
using MP2F    = MPNX<float, 2 , bsize>;
using MP3F    = MPNX<float, 3 , bsize>;
using MP6F    = MPNX<float, 6 , bsize>;
using MP2x2SF = MPNX<float, 3 , bsize>;
using MP3x3SF = MPNX<float, 6 , bsize>;
using MP6x6SF = MPNX<float, 21, bsize>;
using MP6x6F  = MPNX<float, 36, bsize>;
using MP3x3   = MPNX<float, 9 , bsize>;
using MP3x6   = MPNX<float, 18, bsize>;

// internal data formats:
using MP1I_    = MPNX<int,   1 >;
using MP1F_    = MPNX<float, 1 >;
using MP2F_    = MPNX<float, 2 >;
using MP3F_    = MPNX<float, 3 >;
using MP6F_    = MPNX<float, 6 >;
using MP2x2SF_ = MPNX<float, 3 >;
using MP3x3SF_ = MPNX<float, 6 >;
using MP6x6SF_ = MPNX<float, 21>;
using MP6x6F_  = MPNX<float, 36>;
using MP3x3_   = MPNX<float, 9 >;
using MP3x6_   = MPNX<float, 18>;

struct MPTRK_ {
  MP6F_    par;
  MP6x6SF_ cov;
  MP1I_    q;
};

struct MPHIT_ {
  MP3F_    pos;
  MP3x3SF_ cov;
};

struct MPTRK {
  MP6F    par;
  MP6x6SF cov;
  MP1I    q;

  __device__ __host__ inline const auto load_component (const int batch_id) const{//b is a batch idx
  
    MPTRK_ dst;

    this->par.load(dst.par, batch_id);
    this->cov.load(dst.cov, batch_id);
    this->q.load(dst.q, batch_id);
    
    return dst;  
  }
  
  __device__ __host__ inline void save_component(MPTRK_ &src, const int batch_id) {
    this->par.save(src.par, batch_id);
    this->cov.save(src.cov, batch_id);
    this->q.save(src.q, batch_id);
    
    return;
  }  
};

struct MPHIT {
  MP3F    pos;
  MP3x3SF cov;
  //
  __device__ __host__ inline const auto load_component(const int batch_id) const {
    MPHIT_ dst;

    this->pos.load(dst.pos, batch_id);
    this->cov.load(dst.cov, batch_id);
    
    return dst;
  }
};

using MPTRKAllocator = impl::UVMAllocator<MPTRK>;
using MPHITAllocator = impl::UVMAllocator<MPHIT>;

///////////////////////////////////////
//Gen. utils

float randn(float mu, float sigma) {
  float U1, U2, W, mult;
  static float X1, X2;
  static int call = 0;
  if (call == 1) {
    call = !call;
    return (mu + sigma * (float) X2);
  } do {
    U1 = -1 + ((float) rand () / RAND_MAX) * 2;
    U2 = -1 + ((float) rand () / RAND_MAX) * 2;
    W = pow (U1, 2) + pow (U2, 2);
  }
  while (W >= 1 || W == 0); 
  mult = sqrt ((-2 * log (W)) / W);
  X1 = U1 * mult;
  X2 = U2 * mult; 
  call = !call; 
  return (mu + sigma * (float) X1);
}

template<typename MPTRKAllocator>
void prepareTracks(std::vector<MPTRK, MPTRKAllocator> &trcks, ATRK &inputtrk) {
  //
  for (size_t ie=0;ie<nevts;++ie) {
    for (size_t ib=0;ib<nb;++ib) {
      for (size_t it=0;it<bsize;++it) {
        //par
	 for (size_t ip=0;ip<6;++ip) {
	   trcks[ib + nb*ie].par.data[it + ip*bsize] = (1+smear*randn(0,1))*inputtrk.par[ip];
	 }
	 //cov, scaled by factor 100 
	 for (size_t ip=0;ip<21;++ip) {
	   trcks[ib + nb*ie].cov.data[it + ip*bsize] = (1+smear*randn(0,1))*inputtrk.cov[ip];
	 }
	 //q
	 trcks[ib + nb*ie].q.data[it] = inputtrk.q-2*ceil(-0.5 + (float)rand() / RAND_MAX);//can't really smear this or fit will be wrong
        //if((ib + nb*ie)%10==0 ) printf("prep trk index = %i ,track = (%.3f)\n ", ib+nb*ie);
      }
    }
  }
  //
  return;
}

template<typename MPHITAllocator>
void prepareHits(std::vector<MPHIT, MPHITAllocator> &hits, AHIT& inputhit) {
  // store in element order for bunches of bsize matrices (a la matriplex)
  for (size_t lay=0;lay<nlayer;++lay) {
    for (size_t ie=0;ie<nevts;++ie) {
      for (size_t ib=0;ib<nb;++ib) {
        for (size_t it=0;it<bsize;++it) {
          //pos
          for (size_t ip=0;ip<3;++ip) {
            hits[lay+nlayer*(ib + nb*ie)].pos.data[it + ip*bsize] = (1+smear*randn(0,1))*inputhit.pos[ip];
          }
          //cov
          for (size_t ip=0;ip<6;++ip) {
            hits[lay+nlayer*(ib + nb*ie)].cov.data[it + ip*bsize] = (1+smear*randn(0,1))*inputhit.cov[ip];
          }
        }
      }
    }
  }
  return;
}

//////////////////////////////////////////////////////////////////////////////////////
// Aux utils 
MPTRK* bTk(MPTRK* tracks, size_t ev, size_t ib) {
  return &(tracks[ib + nb*ev]);
}

const MPTRK* bTk(const MPTRK* tracks, size_t ev, size_t ib) {
  return &(tracks[ib + nb*ev]);
}

float q(const MP1I* bq, size_t it){
  return (*bq).data[it];
}
//
float par(const MP6F* bpars, size_t it, size_t ipar){
  return (*bpars).data[it + ipar*bsize];
}
float x    (const MP6F* bpars, size_t it){ return par(bpars, it, 0); }
float y    (const MP6F* bpars, size_t it){ return par(bpars, it, 1); }
float z    (const MP6F* bpars, size_t it){ return par(bpars, it, 2); }
float ipt  (const MP6F* bpars, size_t it){ return par(bpars, it, 3); }
float phi  (const MP6F* bpars, size_t it){ return par(bpars, it, 4); }
float theta(const MP6F* bpars, size_t it){ return par(bpars, it, 5); }
//
float par(const MPTRK* btracks, size_t it, size_t ipar){
  return par(&(*btracks).par,it,ipar);
}
float x    (const MPTRK* btracks, size_t it){ return par(btracks, it, 0); }
float y    (const MPTRK* btracks, size_t it){ return par(btracks, it, 1); }
float z    (const MPTRK* btracks, size_t it){ return par(btracks, it, 2); }
float ipt  (const MPTRK* btracks, size_t it){ return par(btracks, it, 3); }
float phi  (const MPTRK* btracks, size_t it){ return par(btracks, it, 4); }
float theta(const MPTRK* btracks, size_t it){ return par(btracks, it, 5); }
//
float par(const MPTRK* tracks, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  const MPTRK* btracks = bTk(tracks, ev, ib);
  size_t it = tk % bsize;
  return par(btracks, it, ipar);
}
float x    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 0); }
float y    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 1); }
float z    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 2); }
float ipt  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 3); }
float phi  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 4); }
float theta(const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 5); }
//

const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib) {
  return &(hits[ib + nb*ev]);
}
const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib,size_t lay) {
return &(hits[lay + (ib*nlayer) +(ev*nlayer*nb)]);
}
//
float Pos(const MP3F* hpos, size_t it, size_t ipar){
  return (*hpos).data[it + ipar*bsize];
}
float x(const MP3F* hpos, size_t it)    { return Pos(hpos, it, 0); }
float y(const MP3F* hpos, size_t it)    { return Pos(hpos, it, 1); }
float z(const MP3F* hpos, size_t it)    { return Pos(hpos, it, 2); }
//
float Pos(const MPHIT* hits, size_t it, size_t ipar){
  return Pos(&(*hits).pos,it,ipar);
}
float x(const MPHIT* hits, size_t it)    { return Pos(hits, it, 0); }
float y(const MPHIT* hits, size_t it)    { return Pos(hits, it, 1); }
float z(const MPHIT* hits, size_t it)    { return Pos(hits, it, 2); }
//
float Pos(const MPHIT* hits, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  const MPHIT* bhits = bHit(hits, ev, ib, nlayer-1);
  size_t it = tk % bsize;
  return Pos(bhits,it,ipar);
}
float x(const MPHIT* hits, size_t ev, size_t tk)    { return Pos(hits, ev, tk, 0); }
float y(const MPHIT* hits, size_t ev, size_t tk)    { return Pos(hits, ev, tk, 1); }
float z(const MPHIT* hits, size_t ev, size_t tk)    { return Pos(hits, ev, tk, 2); }


////////////////////////////////////////////////////////////////////////
///MAIN compute kernels

__device__ inline void MultHelixPropEndcap(const MP6x6F_ &a, const MP6x6SF_ &b, MP6x6F_ &c) {
 
  c[ 0] = b[ 0] + a[ 2]*b[ 3] + a[ 3]*b[ 6] + a[ 4]*b[10] + a[ 5]*b[15];
  c[ 1] = b[ 1] + a[ 2]*b[ 4] + a[ 3]*b[ 7] + a[ 4]*b[11] + a[ 5]*b[16];
  c[ 2] = b[ 3] + a[ 2]*b[ 5] + a[ 3]*b[ 8] + a[ 4]*b[12] + a[ 5]*b[17];
  c[ 3] = b[ 6] + a[ 2]*b[ 8] + a[ 3]*b[ 9] + a[ 4]*b[13] + a[ 5]*b[18];
  c[ 4] = b[10] + a[ 2]*b[12] + a[ 3]*b[13] + a[ 4]*b[14] + a[ 5]*b[19];
  c[ 5] = b[15] + a[ 2]*b[17] + a[ 3]*b[18] + a[ 4]*b[19] + a[ 5]*b[20];
  c[ 6] = b[ 1] + a[ 8]*b[ 3] + a[ 9]*b[ 6] + a[10]*b[10] + a[11]*b[15];
  c[ 7] = b[ 2] + a[ 8]*b[ 4] + a[ 9]*b[ 7] + a[10]*b[11] + a[11]*b[16];
  c[ 8] = b[ 4] + a[ 8]*b[ 5] + a[ 9]*b[ 8] + a[10]*b[12] + a[11]*b[17];
  c[ 9] = b[ 7] + a[ 8]*b[ 8] + a[ 9]*b[ 9] + a[10]*b[13] + a[11]*b[18];
  c[10] = b[11] + a[ 8]*b[12] + a[ 9]*b[13] + a[10]*b[14] + a[11]*b[19];
  c[11] = b[16] + a[ 8]*b[17] + a[ 9]*b[18] + a[10]*b[19] + a[11]*b[20];
  c[12] = 0.f;
  c[13] = 0.f;
  c[14] = 0.f;
  c[15] = 0.f;
  c[16] = 0.f;
  c[17] = 0.f;
  c[18] = b[ 6];
  c[19] = b[ 7];
  c[20] = b[ 8];
  c[21] = b[ 9];
  c[22] = b[13];
  c[23] = b[18];
  c[24] = a[26]*b[ 3] + a[27]*b[ 6] + b[10] + a[29]*b[15];
  c[25] = a[26]*b[ 4] + a[27]*b[ 7] + b[11] + a[29]*b[16];
  c[26] = a[26]*b[ 5] + a[27]*b[ 8] + b[12] + a[29]*b[17];
  c[27] = a[26]*b[ 8] + a[27]*b[ 9] + b[13] + a[29]*b[18];
  c[28] = a[26]*b[12] + a[27]*b[13] + b[14] + a[29]*b[19];
  c[29] = a[26]*b[17] + a[27]*b[18] + b[19] + a[29]*b[20];
  c[30] = b[15];
  c[31] = b[16];
  c[32] = b[17];
  c[33] = b[18];
  c[34] = b[19];
  c[35] = b[20];
 
  return;
}

__device__ inline void MultHelixPropTranspEndcap(const MP6x6F_ &a, const MP6x6F_ &b, MP6x6SF_ &c) {
  
  c[ 0] = b[ 0] + b[ 2]*a[ 2] + b[ 3]*a[ 3] + b[ 4]*a[ 4] + b[ 5]*a[ 5];
  c[ 1] = b[ 6] + b[ 8]*a[ 2] + b[ 9]*a[ 3] + b[10]*a[ 4] + b[11]*a[ 5];
  c[ 2] = b[ 7] + b[ 8]*a[ 8] + b[ 9]*a[ 9] + b[10]*a[10] + b[11]*a[11];
  c[ 3] = b[12] + b[14]*a[ 2] + b[15]*a[ 3] + b[16]*a[ 4] + b[17]*a[ 5];
  c[ 4] = b[13] + b[14]*a[ 8] + b[15]*a[ 9] + b[16]*a[10] + b[17]*a[11];
  c[ 5] = 0.f;
  c[ 6] = b[18] + b[20]*a[ 2] + b[21]*a[ 3] + b[22]*a[ 4] + b[23]*a[ 5];
  c[ 7] = b[19] + b[20]*a[ 8] + b[21]*a[ 9] + b[22]*a[10] + b[23]*a[11];
  c[ 8] = 0.f;
  c[ 9] = b[21];
  c[10] = b[24] + b[26]*a[ 2] + b[27]*a[ 3] + b[28]*a[ 4] + b[29]*a[ 5];
  c[11] = b[25] + b[26]*a[ 8] + b[27]*a[ 9] + b[28]*a[10] + b[29]*a[11];
  c[12] = 0.f;
  c[13] = b[27];
  c[14] = b[26]*a[26] + b[27]*a[27] + b[28] + b[29]*a[29];
  c[15] = b[30] + b[32]*a[ 2] + b[33]*a[ 3] + b[34]*a[ 4] + b[35]*a[ 5];
  c[16] = b[31] + b[32]*a[ 8] + b[33]*a[ 9] + b[34]*a[10] + b[35]*a[11];
  c[17] = 0.f;
  c[18] = b[33];
  c[19] = b[32]*a[26] + b[33]*a[27] + b[34] + b[35]*a[29];
  c[20] = b[35];
  
  return;
}

__device__ inline void KalmanGainInv(const MP6x6SF_ &a, const MP3x3SF_ &b, MP3x3_ &c) {

  double det =
    ((a[0]+b[0])*(((a[ 6]+b[ 3]) *(a[11]+b[5])) - ((a[7]+b[4]) *(a[7]+b[4])))) -
    ((a[1]+b[1])*(((a[ 1]+b[ 1]) *(a[11]+b[5])) - ((a[7]+b[4]) *(a[2]+b[2])))) +
    ((a[2]+b[2])*(((a[ 1]+b[ 1]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[6]+b[3]))));
  double invdet = 1.0/det;

  c[ 0] =   invdet*(((a[ 6]+b[ 3]) *(a[11]+b[5])) - ((a[7]+b[4]) *(a[7]+b[4])));
  c[ 1] =  -invdet*(((a[ 1]+b[ 1]) *(a[11]+b[5])) - ((a[2]+b[2]) *(a[7]+b[4])));
  c[ 2] =   invdet*(((a[ 1]+b[ 1]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[7]+b[4])));
  c[ 3] =  -invdet*(((a[ 1]+b[ 1]) *(a[11]+b[5])) - ((a[7]+b[4]) *(a[2]+b[2])));
  c[ 4] =   invdet*(((a[ 0]+b[ 0]) *(a[11]+b[5])) - ((a[2]+b[2]) *(a[2]+b[2])));
  c[ 5] =  -invdet*(((a[ 0]+b[ 0]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[1]+b[1])));
  c[ 6] =   invdet*(((a[ 1]+b[ 1]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[6]+b[3])));
  c[ 7] =  -invdet*(((a[ 0]+b[ 0]) *(a[7]+b[4])) - ((a[2]+b[2]) *(a[1]+b[1])));
  c[ 8] =   invdet*(((a[ 0]+b[ 0]) *(a[6]+b[3])) - ((a[1]+b[1]) *(a[1]+b[1])));

  
  return;
}

__device__ inline void KalmanGain(const MP6x6SF_ &a, const MP3x3_ &b, MP3x6_ &c) {

  c[ 0] = a[0]*b[0] + a[ 1]*b[3] + a[2]*b[6];
  c[ 1] = a[0]*b[1] + a[ 1]*b[4] + a[2]*b[7];
  c[ 2] = a[0]*b[2] + a[ 1]*b[5] + a[2]*b[8];
  c[ 3] = a[1]*b[0] + a[ 6]*b[3] + a[7]*b[6];
  c[ 4] = a[1]*b[1] + a[ 6]*b[4] + a[7]*b[7];
  c[ 5] = a[1]*b[2] + a[ 6]*b[5] + a[7]*b[8];
  c[ 6] = a[2]*b[0] + a[ 7]*b[3] + a[11]*b[6];
  c[ 7] = a[2]*b[1] + a[ 7]*b[4] + a[11]*b[7];
  c[ 8] = a[2]*b[2] + a[ 7]*b[5] + a[11]*b[8];
  c[ 9] = a[3]*b[0] + a[ 8]*b[3] + a[12]*b[6];
  c[10] = a[3]*b[1] + a[ 8]*b[4] + a[12]*b[7];
  c[11] = a[3]*b[2] + a[ 8]*b[5] + a[12]*b[8];
  c[12] = a[4]*b[0] + a[ 9]*b[3] + a[13]*b[6];
  c[13] = a[4]*b[1] + a[ 9]*b[4] + a[13]*b[7];
  c[14] = a[4]*b[2] + a[ 9]*b[5] + a[13]*b[8];
  c[15] = a[5]*b[0] + a[10]*b[3] + a[14]*b[6];
  c[16] = a[5]*b[1] + a[10]*b[4] + a[14]*b[7];
  c[17] = a[5]*b[2] + a[10]*b[5] + a[14]*b[8];

  
  return;
}

__device__ void KalmanUpdate(MP6x6SF_ &trkErr, MP6F_ &inPar, const MP3x3SF_ &hitErr, const MP3F_ &msP){

  MP3x3_ inverse_temp;
  MP3x6_ kGain;
  MP6x6SF_ newErr;
  
  KalmanGainInv(trkErr, hitErr, inverse_temp);
  KalmanGain(trkErr, inverse_temp, kGain);

  {
    const auto xin     = inPar[iparX];
    const auto yin     = inPar[iparY];
    const auto zin     = inPar[iparZ];
    const auto ptin    = 1.f/ inPar[iparIpt];
    const auto phiin   = inPar[iparPhi];
    const auto thetain = inPar[iparTheta];
    const auto xout    = msP[iparX];
    const auto yout    = msP[iparY];
    //const auto zout    = msP[iparZ];

    auto xnew     = xin + (kGain[0]*(xout-xin)) +(kGain[1]*(yout-yin)); 
    auto ynew     = yin + (kGain[3]*(xout-xin)) +(kGain[4]*(yout-yin)); 
    auto znew     = zin + (kGain[6]*(xout-xin)) +(kGain[7]*(yout-yin)); 
    auto ptnew    = ptin + (kGain[9]*(xout-xin)) +(kGain[10]*(yout-yin)); 
    auto phinew   = phiin + (kGain[12]*(xout-xin)) +(kGain[13]*(yout-yin)); 
    auto thetanew = thetain + (kGain[15]*(xout-xin)) +(kGain[16]*(yout-yin)); 

    newErr[ 0] = trkErr[ 0] - (kGain[ 0]*trkErr[0]+kGain[1]*trkErr[1]+kGain[2]*trkErr[2]);
    newErr[ 1] = trkErr[ 1] - (kGain[ 0]*trkErr[1]+kGain[1]*trkErr[6]+kGain[2]*trkErr[7]);
    newErr[ 2] = trkErr[ 2] - (kGain[ 0]*trkErr[2]+kGain[1]*trkErr[7]+kGain[2]*trkErr[11]);
    newErr[ 3] = trkErr[ 3] - (kGain[ 0]*trkErr[3]+kGain[1]*trkErr[8]+kGain[2]*trkErr[12]);
    newErr[ 4] = trkErr[ 4] - (kGain[ 0]*trkErr[4]+kGain[1]*trkErr[9]+kGain[2]*trkErr[13]);
    newErr[ 5] = trkErr[ 5] - (kGain[ 0]*trkErr[5]+kGain[1]*trkErr[10]+kGain[2]*trkErr[14]);

    newErr[ 6] = trkErr[ 6] - (kGain[ 3]*trkErr[1]+kGain[4]*trkErr[6]+kGain[5]*trkErr[7]);
    newErr[ 7] = trkErr[ 7] - (kGain[ 3]*trkErr[2]+kGain[4]*trkErr[7]+kGain[5]*trkErr[11]);
    newErr[ 8] = trkErr[ 8] - (kGain[ 3]*trkErr[3]+kGain[4]*trkErr[8]+kGain[5]*trkErr[12]);
    newErr[ 9] = trkErr[ 9] - (kGain[ 3]*trkErr[4]+kGain[4]*trkErr[9]+kGain[5]*trkErr[13]);
    newErr[10] = trkErr[10] - (kGain[ 3]*trkErr[5]+kGain[4]*trkErr[10]+kGain[5]*trkErr[14]);

    newErr[11] = trkErr[11] - (kGain[ 6]*trkErr[2]+kGain[7]*trkErr[7]+kGain[8]*trkErr[11]);
    newErr[12] = trkErr[12] - (kGain[ 6]*trkErr[3]+kGain[7]*trkErr[8]+kGain[8]*trkErr[12]);
    newErr[13] = trkErr[13] - (kGain[ 6]*trkErr[4]+kGain[7]*trkErr[9]+kGain[8]*trkErr[13]);
    newErr[14] = trkErr[14] - (kGain[ 6]*trkErr[5]+kGain[7]*trkErr[10]+kGain[8]*trkErr[14]);

    newErr[15] = trkErr[15] - (kGain[ 9]*trkErr[3]+kGain[10]*trkErr[8]+kGain[11]*trkErr[12]);
    newErr[16] = trkErr[16] - (kGain[ 9]*trkErr[4]+kGain[10]*trkErr[9]+kGain[11]*trkErr[13]);
    newErr[17] = trkErr[17] - (kGain[ 9]*trkErr[5]+kGain[10]*trkErr[10]+kGain[11]*trkErr[14]);

    newErr[18] = trkErr[18] - (kGain[12]*trkErr[4]+kGain[13]*trkErr[9]+kGain[14]*trkErr[13]);
    newErr[19] = trkErr[19] - (kGain[12]*trkErr[5]+kGain[13]*trkErr[10]+kGain[14]*trkErr[14]);

    newErr[20] = trkErr[20] - (kGain[15]*trkErr[5]+kGain[16]*trkErr[10]+kGain[17]*trkErr[14]);
    
    inPar[iparX]     = xnew;
    inPar[iparY]     = ynew;
    inPar[iparZ]     = znew;
    inPar[iparIpt]   = ptnew;
    inPar[iparPhi]   = phinew;
    inPar[iparTheta] = thetanew;
    
 #pragma unroll
    for (int i = 0; i < 21; i++){
      trkErr[ i] = trkErr[ i] - newErr[ i];
    }

  }
  
  return;
}              

//constexpr auto kfact= 100/(-0.299792458*3.8112);
constexpr auto kfact= 100/3.8;

__device__ void propagateToZ(const MP6x6SF_ &inErr, const MP6F_ &inPar, const MP1I_ &inChg, 
                  const MP3F_ &msP, MP6x6SF_ &outErr, MP6F_ &outPar) {
  
  MP6x6F_ errorProp;
  MP6x6F_ temp;

  auto PosInMtrx = [=] (int i, int j, int D) constexpr {return (i*D+j);};
//#pragma omp simd
  {	
    const auto zout = msP[iparZ];
    //note: in principle charge is not needed and could be the sign of ipt
    const auto k = inChg[0]*kfact;
    const auto deltaZ = zout - inPar[iparZ];
    const auto ipt  = inPar[iparIpt];
    const auto pt   = 1.f/ipt;
    const auto phi  = inPar[iparPhi];
    const auto cosP = cosf(phi);
    const auto sinP = sinf(phi);
    const auto theta= inPar[iparTheta];
    const auto cosT = cosf(theta);
    const auto sinT = sinf(theta);
    const auto pxin = cosP*pt;
    const auto pyin = sinP*pt;
    const auto icosT  = 1.f/cosT;
    const auto icosTk = icosT/k;
    const auto alpha  = deltaZ*sinT*ipt*icosTk;
    //const auto alpha = deltaZ*sinT*ipt(inPar]/(cosT*k);
    const auto sina = sinf(alpha); // this can be approximated;
    const auto cosa = cosf(alpha); // this can be approximated;
    //
    outPar[iparX]     = inPar[iparX] + k*(pxin*sina - pyin*(1.f-cosa));
    outPar[iparY]     = inPar[iparY] + k*(pyin*sina + pxin*(1.f-cosa));
    outPar[iparZ]     = zout;
    outPar[iparIpt]   = ipt;
    outPar[iparPhi]   = phi +alpha;
    outPar[iparTheta] = theta;
    
    const auto sCosPsina = sinf(cosP*sina);
    const auto cCosPsina = cosf(cosP*sina);
    
    //for (size_t i=0;i<6;++i) errorProp[bsize*PosInMtrx(i,i,6) + it] = 1.;
    errorProp[PosInMtrx(0,0,6)] = 1.0f;
    errorProp[PosInMtrx(1,1,6)] = 1.0f;
    errorProp[PosInMtrx(2,2,6)] = 1.0f;
    errorProp[PosInMtrx(3,3,6)] = 1.0f;
    errorProp[PosInMtrx(4,4,6)] = 1.0f;
    errorProp[PosInMtrx(5,5,6)] = 1.0f;
    //
    errorProp[PosInMtrx(0,1,6)] = 0.f;
    errorProp[PosInMtrx(0,2,6)] = cosP*sinT*(sinP*cosa*sCosPsina-cosa)*icosT;
    errorProp[PosInMtrx(0,3,6)] = cosP*sinT*deltaZ*cosa*(1.f-sinP*sCosPsina)*(icosT*pt)-k*(cosP*sina-sinP*(1.f-cCosPsina))*(pt*pt);
    errorProp[PosInMtrx(0,4,6)] = (k*pt)*(-sinP*sina+sinP*sinP*sina*sCosPsina-cosP*(1.f-cCosPsina));
    errorProp[PosInMtrx(0,5,6)] = cosP*deltaZ*cosa*(1.f-sinP*sCosPsina)*(icosT*icosT);
    errorProp[PosInMtrx(1,2,6)] = cosa*sinT*(cosP*cosP*sCosPsina-sinP)*icosT;
    errorProp[PosInMtrx(1,3,6)] = sinT*deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*pt)-k*(sinP*sina+cosP*(1.f-cCosPsina))*(pt*pt);
    errorProp[PosInMtrx(1,4,6)] = (k*pt)*(-sinP*(1.f-cCosPsina)-sinP*cosP*sina*sCosPsina+cosP*sina);
    errorProp[PosInMtrx(1,5,6)] = deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*icosT);
    errorProp[PosInMtrx(4,2,6)] = -ipt*sinT*(icosTk);//!
    errorProp[PosInMtrx(4,3,6)] = sinT*deltaZ*(icosTk);
    errorProp[PosInMtrx(4,5,6)] = ipt*deltaZ*(icosT*icosTk);//!
  }
  
  MultHelixPropEndcap(errorProp, inErr, temp);
  MultHelixPropTranspEndcap(errorProp, temp, outErr);
  
  return;
}


template <int bSize, int layers, bool grid_stride = true>
__global__ void launch_p2z_kernels(MPTRK *obtracks_, MPTRK *btracks_, MPHIT *bhits_, const int length){
   auto i = threadIdx.x + blockIdx.x * blockDim.x;

   while (i < length) {
     //
     MPTRK_ obtracks;
     //
     const auto tid        = i / bSize;
     const auto batch_id   = i % bSize;
     //
     const auto& btracks = btracks_[tid].load_component(batch_id);
     //
#pragma unroll     
     for(int layer = 0; layer < layers; ++layer) {  
       //
       const auto& bhits = bhits_[layer+layers*tid].load_component(batch_id);
       //
       propagateToZ(btracks.cov, btracks.par, btracks.q, bhits.pos, obtracks.cov, obtracks.par);
       KalmanUpdate(obtracks.cov, obtracks.par, bhits.cov, bhits.pos);
       //
     }
     //
     obtracks_[tid].save_component(obtracks, batch_id);
     
     if constexpr (grid_stride) i += gridDim.x * blockDim.x;
     else break;
  }
  return;
}


int main (int argc, char* argv[]) {

   ATRK inputtrk = {
     {-12.806846618652344, -7.723824977874756, 38.13014221191406,0.23732035065189902, -2.613372802734375, 0.35594117641448975},
     {6.290299552347278e-07,4.1375109560704004e-08,7.526661534029699e-07,2.0973730840978533e-07,1.5431574240665213e-07,9.626245400795597e-08,-2.804026640189443e-06,
      6.219111130687595e-06,2.649119409845118e-07,0.00253512163402557,-2.419662877381737e-07,4.3124190760040646e-07,3.1068903991780678e-09,0.000923913115050627,
      0.00040678296006807003,-7.755406890332818e-07,1.68539375883925e-06,6.676875566525437e-08,0.0008420574605423793,7.356584799406111e-05,0.0002306247719158348},
     1
   };

   AHIT inputhit = {
     {-20.7824649810791, -12.24150276184082, 57.8067626953125},
     {2.545517190810642e-06,-2.6680759219743777e-06,2.8030024168401724e-06,0.00014160551654640585,0.00012282167153898627,11.385087966918945}
   };

   printf("track in pos: %f, %f, %f \n", inputtrk.par[0], inputtrk.par[1], inputtrk.par[2]);
   printf("track in cov: %.2e, %.2e, %.2e \n", inputtrk.cov[SymOffsets66[(0)]],
                                              inputtrk.cov[SymOffsets66[(1*6+1)]],
                                              inputtrk.cov[SymOffsets66[(2*6+2)]]);
   printf("hit in pos: %f %f %f \n", inputhit.pos[0], inputhit.pos[1], inputhit.pos[2]);
   
   printf("produce nevts=%i ntrks=%i smearing by=%f \n", nevts, ntrks, smear);
   printf("NITER=%d\n", NITER);

   long setup_start, setup_stop;
   struct timeval timecheck;
   
   gettimeofday(&timecheck, NULL);
   setup_start = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;   

   auto dev_id = p2z_get_compute_device_id();
   auto streams= p2z_get_streams(nstreams);

   auto stream = streams[0];//with UVM, we use only one compute stream    
   //
   std::vector<MPTRK, MPTRKAllocator> outtrcks(nevts*nb);
   // migrate output object to dev memory:
   p2z_prefetch<MPTRK, MPTRKAllocator>(outtrcks, dev_id, stream);

   std::vector<MPTRK, MPTRKAllocator > trcks(nevts*nb); 
   prepareTracks<MPTRKAllocator>(trcks, inputtrk);
   //
   std::vector<MPHIT, MPHITAllocator> hits(nlayer*nevts*nb);
   prepareHits<MPHITAllocator>(hits, inputhit);
   //
   //
   if constexpr (include_data_transfer == false) {
     p2z_prefetch<MPTRK, MPTRKAllocator>(trcks, dev_id, stream);
     p2z_prefetch<MPHIT, MPHITAllocator>(hits,  dev_id, stream);
   }

   // synchronize to ensure that all needed data is on the device:
   p2z_wait();

   gettimeofday(&timecheck, NULL);
   setup_stop = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;

   printf("done preparing!\n");

   printf("Size of struct MPTRK trk[] = %ld\n", nevts*nb*sizeof(MPTRK));
   printf("Size of struct MPTRK outtrk[] = %ld\n", nevts*nb*sizeof(MPTRK));
   printf("Size of struct struct MPHIT hit[] = %ld\n", nevts*nb*sizeof(MPHIT));

   const int phys_length      = nevts*nb;
   const int outer_loop_range = phys_length*bsize;
   //
   dim3 blocks(threadsperblock, 1, 1);
   dim3 grid(((outer_loop_range + threadsperblock - 1)/ threadsperblock),1,1);

   double wall_time = 0.0;

   for(int itr=0; itr<NITER; itr++) {
     auto wall_start = std::chrono::high_resolution_clock::now();
     //
     if constexpr (include_data_transfer) {
       p2z_prefetch<MPTRK, MPTRKAllocator>(trcks, dev_id, stream);
       p2z_prefetch<MPHIT, MPHITAllocator>(hits,  dev_id, stream);
     }

     launch_p2z_kernels<bsize, nlayer><<<grid, blocks, 0, stream>>>(outtrcks.data(), trcks.data(), hits.data(), outer_loop_range);

     if constexpr (include_data_transfer) {
       p2z_prefetch<MPTRK, MPTRKAllocator>(outtrcks, host_id, stream);
     }
     //
     p2z_wait();
     //
     auto wall_stop = std::chrono::high_resolution_clock::now();
     //
     auto wall_diff = wall_stop - wall_start;
     //
     wall_time += static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(wall_diff).count()) / 1e6;
     // reset initial states (don't need if we won't measure data migrations):
     if constexpr (include_data_transfer) {

       p2z_prefetch<MPTRK, MPTRKAllocator>(trcks, host_id, stream);
       p2z_prefetch<MPHIT, MPHITAllocator>(hits,  host_id, stream);
       //
       p2z_prefetch<MPTRK, MPTRKAllocator, decltype(stream), true>(outtrcks, dev_id, stream);
     }
   } //end of itr loop

   printf("setup time time=%f (s)\n", (setup_stop-setup_start)*0.001);
   printf("done ntracks=%i tot time=%f (s) time/trk=%e (s)\n", nevts*ntrks*int(NITER), wall_time, wall_time/(nevts*ntrks*int(NITER)));
   printf("formatted %i %i %i %i %i %f 0 %f %i\n",int(NITER),nevts, ntrks, bsize, ntrks, wall_time, (setup_stop-setup_start)*0.001, -1);

   auto outtrk = outtrcks.data();
   auto hit    = hits.data();

   double avgx = 0, avgy = 0, avgz = 0;
   double avgpt = 0, avgphi = 0, avgtheta = 0;
   double avgdx = 0, avgdy = 0, avgdz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       float pt_ = 1./ipt(outtrk,ie,it);
       float phi_ = phi(outtrk,ie,it);
       float theta_ = theta(outtrk,ie,it);
       avgpt += pt_;
       avgphi += phi_;
       avgtheta += theta_;
       avgx += x_;
       avgy += y_;
       avgz += z_;
       float hx_ = x(hit,ie,it);
       float hy_ = y(hit,ie,it);
       float hz_ = z(hit,ie,it);
       avgdx += (x_-hx_)/x_;
       avgdy += (y_-hy_)/y_;
       avgdz += (z_-hz_)/z_;
     }
   }
   avgpt = avgpt/double(nevts*ntrks);
   avgphi = avgphi/double(nevts*ntrks);
   avgtheta = avgtheta/double(nevts*ntrks);
   avgx = avgx/double(nevts*ntrks);
   avgy = avgy/double(nevts*ntrks);
   avgz = avgz/double(nevts*ntrks);
   avgdx = avgdx/double(nevts*ntrks);
   avgdy = avgdy/double(nevts*ntrks);
   avgdz = avgdz/double(nevts*ntrks);

   double stdx = 0, stdy = 0, stdz = 0;
   double stddx = 0, stddy = 0, stddz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       stdx += (x_-avgx)*(x_-avgx);
       stdy += (y_-avgy)*(y_-avgy);
       stdz += (z_-avgz)*(z_-avgz);
       float hx_ = x(hit,ie,it);
       float hy_ = y(hit,ie,it);
       float hz_ = z(hit,ie,it);
       stddx += ((x_-hx_)/x_-avgdx)*((x_-hx_)/x_-avgdx);
       stddy += ((y_-hy_)/y_-avgdy)*((y_-hy_)/y_-avgdy);
       stddz += ((z_-hz_)/z_-avgdz)*((z_-hz_)/z_-avgdz);
     }
   }

   stdx = sqrtf(stdx/double(nevts*ntrks));
   stdy = sqrtf(stdy/double(nevts*ntrks));
   stdz = sqrtf(stdz/double(nevts*ntrks));
   stddx = sqrtf(stddx/double(nevts*ntrks));
   stddy = sqrtf(stddy/double(nevts*ntrks));
   stddz = sqrtf(stddz/double(nevts*ntrks));

   printf("track x avg=%f std/avg=%f\n", avgx, fabs(stdx/avgx));
   printf("track y avg=%f std/avg=%f\n", avgy, fabs(stdy/avgy));
   printf("track z avg=%f std/avg=%f\n", avgz, fabs(stdz/avgz));
   printf("track dx/x avg=%f std=%f\n", avgdx, stddx);
   printf("track dy/y avg=%f std=%f\n", avgdy, stddy);
   printf("track dz/z avg=%f std=%f\n", avgdz, stddz);
   printf("track pt avg=%f\n", avgpt);
   printf("track phi avg=%f\n", avgphi);
   printf("track theta avg=%f\n", avgtheta);

   return 0;
}
